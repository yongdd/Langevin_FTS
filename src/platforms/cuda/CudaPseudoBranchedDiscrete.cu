#include "hip/hip_runtime.h"
#define THRUST_IGNORE_DEPRECATED_CPP_DIALECToptimal
#define CUB_IGNORE_DEPRECATED_CPP_DIALECT

#include <complex>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <iostream>
#include "CudaPseudoBranchedDiscrete.h"
#include "CudaComputationBox.h"

CudaPseudoBranchedDiscrete::CudaPseudoBranchedDiscrete(
    ComputationBox *cb,
    PolymerChain *pc)
    : Pseudo(cb, pc)
{
    try
    {
        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        // create reduced_edges, which contain partition function
        for(const auto& item: pc->get_reduced_branches_max_segment())
        {
            std::string dep = item.first;
            d_reduced_edges[dep].max_n_segment = item.second;
             // There are N segments

             // Illustration (N==5)
             // O--O--O--O--O
             // 0  1  2  3  4 

             // Legend)
             // -- : full bond
             // O  : full segment
             
            d_reduced_edges[dep].species = pc->key_to_species(dep);
            d_reduced_edges[dep].deps    = pc->key_to_deps(dep);
            d_reduced_edges[dep].partition = nullptr;
            gpu_error_check(hipMalloc((void**)&d_reduced_edges[dep].partition, sizeof(double)*M*item.second));
        }

        // create reduced_q_junctions, which contain partition function at junction of discrete chain
        for(const auto& item: pc->get_reduced_branches_max_segment())
        {
            std::string dep = item.first;
            d_reduced_q_junctions[dep] = nullptr;
            gpu_error_check(hipMalloc((void**)&d_reduced_q_junctions[dep], sizeof(double)*M));
        }

        // create reduced_blocks, which contains concentration
        std::vector<polymer_chain_block>& blocks = pc->get_blocks();
        for(int i=0; i<blocks.size(); i++)
        {
            std::string dep_v = pc->get_dep(blocks[i].v, blocks[i].u);
            std::string dep_u = pc->get_dep(blocks[i].u, blocks[i].v);
            if (dep_v > dep_u)
                dep_v.swap(dep_u);
            std::pair<std::string, std::string> key = std::make_pair(dep_v, dep_u);
            d_reduced_blocks[key].n_segment = blocks[i].n_segment;
            d_reduced_blocks[key].species   = blocks[i].species;
        }
        for(const auto& item: d_reduced_blocks)
        {
            //std::cout << "reduced_blocks: " << dep_v << ", " << dep_u << std::endl;
            d_reduced_blocks[item.first].phi = nullptr;
            gpu_error_check(hipMalloc((void**)&d_reduced_blocks[item.first].phi, sizeof(double)*M));
        }

        // create boltz_bond, boltz_bond_half, and exp_dw
        for(const auto& item: pc->get_dict_bond_lengths())
        {
            std::string species = item.first;
            d_boltz_bond     [species] = nullptr;
            d_boltz_bond_half[species] = nullptr;
            d_exp_dw         [species] = nullptr;

            gpu_error_check(hipMalloc((void**)&d_boltz_bond     [species], sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_boltz_bond_half[species], sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_exp_dw         [species], sizeof(double)*M));
        }

        // Create FFT plan
        const int BATCH{1};
        const int NRANK{cb->get_dim()};
        int n_grid[NRANK];

        if(cb->get_dim() == 3)
        {
            n_grid[0] = cb->get_nx(0);
            n_grid[1] = cb->get_nx(1);
            n_grid[2] = cb->get_nx(2);
        }
        else if(cb->get_dim() == 2)
        {
            n_grid[0] = cb->get_nx(1);
            n_grid[1] = cb->get_nx(2);
        }
        else if(cb->get_dim() == 1)
        {
            n_grid[0] = cb->get_nx(2);
        }
        hipfftPlanMany(&plan_for, NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z,BATCH);
        hipfftPlanMany(&plan_bak, NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D,BATCH);

        // allocate memory for pseudo-spectral: one-step()
        gpu_error_check(hipMalloc((void**)&d_qk_in, sizeof(double)*M));
        gpu_error_check(hipMalloc((void**)&d_q_half_step, sizeof(double)*M));
        gpu_error_check(hipMalloc((void**)&d_q_junction,  sizeof(ftsComplex)*M_COMPLEX));
        
        // allocate memory for stress calculation: dq_dl()
        gpu_error_check(hipMalloc((void**)&d_fourier_basis_x, sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_fourier_basis_y, sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_fourier_basis_z, sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_qk_1,        sizeof(ftsComplex)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_qk_2,        sizeof(ftsComplex)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_q_multi,         sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_stress_sum,      sizeof(double)*M_COMPLEX));

        update();
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
CudaPseudoBranchedDiscrete::~CudaPseudoBranchedDiscrete()
{
    hipfftDestroy(plan_for);
    hipfftDestroy(plan_bak);

    for(const auto& item: d_boltz_bond)
        hipFree(item.second);
    for(const auto& item: d_boltz_bond_half)
        hipFree(item.second);
    for(const auto& item: d_exp_dw)
        hipFree(item.second);
    for(const auto& item: d_reduced_edges)
        hipFree(item.second.partition);
    for(const auto& item: d_reduced_blocks)
        hipFree(item.second.phi);
    for(const auto& item: d_reduced_q_junctions)
        hipFree(item.second);

    // for pseudo-spectral: one-step()
    hipFree(d_qk_in);
    hipFree(d_q_half_step);
    hipFree(d_q_junction);

    // for stress calculation: dq_dl()
    hipFree(d_fourier_basis_x);
    hipFree(d_fourier_basis_y);
    hipFree(d_fourier_basis_z);
    hipFree(d_qk_1);
    hipFree(d_qk_2);
    hipFree(d_q_multi);
    hipFree(d_stress_sum);
}

void CudaPseudoBranchedDiscrete::update()
{
    try
    {
        // for pseudo-spectral: one-step()
        const int N_B = pc->get_n_block();
        const int M_COMPLEX = this->n_complex_grid;
        double boltz_bond[M_COMPLEX], boltz_bond_half[M_COMPLEX];
        
        for(const auto& item: pc->get_dict_bond_lengths())
        {
            std::string species = item.first;
            double bond_length_sq = item.second*item.second;

            get_boltz_bond(boltz_bond     , bond_length_sq,   cb->get_nx(), cb->get_dx(), pc->get_ds());
            get_boltz_bond(boltz_bond_half, bond_length_sq/2, cb->get_nx(), cb->get_dx(), pc->get_ds());

            gpu_error_check(hipMemcpy(d_boltz_bond     [species], boltz_bond,      sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_boltz_bond_half[species], boltz_bond_half, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
        }

        // for stress calculation: dq_dl()
        double fourier_basis_x[M_COMPLEX];
        double fourier_basis_y[M_COMPLEX];
        double fourier_basis_z[M_COMPLEX];
        get_weighted_fourier_basis(fourier_basis_x, fourier_basis_y, fourier_basis_z, cb->get_nx(), cb->get_dx());
        gpu_error_check(hipMemcpy(d_fourier_basis_x, fourier_basis_x, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
        gpu_error_check(hipMemcpy(d_fourier_basis_y, fourier_basis_y, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
        gpu_error_check(hipMemcpy(d_fourier_basis_z, fourier_basis_z, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoBranchedDiscrete::compute_statistics(
    std::map<std::string, double*> q_init,
    std::map<std::string, double*> w_block,
    double *phi, double &single_partition)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const double ds = pc->get_ds();

        for(const auto& item: d_reduced_edges)
        {
            if( w_block.count(item.second.species) == 0)
                throw_with_line_number("\"" + item.second.species + "\" species is not in w_block.");
        }

        if( q_init.size() > 0)
            throw_with_line_number("Currently, \'q_init\' is not supported for branched polymers.");

        // exp_dw
        double exp_dw[M];
        for(const auto& item: w_block)
        {
            std::string species = item.first;
            double *w = item.second;
            for(int i=0; i<M; i++)
                exp_dw[i] = exp(-w[i]*ds);
            gpu_error_check(hipMemcpy(d_exp_dw[species], exp_dw, sizeof(double)*M,hipMemcpyHostToDevice));
        }

        double q_uniform[M];
        for(int i=0; i<M; i++)
            q_uniform[i] = 1.0;
        for(const auto& item: d_reduced_edges)
        {
            // calculate one block end
            if (item.second.deps.size() > 0) // if it is not leaf node
            { 
                // Illustration (four branches)
                //     A
                //     |
                // O - . - B
                //     |
                //     C

                // Legend)
                // .       : junction
                // O       : full segment
                // -, |    : half bonds
                // A, B, C : other full segments

                // combine branches
                gpu_error_check(hipMemcpy(d_q_junction, q_uniform, sizeof(double)*M,hipMemcpyHostToDevice));

                for(int p=0; p<item.second.deps.size(); p++)
                {
                    std::string sub_dep = item.second.deps[p].first;
                    int sub_n_segment   = item.second.deps[p].second;

                    half_bond_step(&d_reduced_edges[sub_dep].partition[(sub_n_segment-1)*M],
                        d_q_half_step, d_boltz_bond_half[d_reduced_edges[sub_dep].species]);

                    multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_junction, d_q_junction, d_q_half_step, 1.0, M);
                }
                gpu_error_check(hipMemcpy(d_reduced_q_junctions[item.first], d_q_junction, sizeof(double)*M,hipMemcpyDeviceToDevice));

                // add half bond
                half_bond_step(d_reduced_q_junctions[item.first], &item.second.partition[0], d_boltz_bond_half[item.second.species]);

                // add full segment
                multi_real<<<N_BLOCKS, N_THREADS>>>(&item.second.partition[0], &item.second.partition[0], d_exp_dw[item.second.species], 1.0, M);
            }
            else  // if it is leaf node
            {
                //* q_init
                gpu_error_check(hipMemcpy(&item.second.partition[0], d_exp_dw[item.second.species], sizeof(double)*M,hipMemcpyDeviceToDevice));
            }

            // diffusion of each blocks
            for(int n=1; n<item.second.max_n_segment; n++)
            {
                one_step(&item.second.partition[(n-1)*M],
                         &item.second.partition[n*M],
                         d_boltz_bond[item.second.species],
                         d_exp_dw[item.second.species]);
            }
        }

        // calculate the single chain partition function
        std::string dep_v = d_reduced_blocks.begin()->first.first;
        std::string dep_u = d_reduced_blocks.begin()->first.second;
        int n_segment = d_reduced_blocks.begin()->second.n_segment;
        single_partition = ((CudaComputationBox *)cb)->inner_product_inverse_weight_gpu(
            &d_reduced_edges[dep_v].partition[(n_segment-1)*M],  // q
            &d_reduced_edges[dep_u].partition[0],                // q^dagger
            d_exp_dw[d_reduced_blocks.begin()->second.species]);

        // calculate segment concentration
        for(const auto& item: d_reduced_blocks)
        {
            calculate_phi_one_type(
                item.second.phi,                                // phi
                d_reduced_edges[item.first.first].partition,    // dependency v
                d_reduced_edges[item.first.second].partition,   // dependency u
                item.second.n_segment);                         // n_segment
        }

        // normalize the concentration
        for(const auto& item: d_reduced_blocks)
            divide_real<<<N_BLOCKS, N_THREADS>>>(item.second.phi, item.second.phi,
            d_exp_dw[item.second.species], cb->get_volume()*pc->get_ds()/single_partition, M);

        // copy phi
        std::vector<polymer_chain_block>& blocks = pc->get_blocks();
        for(int n=0; n<blocks.size(); n++)
        {
            std::string dep_v = pc->get_dep(blocks[n].v, blocks[n].u);
            std::string dep_u = pc->get_dep(blocks[n].u, blocks[n].v);
            if (dep_v > dep_u)
                dep_v.swap(dep_u);
            gpu_error_check(hipMemcpy(
                &phi[n*M],d_reduced_blocks[std::make_pair(dep_v, dep_u)].phi,
                sizeof(double)*M, hipMemcpyDeviceToHost));
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}

void CudaPseudoBranchedDiscrete::one_step(
    double *d_q_in, double *d_q_out,
    double *d_boltz_bond, double *d_exp_dw)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        //-------------- step 1 ----------
        // Execute a Forward FFT
        hipfftExecD2Z(plan_for, d_q_in, d_qk_in);

        // Multiply e^(-k^2 ds/6) in fourier space
        multi_complex_real<<<N_BLOCKS, N_THREADS>>>(d_qk_in, d_boltz_bond, M_COMPLEX);

        // Execute a backward FFT
        hipfftExecZ2D(plan_bak, d_qk_in, d_q_out);

        // Evaluate e^(-w*ds) in real space
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_out, d_q_out, d_exp_dw, 1.0/((double)M), M);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoBranchedDiscrete::half_bond_step(double *d_q_in, double *d_q_out, double *d_boltz_bond_half)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        // 3D fourier discrete transform, forward and inplace
        hipfftExecD2Z(plan_for, d_q_in, d_qk_in);
        // multiply e^(-k^2 ds/12) in fourier space, in all 3 directions
        multi_complex_real<<<N_BLOCKS, N_THREADS>>>(d_qk_in, d_boltz_bond_half, 1.0/((double)M), M_COMPLEX);
        // 3D fourier discrete transform, backward and inplace
        hipfftExecZ2D(plan_bak, d_qk_in, d_q_out);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoBranchedDiscrete::calculate_phi_one_type(
    double *d_phi, double *d_q_1, double *d_q_2, const int N)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        // Compute segment concentration
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_phi, &d_q_1[0], &d_q_2[(N-1)*M], 1.0, M);
        for(int n=1; n<N; n++)
        {
            add_multi_real<<<N_BLOCKS, N_THREADS>>>(d_phi, &d_q_1[n*M], &d_q_2[(N-n-1)*M], 1.0, M);
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
std::array<double,3> CudaPseudoBranchedDiscrete::dq_dl()
{
    // This method should be invoked after invoking compute_statistics().

    // To calculate stress, we multiply weighted fourier basis to q(k)*q^dagger(-k).
    // We only need the real part of stress calculation.
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int DIM  = cb->get_dim();
        const int M    = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        std::array<double,3> dq_dl;
        std::map<std::pair<std::string, std::string>, std::array<double,3>> reduced_dq_dl;
        std::map<std::string, double>& dict_bond_lengths = pc->get_dict_bond_lengths();
        thrust::device_ptr<double> temp_gpu_ptr(d_stress_sum);

        // compute stress for reduced key pairs
        for(const auto& item: d_reduced_blocks)
        {
            const int N = item.second.n_segment;
            auto key = item.first;
            std::string dep_v = key.first;
            std::string dep_u = key.second; 
            std::string species = item.second.species;
            double* d_q_1 = d_reduced_edges[dep_v].partition;   // dependency v
            double* d_q_2 = d_reduced_edges[dep_u].partition;   // dependency u

            double bond_length_sq;
            double *d_boltz_bond_now;

            // reset
            for(int d=0; d<3; d++)
                reduced_dq_dl[key][d] = 0.0;

            // std::cout << "dep_v: " << dep_v << std::endl;
            // std::cout << "dep_u: " << dep_u << std::endl;

            // compute stress
            for(int n=0; n<=N; n++)
            {
                std::cout<< "reduced_dq_dl[key][0]: " << reduced_dq_dl[key][0] << std::endl;
                // at v
                if (n == 0){
                    if (d_reduced_edges[dep_v].deps.size() == 0) // if v is leaf node, skip
                        continue;
                    hipfftExecD2Z(plan_for, d_reduced_q_junctions[dep_v], d_qk_1);
                    hipfftExecD2Z(plan_for, &d_q_2[(N-1)*M],              d_qk_2);
                    bond_length_sq = 0.5*dict_bond_lengths[species]*dict_bond_lengths[species];
                    d_boltz_bond_now = d_boltz_bond_half[species];
                }
                // at u  
                else if (n == N)
                {
                    if (d_reduced_edges[dep_u].deps.size() == 0) // if u is leaf node, skip
                        continue; 
                    hipfftExecD2Z(plan_for, &d_q_1[(N-1)*M],              d_qk_1);
                    hipfftExecD2Z(plan_for, d_reduced_q_junctions[dep_u], d_qk_2);
                    bond_length_sq = 0.5*dict_bond_lengths[species]*dict_bond_lengths[species];
                    d_boltz_bond_now = d_boltz_bond_half[species];
                }
                // within the blocks
                else
                {
                    hipfftExecD2Z(plan_for, &d_q_1[(n-1)*M],   d_qk_1);
                    hipfftExecD2Z(plan_for, &d_q_2[(N-n-1)*M], d_qk_2);
                    bond_length_sq = dict_bond_lengths[species]*dict_bond_lengths[species];
                    d_boltz_bond_now = d_boltz_bond[species];
                }

                // compute
                multi_complex_conjugate<<<N_BLOCKS, N_THREADS>>>(d_q_multi, d_qk_1, d_qk_2, M_COMPLEX);
                multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_multi, d_q_multi, d_boltz_bond_now, bond_length_sq, M_COMPLEX);
                if ( DIM >= 3 )
                {
                    multi_real<<<N_BLOCKS, N_THREADS>>>(d_stress_sum, d_q_multi, d_fourier_basis_x, 1.0, M_COMPLEX);
                    reduced_dq_dl[key][0] += thrust::reduce(temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX);
                }
                if ( DIM >= 2 )
                {
                    multi_real<<<N_BLOCKS, N_THREADS>>>(d_stress_sum, d_q_multi, d_fourier_basis_y, 1.0, M_COMPLEX);
                    reduced_dq_dl[key][1] += thrust::reduce(temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX);
                }
                if ( DIM >= 1 )
                {
                    multi_real<<<N_BLOCKS, N_THREADS>>>(d_stress_sum, d_q_multi, d_fourier_basis_z, 1.0, M_COMPLEX);
                    reduced_dq_dl[key][2] += thrust::reduce(temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX);
                }
            }
        }

        // compute total stress
        std::vector<polymer_chain_block>& blocks = pc->get_blocks();
        for(int d=0; d<3; d++)
            dq_dl[d] = 0.0;
        for(int n=0; n<blocks.size(); n++)
        {
            std::string dep_v = pc->get_dep(blocks[n].v, blocks[n].u);
            std::string dep_u = pc->get_dep(blocks[n].u, blocks[n].v);
            if (dep_v > dep_u)
                dep_v.swap(dep_u);
            for(int d=0; d<3; d++)
                dq_dl[d] += reduced_dq_dl[std::make_pair(dep_v, dep_u)][d];
        }
        for(int d=0; d<3; d++)
            dq_dl[d] /= 3.0*cb->get_lx(d)*M*M/pc->get_ds()/cb->get_volume();

        return dq_dl;
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoBranchedDiscrete::get_partition(double *q_out, int v, int u, int n)
{
    // This method should be invoked after invoking compute_statistics()

    // Get partial partition functions
    // This is made for debugging and testing
    try
    {
        const int M = cb->get_n_grid();
        std::string dep = pc->get_dep(v,u);
        const int N = d_reduced_edges[dep].max_n_segment;
        if (n < 1 || n > N)
            throw_with_line_number("n (" + std::to_string(n) + ") must be in range [1, " + std::to_string(N) + "]");

        double* d_partition = d_reduced_edges[dep].partition;
        gpu_error_check(hipMemcpy(q_out, &d_partition[(n-1)*M], sizeof(double)*M,hipMemcpyDeviceToHost));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
