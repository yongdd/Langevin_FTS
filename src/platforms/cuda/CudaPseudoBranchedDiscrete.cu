#include "hip/hip_runtime.h"
#define THRUST_IGNORE_DEPRECATED_CPP_DIALECToptimal
#define CUB_IGNORE_DEPRECATED_CPP_DIALECT

#include <complex>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <iostream>
#include "CudaPseudoBranchedDiscrete.h"
#include "CudaComputationBox.h"

CudaPseudoBranchedDiscrete::CudaPseudoBranchedDiscrete(
    ComputationBox *cb,
    Mixture *mx)
    : Pseudo(cb, mx)
{
    try
    {
        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        // allocate memory for partition functions
        for(const auto& item: mx->get_reduced_branches())
        {
            std::string dep = item.first;
            int max_n_segment = item.second.max_n_segment;
             // There are N segments

             // Illustration (N==5)
             // O--O--O--O--O
             // 0  1  2  3  4 reduced_blocks

             // Legend)
             // -- : full bond
             // O  : full segment
            d_reduced_partition[dep] = nullptr;
            gpu_error_check(hipMalloc((void**)&d_reduced_partition[dep], sizeof(double)*M*max_n_segment));
        }

        // allocate memory for reduced_q_junctions, which contain partition function at junction of discrete chain
        for(const auto& item: mx->get_reduced_branches())
        {
            std::string dep = item.first;
            d_reduced_q_junctions[dep] = nullptr;
            gpu_error_check(hipMalloc((void**)&d_reduced_q_junctions[dep], sizeof(double)*M));
        }

        // allocate memory for concentrations
        for(const auto& item: mx->get_reduced_blocks())
        {
            d_reduced_phi[item.first] = nullptr;
            gpu_error_check(hipMalloc((void**)&d_reduced_phi[item.first], sizeof(double)*M));
        }

        // create boltz_bond, boltz_bond_half, and exp_dw
        for(const auto& item: mx->get_bond_lengths())
        {
            std::string species = item.first;
            d_boltz_bond     [species] = nullptr;
            d_boltz_bond_half[species] = nullptr;
            d_exp_dw         [species] = nullptr;

            gpu_error_check(hipMalloc((void**)&d_boltz_bond     [species], sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_boltz_bond_half[species], sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_exp_dw         [species], sizeof(double)*M));
        }

        // Create FFT plan
        const int BATCH{1};
        const int NRANK{cb->get_dim()};
        int n_grid[NRANK];

        if(cb->get_dim() == 3)
        {
            n_grid[0] = cb->get_nx(0);
            n_grid[1] = cb->get_nx(1);
            n_grid[2] = cb->get_nx(2);
        }
        else if(cb->get_dim() == 2)
        {
            n_grid[0] = cb->get_nx(1);
            n_grid[1] = cb->get_nx(2);
        }
        else if(cb->get_dim() == 1)
        {
            n_grid[0] = cb->get_nx(2);
        }
        hipfftPlanMany(&plan_for, NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z,BATCH);
        hipfftPlanMany(&plan_bak, NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D,BATCH);

        // allocate memory for pseudo-spectral: one-step()
        gpu_error_check(hipMalloc((void**)&d_qk_in, sizeof(double)*M));
        gpu_error_check(hipMalloc((void**)&d_q_half_step, sizeof(double)*M));
        gpu_error_check(hipMalloc((void**)&d_q_junction,  sizeof(ftsComplex)*M_COMPLEX));
        
        // allocate memory for stress calculation: dq_dl()
        gpu_error_check(hipMalloc((void**)&d_fourier_basis_x, sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_fourier_basis_y, sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_fourier_basis_z, sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_qk_1,        sizeof(ftsComplex)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_qk_2,        sizeof(ftsComplex)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_q_multi,         sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_stress_sum,      sizeof(double)*M_COMPLEX));

        update();
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
CudaPseudoBranchedDiscrete::~CudaPseudoBranchedDiscrete()
{
    hipfftDestroy(plan_for);
    hipfftDestroy(plan_bak);

    for(const auto& item: d_boltz_bond)
        hipFree(item.second);
    for(const auto& item: d_boltz_bond_half)
        hipFree(item.second);
    for(const auto& item: d_exp_dw)
        hipFree(item.second);
    for(const auto& item: d_reduced_partition)
        hipFree(item.second);
    for(const auto& item: d_reduced_phi)
        hipFree(item.second);
    for(const auto& item: d_reduced_q_junctions)
        hipFree(item.second);

    // for pseudo-spectral: one-step()
    hipFree(d_qk_in);
    hipFree(d_q_half_step);
    hipFree(d_q_junction);

    // for stress calculation: dq_dl()
    hipFree(d_fourier_basis_x);
    hipFree(d_fourier_basis_y);
    hipFree(d_fourier_basis_z);
    hipFree(d_qk_1);
    hipFree(d_qk_2);
    hipFree(d_q_multi);
    hipFree(d_stress_sum);
}

void CudaPseudoBranchedDiscrete::update()
{
    try
    {
        // for pseudo-spectral: one-step()
        const int M_COMPLEX = this->n_complex_grid;
        double boltz_bond[M_COMPLEX], boltz_bond_half[M_COMPLEX];
        
        for(const auto& item: mx->get_bond_lengths())
        {
            std::string species = item.first;
            double bond_length_sq = item.second*item.second;

            get_boltz_bond(boltz_bond     , bond_length_sq,   cb->get_nx(), cb->get_dx(), mx->get_ds());
            get_boltz_bond(boltz_bond_half, bond_length_sq/2, cb->get_nx(), cb->get_dx(), mx->get_ds());

            gpu_error_check(hipMemcpy(d_boltz_bond     [species], boltz_bond,      sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_boltz_bond_half[species], boltz_bond_half, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
        }

        // for stress calculation: dq_dl()
        double fourier_basis_x[M_COMPLEX];
        double fourier_basis_y[M_COMPLEX];
        double fourier_basis_z[M_COMPLEX];
        get_weighted_fourier_basis(fourier_basis_x, fourier_basis_y, fourier_basis_z, cb->get_nx(), cb->get_dx());
        gpu_error_check(hipMemcpy(d_fourier_basis_x, fourier_basis_x, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
        gpu_error_check(hipMemcpy(d_fourier_basis_y, fourier_basis_y, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
        gpu_error_check(hipMemcpy(d_fourier_basis_z, fourier_basis_z, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
std::vector<double> CudaPseudoBranchedDiscrete::compute_statistics(
    std::map<std::string, double*> q_init,
    std::map<std::string, double*> w_block,
    std::vector<double *> phi)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const double ds = mx->get_ds();

        for(const auto& item: mx->get_reduced_branches())
        {
            if( w_block.count(item.second.species) == 0)
                throw_with_line_number("\"" + item.second.species + "\" species is not in w_block.");
        }

        if( q_init.size() > 0)
            throw_with_line_number("Currently, \'q_init\' is not supported for branched polymers.");

        // exp_dw
        double exp_dw[M];
        for(const auto& item: w_block)
        {
            std::string species = item.first;
            double *w = item.second;
            for(int i=0; i<M; i++)
                exp_dw[i] = exp(-w[i]*ds);
            gpu_error_check(hipMemcpy(d_exp_dw[species], exp_dw, sizeof(double)*M,hipMemcpyHostToDevice));
        }

        double q_uniform[M];
        for(int i=0; i<M; i++)
            q_uniform[i] = 1.0;
        for(const auto& item: mx->get_reduced_branches())
        {
            auto& key = item.first;
            // calculate one block end
            if (item.second.deps.size() > 0) // if it is not leaf node
            { 
                // Illustration (four branches)
                //     A
                //     |
                // O - . - B
                //     |
                //     C

                // Legend)
                // .       : junction
                // O       : full segment
                // -, |    : half bonds
                // A, B, C : other full segments

                // combine branches
                gpu_error_check(hipMemcpy(d_q_junction, q_uniform, sizeof(double)*M,hipMemcpyHostToDevice));

                for(int p=0; p<item.second.deps.size(); p++)
                {
                    std::string sub_dep = item.second.deps[p].first;
                    int sub_n_segment   = item.second.deps[p].second;

                    half_bond_step(&d_reduced_partition[sub_dep][(sub_n_segment-1)*M],
                        d_q_half_step, d_boltz_bond_half[mx->get_reduced_branch(sub_dep).species]);

                    multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_junction, d_q_junction, d_q_half_step, 1.0, M);
                }
                gpu_error_check(hipMemcpy(d_reduced_q_junctions[item.first], d_q_junction, sizeof(double)*M,hipMemcpyDeviceToDevice));

                // add half bond
                half_bond_step(d_reduced_q_junctions[item.first], &d_reduced_partition[key][0], d_boltz_bond_half[item.second.species]);

                // add full segment
                multi_real<<<N_BLOCKS, N_THREADS>>>(&d_reduced_partition[key][0], &d_reduced_partition[key][0], d_exp_dw[item.second.species], 1.0, M);
            }
            else  // if it is leaf node
            {
                //* q_init
                gpu_error_check(hipMemcpy(&d_reduced_partition[key][0], d_exp_dw[item.second.species], sizeof(double)*M,hipMemcpyDeviceToDevice));
            }

            // diffusion of each blocks
            for(int n=1; n<item.second.max_n_segment; n++)
            {
                one_step(&d_reduced_partition[key][(n-1)*M],
                         &d_reduced_partition[key][n*M],
                         d_boltz_bond[item.second.species],
                         d_exp_dw[item.second.species]);
            }
        }

        // calculate segment concentrations
        for(const auto& item: mx->get_reduced_blocks())
        {
            auto& key = item.first;
            calculate_phi_one_type(
                d_reduced_phi[key],                     // phi
                d_reduced_partition[std::get<0>(key)],  // dependency v
                d_reduced_partition[std::get<1>(key)],  // dependency u
                d_exp_dw[item.second.species],          // d_exp_dw
                std::get<2>(key));                      // n_segment
        }

        // for each distinct polymers 
        std::vector<double> single_partitions(mx->get_n_distinct_polymers());
        for(int p=0; p<mx->get_n_distinct_polymers(); p++)
        {
            PolymerChain *pc = mx->get_polymer_chain(p);
            std::vector<PolymerChainBlock>& blocks = pc->get_blocks();

            // calculate the single chain partition function at block 0
            std::string dep_v = pc->get_dep(blocks[0].v, blocks[0].u);
            std::string dep_u = pc->get_dep(blocks[0].u, blocks[0].v);
            int n_segment = blocks[0].n_segment;
            single_partitions[p] = ((CudaComputationBox *)cb)->inner_product_inverse_weight_gpu(
                &d_reduced_partition[dep_v][(n_segment-1)*M],  // q
                &d_reduced_partition[dep_u][0],                // q^dagger
                d_exp_dw[blocks[0].species]);        

            // copy phi
            double* phi_p = phi[p];
            for(int b=0; b<blocks.size(); b++)
            {
                std::string dep_v = pc->get_dep(blocks[b].v, blocks[b].u);
                std::string dep_u = pc->get_dep(blocks[b].u, blocks[b].v);
                if (dep_v > dep_u)
                    dep_v.swap(dep_u);
                gpu_error_check(hipMemcpy(
                    &phi_p[b*M], d_reduced_phi[std::make_tuple(dep_v, dep_u, blocks[b].n_segment)],
                    sizeof(double)*M, hipMemcpyDeviceToHost));

                // normalize the concentration
                double norm = cb->get_volume()*mx->get_ds()/single_partitions[p];
                for(int i=0; i<M; i++)
                    phi_p[i+b*M] = norm*phi_p[i+b*M]; 
            }
        }
        return single_partitions;
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}

void CudaPseudoBranchedDiscrete::one_step(
    double *d_q_in, double *d_q_out,
    double *d_boltz_bond, double *d_exp_dw)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        //-------------- step 1 ----------
        // Execute a Forward FFT
        hipfftExecD2Z(plan_for, d_q_in, d_qk_in);

        // Multiply e^(-k^2 ds/6) in fourier space
        multi_complex_real<<<N_BLOCKS, N_THREADS>>>(d_qk_in, d_boltz_bond, M_COMPLEX);

        // Execute a backward FFT
        hipfftExecZ2D(plan_bak, d_qk_in, d_q_out);

        // Evaluate e^(-w*ds) in real space
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_out, d_q_out, d_exp_dw, 1.0/((double)M), M);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoBranchedDiscrete::half_bond_step(double *d_q_in, double *d_q_out, double *d_boltz_bond_half)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        // 3D fourier discrete transform, forward and inplace
        hipfftExecD2Z(plan_for, d_q_in, d_qk_in);
        // multiply e^(-k^2 ds/12) in fourier space, in all 3 directions
        multi_complex_real<<<N_BLOCKS, N_THREADS>>>(d_qk_in, d_boltz_bond_half, 1.0/((double)M), M_COMPLEX);
        // 3D fourier discrete transform, backward and inplace
        hipfftExecZ2D(plan_bak, d_qk_in, d_q_out);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoBranchedDiscrete::calculate_phi_one_type(
    double *d_phi, double *d_q_1, double *d_q_2, double *d_exp_dw, const int N)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        // Compute segment concentration
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_phi, &d_q_1[0], &d_q_2[(N-1)*M], 1.0, M);
        for(int n=1; n<N; n++)
        {
            add_multi_real<<<N_BLOCKS, N_THREADS>>>(d_phi, &d_q_1[n*M], &d_q_2[(N-n-1)*M], 1.0, M);
        }
        divide_real<<<N_BLOCKS, N_THREADS>>>(d_phi, d_phi, d_exp_dw, 1.0, M);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
std::vector<std::array<double,3>> CudaPseudoBranchedDiscrete::dq_dl()
{
    // This method should be invoked after invoking compute_statistics().

    // To calculate stress, we multiply weighted fourier basis to q(k)*q^dagger(-k).
    // We only need the real part of stress calculation.
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int DIM  = cb->get_dim();
        const int M    = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        std::map<std::string, double>& bond_lengths = mx->get_bond_lengths();
        std::vector<std::array<double,3>> dq_dl(mx->get_n_distinct_polymers());
        std::map<std::tuple<std::string, std::string, int>, std::array<double,3>> reduced_dq_dl;
        thrust::device_ptr<double> temp_gpu_ptr(d_stress_sum);
        
        // compute stress for reduced key pairs
        for(const auto& item: mx->get_reduced_blocks())
        {
            auto& key = item.first;
            std::string dep_v = std::get<0>(key);
            std::string dep_u = std::get<1>(key);
            const int N       = std::get<2>(key);
            std::string species = item.second.species;

            double* d_q_1 = d_reduced_partition[dep_v];    // dependency v
            double* d_q_2 = d_reduced_partition[dep_u];    // dependency u

            double bond_length_sq;
            double *d_boltz_bond_now;

            // reset
            for(int d=0; d<3; d++)
                reduced_dq_dl[key][d] = 0.0;

            // std::cout << "dep_v: " << dep_v << std::endl;
            // std::cout << "dep_u: " << dep_u << std::endl;

            // compute stress
            for(int n=0; n<=N; n++)
            {
                // at v
                if (n == 0){
                    if (mx->get_reduced_branch(dep_v).deps.size() == 0) // if v is leaf node, skip
                        continue;
                    hipfftExecD2Z(plan_for, d_reduced_q_junctions[dep_v], d_qk_1);
                    hipfftExecD2Z(plan_for, &d_q_2[(N-1)*M],              d_qk_2);
                    bond_length_sq = 0.5*bond_lengths[species]*bond_lengths[species];
                    d_boltz_bond_now = d_boltz_bond_half[species];
                }
                // at u  
                else if (n == N)
                {
                    if (mx->get_reduced_branch(dep_u).deps.size() == 0) // if u is leaf node, skip
                        continue; 
                    hipfftExecD2Z(plan_for, &d_q_1[(N-1)*M],              d_qk_1);
                    hipfftExecD2Z(plan_for, d_reduced_q_junctions[dep_u], d_qk_2);
                    bond_length_sq = 0.5*bond_lengths[species]*bond_lengths[species];
                    d_boltz_bond_now = d_boltz_bond_half[species];
                }
                // within the blocks
                else
                {
                    hipfftExecD2Z(plan_for, &d_q_1[(n-1)*M],   d_qk_1);
                    hipfftExecD2Z(plan_for, &d_q_2[(N-n-1)*M], d_qk_2);
                    bond_length_sq = bond_lengths[species]*bond_lengths[species];
                    d_boltz_bond_now = d_boltz_bond[species];
                }

                // compute
                multi_complex_conjugate<<<N_BLOCKS, N_THREADS>>>(d_q_multi, d_qk_1, d_qk_2, M_COMPLEX);
                multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_multi, d_q_multi, d_boltz_bond_now, bond_length_sq, M_COMPLEX);
                if ( DIM >= 3 )
                {
                    multi_real<<<N_BLOCKS, N_THREADS>>>(d_stress_sum, d_q_multi, d_fourier_basis_x, 1.0, M_COMPLEX);
                    reduced_dq_dl[key][0] += thrust::reduce(temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX);
                }
                if ( DIM >= 2 )
                {
                    multi_real<<<N_BLOCKS, N_THREADS>>>(d_stress_sum, d_q_multi, d_fourier_basis_y, 1.0, M_COMPLEX);
                    reduced_dq_dl[key][1] += thrust::reduce(temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX);
                }
                if ( DIM >= 1 )
                {
                    multi_real<<<N_BLOCKS, N_THREADS>>>(d_stress_sum, d_q_multi, d_fourier_basis_z, 1.0, M_COMPLEX);
                    reduced_dq_dl[key][2] += thrust::reduce(temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX);
                }
            }
        }

        // compute total stress for each distinct polymers 
        for(int p=0; p < mx->get_n_distinct_polymers(); p++)
        {
            for(int d=0; d<3; d++)
                dq_dl[p][d] = 0.0;
            PolymerChain *pc = mx->get_polymer_chain(p);
            std::vector<PolymerChainBlock>& blocks = pc->get_blocks();
            for(int b=0; b<blocks.size(); b++)
            {
                std::string dep_v = pc->get_dep(blocks[b].v, blocks[b].u);
                std::string dep_u = pc->get_dep(blocks[b].u, blocks[b].v);
                if (dep_v > dep_u)
                    dep_v.swap(dep_u);
                for(int d=0; d<3; d++)
                    dq_dl[p][d] += reduced_dq_dl[std::make_tuple(dep_v, dep_u, blocks[b].n_segment)][d];
            }
            for(int d=0; d<3; d++)
                dq_dl[p][d] /= 3.0*cb->get_lx(d)*M*M/mx->get_ds()/cb->get_volume();
        }

        return dq_dl;
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoBranchedDiscrete::get_partition(double *q_out, int polymer, int v, int u, int n)
{ 
    // This method should be invoked after invoking compute_statistics()

    // Get partial partition functions
    // This is made for debugging and testing
    try
    {
        const int M = cb->get_n_grid();
        PolymerChain *pc = mx->get_polymer_chain(polymer);
        std::string dep = pc->get_dep(v,u);
        const int N = mx->get_reduced_branches()[dep].max_n_segment;
        if (n < 1 || n > N)
            throw_with_line_number("n (" + std::to_string(n) + ") must be in range [1, " + std::to_string(N) + "]");

        double* d_partition = d_reduced_partition[dep];
        gpu_error_check(hipMemcpy(q_out, &d_partition[(n-1)*M], sizeof(double)*M,hipMemcpyDeviceToHost));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
