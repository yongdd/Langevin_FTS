#include "hip/hip_runtime.h"
#define THRUST_IGNORE_DEPRECATED_CPP_DIALECT
#define CUB_IGNORE_DEPRECATED_CPP_DIALECT

#include <complex>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include "CudaPseudoReduceMemoryContinuous.h"
#include "CudaComputationBox.h"
#include "SimpsonRule.h"

CudaPseudoReduceMemoryContinuous::CudaPseudoReduceMemoryContinuous(
    ComputationBox *cb,
    Mixture *mx)
    : Pseudo(cb, mx)
{
    try{
        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        // allocate memory for propagators
        if( mx->get_essential_propagator_codes().size() == 0)
            throw_with_line_number("There is no propagator code. Add polymers first.");
        for(const auto& item: mx->get_essential_propagator_codes())
        {
            std::string dep = item.first;
            int max_n_segment = item.second.max_n_segment;

            // allocate pinned memory for device overlapping
            hipHostMalloc((void**)&propagator[dep], sizeof(double)*(max_n_segment+1)*M);

            #ifndef NDEBUG
            propagator_finished[dep] = new bool[max_n_segment+1];
            for(int i=0; i<=max_n_segment;i++)
                propagator_finished[dep][i] = false;
            #endif
        }

        // allocate memory for concentrations
        if( mx->get_essential_blocks().size() == 0)
            throw_with_line_number("There is no block. Add polymers first.");
        for(const auto& item: mx->get_essential_blocks())
        {
            block_phi[item.first] = new double[M];
        }

        // create boltz_bond, boltz_bond_half, exp_dw, and exp_dw_half
        for(const auto& item: mx->get_bond_lengths()){
            std::string monomer_type = item.first;
            d_boltz_bond     [monomer_type] = nullptr;
            d_boltz_bond_half[monomer_type] = nullptr;
            d_exp_dw         [monomer_type] = nullptr;
            d_exp_dw_half    [monomer_type] = nullptr;

            gpu_error_check(hipMalloc((void**)&d_exp_dw         [monomer_type], sizeof(double)*M));
            gpu_error_check(hipMalloc((void**)&d_exp_dw_half    [monomer_type], sizeof(double)*M));
            gpu_error_check(hipMalloc((void**)&d_boltz_bond     [monomer_type], sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_boltz_bond_half[monomer_type], sizeof(double)*M_COMPLEX));
        }

        // total partition functions for each polymer
        single_partitions = new double[mx->get_n_polymers()];

        // create FFT plan
        const int NRANK{cb->get_dim()};
        int n_grid[NRANK];

        if(cb->get_dim() == 3)
        {
            n_grid[0] = cb->get_nx(0);
            n_grid[1] = cb->get_nx(1);
            n_grid[2] = cb->get_nx(2);
        }
        else if(cb->get_dim() == 2)
        {
            n_grid[0] = cb->get_nx(0);
            n_grid[1] = cb->get_nx(1);
        }
        else if(cb->get_dim() == 1)
        {
            n_grid[0] = cb->get_nx(0);
        }

        // cufft plans using one batches for propagators 
        hipfftPlanMany(&plan_for, NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z,1);
        hipfftPlanMany(&plan_bak, NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D,1);

        // cufft plan using two batches for stress computation
        hipfftPlanMany(&plan_for_two, NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z,2);

        // three streams for overlapping kernel execution and data transfers 
        // streams[0] : data transfers
        // streams[1] : compute_statistics() using single batched cufft
        // streams[2] : compute_stress() using double batched cufft
        const int NUM_STREAMS = 3;
        streams = (hipStream_t*) malloc(sizeof(hipStream_t)*NUM_STREAMS);
        for (int i = 0; i < NUM_STREAMS; i++)
            hipStreamCreate(&streams[i]);
        hipfftSetStream(plan_for, streams[1]);
        hipfftSetStream(plan_bak, streams[1]); 
        hipfftSetStream(plan_for_two, streams[2]);

        // allocate memory for pseudo-spectral: one_step()
        d_q = new double*[2]; // one for prev, the other for next
        gpu_error_check(hipMalloc((void**)&d_q[0], sizeof(double)*M));
        gpu_error_check(hipMalloc((void**)&d_q[1], sizeof(double)*M));

        gpu_error_check(hipMalloc((void**)&d_q_step1, sizeof(double)*M));
        gpu_error_check(hipMalloc((void**)&d_q_step2, sizeof(double)*M));
        gpu_error_check(hipMalloc((void**)&d_qk_in,  sizeof(ftsComplex)*M_COMPLEX));

        d_propagator_sub_dep = new double*[2]; // one for prev, the other for next
        gpu_error_check(hipMalloc((void**)&d_propagator_sub_dep[0], sizeof(double)*M));
        gpu_error_check(hipMalloc((void**)&d_propagator_sub_dep[1], sizeof(double)*M));

        // allocate memory for stress calculation: compute_stress()
        gpu_error_check(hipMalloc((void**)&d_fourier_basis_x, sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_fourier_basis_y, sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_fourier_basis_z, sizeof(double)*M_COMPLEX));        
        gpu_error_check(hipMalloc((void**)&d_two_qk_in, sizeof(ftsComplex)*2*M_COMPLEX));

        const int NUM_BATCHES_FOR_STRESS = 2;
        d_q_two_partition = new double*[NUM_BATCHES_FOR_STRESS];
        for (int i=0; i<NUM_BATCHES_FOR_STRESS; i++)
            gpu_error_check(hipMalloc((void**)&d_q_two_partition[i], sizeof(double)*2*M));

        gpu_error_check(hipMalloc((void**)&d_q_multi,         sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_stress_sum,      sizeof(double)*M_COMPLEX));

        update_bond_function();
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
CudaPseudoReduceMemoryContinuous::~CudaPseudoReduceMemoryContinuous()
{
    const int NUM_STREAMS = 3;
    for (int i = 0; i < NUM_STREAMS; i++)
        hipStreamDestroy(streams[i]);
    free(streams);

    hipfftDestroy(plan_for);
    hipfftDestroy(plan_bak);
    hipfftDestroy(plan_for_two);

    delete[] single_partitions;

    for(const auto& item: d_boltz_bond)
        hipFree(item.second);
    for(const auto& item: d_boltz_bond_half)
        hipFree(item.second);
    for(const auto& item: d_exp_dw)
        hipFree(item.second);
    for(const auto& item: d_exp_dw_half)
        hipFree(item.second);

    for(const auto& item: propagator)
        hipHostFree(item.second);
    for(const auto& item: block_phi)
        delete[] item.second;

    #ifndef NDEBUG
    for(const auto& item: propagator_finished)
        delete[] item.second;
    #endif

    // for pseudo-spectral: one_step()
    hipFree(d_q[0]);
    hipFree(d_q[1]);
    delete[] d_q;

    hipFree(d_q_step1);
    hipFree(d_q_step2);
    hipFree(d_qk_in);

    hipFree(d_propagator_sub_dep[0]);
    hipFree(d_propagator_sub_dep[1]);
    delete[] d_propagator_sub_dep;

    // for stress calculation: compute_stress()
    hipFree(d_fourier_basis_x);
    hipFree(d_fourier_basis_y);
    hipFree(d_fourier_basis_z);
    hipFree(d_two_qk_in);

    const int NUM_BATCHES_FOR_STRESS = 2;
    for (int i=0; i<NUM_BATCHES_FOR_STRESS; i++)
        hipFree(d_q_two_partition[i]);
    delete[] d_q_two_partition;

    hipFree(d_q_multi);
    hipFree(d_stress_sum);
}

void CudaPseudoReduceMemoryContinuous::update_bond_function()
{
    try{
        // for pseudo-spectral: one_step()
        const int M_COMPLEX = this->n_complex_grid;
        double boltz_bond[M_COMPLEX], boltz_bond_half[M_COMPLEX];

        for(const auto& item: mx->get_bond_lengths())
        {
            std::string monomer_type = item.first;
            double bond_length_sq = item.second*item.second;
            get_boltz_bond(boltz_bond     , bond_length_sq,   cb->get_nx(), cb->get_dx(), mx->get_ds());
            get_boltz_bond(boltz_bond_half, bond_length_sq/2, cb->get_nx(), cb->get_dx(), mx->get_ds());
        
            gpu_error_check(hipMemcpy(d_boltz_bond[monomer_type],      boltz_bond,      sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_boltz_bond_half[monomer_type], boltz_bond_half, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
        }

        // for stress calculation: compute_stress()
        double fourier_basis_x[M_COMPLEX];
        double fourier_basis_y[M_COMPLEX];
        double fourier_basis_z[M_COMPLEX];
        get_weighted_fourier_basis(fourier_basis_x, fourier_basis_y, fourier_basis_z, cb->get_nx(), cb->get_dx());
        gpu_error_check(hipMemcpy(d_fourier_basis_x, fourier_basis_x, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
        gpu_error_check(hipMemcpy(d_fourier_basis_y, fourier_basis_y, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
        gpu_error_check(hipMemcpy(d_fourier_basis_z, fourier_basis_z, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
    }
    catch(std::exception& exc)
    {
        throw_with_line_number(exc.what());
    }
}
void CudaPseudoReduceMemoryContinuous::compute_statistics(
    std::map<std::string, double*> w_input,
    std::map<std::string, double*> q_init)
{
    try{
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const double ds = mx->get_ds();

        for(const auto& item: mx->get_essential_propagator_codes())
        {
            if( w_input.find(item.second.monomer_type) == w_input.end())
                throw_with_line_number("monomer_type \"" + item.second.monomer_type + "\" is not in w_input.");
        }

        for(const auto& item: w_input)
        {
            if( d_exp_dw.find(item.first) == d_exp_dw.end())
                throw_with_line_number("monomer_type \"" + item.first + "\" is not in d_exp_dw.");     
        }

        // exp_dw and exp_dw_half
        double exp_dw[M];
        double exp_dw_half[M];
        for(const auto& item: w_input)
        {
            std::string monomer_type = item.first;
            double *w = item.second;
            for(int i=0; i<M; i++)
            { 
                exp_dw     [i] = exp(-w[i]*ds*0.5);
                exp_dw_half[i] = exp(-w[i]*ds*0.25);
            }
            gpu_error_check(hipMemcpy(d_exp_dw     [monomer_type], exp_dw,      sizeof(double)*M,hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_exp_dw_half[monomer_type], exp_dw_half, sizeof(double)*M,hipMemcpyHostToDevice));
        }

        double q_uniform[M];
        for(int i=0; i<M; i++)
            q_uniform[i] = 1.0;

        // for each propagator code
        for (auto& item: mx->get_essential_propagator_codes())
        {
            auto& key = item.first;
            auto& deps = item.second.deps;
            int n_segment = item.second.max_n_segment;
            auto monomer_type = item.second.monomer_type;

            // check key
            #ifndef NDEBUG
            if (propagator.find(key) == propagator.end())
                throw_with_line_number("Could not find key '" + key + "'. ");
            #endif
            double *_propagator = propagator[key];

            // if it is leaf node
            if(deps.size() == 0) 
            {
                // q_init
                if (key[0] == '{')
                {
                    std::string g = Mixture::get_q_input_idx_from_key(key);
                    if (q_init.find(g) == q_init.end())
                        throw_with_line_number( "Could not find q_init[\"" + g + "\"].");
                    gpu_error_check(hipMemcpy(d_q[0], q_init[g], sizeof(double)*M, hipMemcpyHostToDevice));
                }
                else
                {
                    gpu_error_check(hipMemcpy(d_q[0], q_uniform, sizeof(double)*M, hipMemcpyHostToDevice));
                }

                #ifndef NDEBUG
                propagator_finished[key][0] = true;
                #endif
            }
            // if it is not leaf node
            else if (deps.size() > 0) 
            {
                // if it is superposed
                if (key[0] == '[')
                {
                    // initialize to zero
                    gpu_error_check(hipMemset(d_q[0], 0, sizeof(double)*M));

                    int prev, next;
                    prev = 0;
                    next = 1;

                    // copy memory from host to device
                    std::string sub_dep = std::get<0>(deps[0]);
                    int sub_n_segment   = std::get<1>(deps[0]);
                    int sub_n_repeated;
                    gpu_error_check(hipMemcpy(d_propagator_sub_dep[prev], &propagator[sub_dep][(sub_n_segment)*M], sizeof(double)*M, hipMemcpyHostToDevice));

                    for(size_t d=0; d<deps.size(); d++)
                    {
                        sub_dep         = std::get<0>(deps[d]);
                        sub_n_segment   = std::get<1>(deps[d]);
                        sub_n_repeated  = std::get<2>(deps[d]);

                        // check sub key
                        #ifndef NDEBUG
                        if (propagator.find(sub_dep) == propagator.end())
                            throw_with_line_number("Could not find sub key '" + sub_dep + "'. ");
                        if (!propagator_finished[sub_dep][sub_n_segment])
                            throw_with_line_number("Could not compute '" + key +  "', since '"+ sub_dep + std::to_string(sub_n_segment) + "' is not prepared.");
                        #endif

                        // STREAM 0: copy memory from host to device
                        if (d < deps.size()-1)
                        {
                            std::string sub_dep_next = std::get<0>(deps[d+1]);
                            int sub_n_segment_next   = std::get<1>(deps[d+1]);

                            gpu_error_check(hipMemcpyAsync(d_propagator_sub_dep[next],
                                            &propagator[sub_dep_next][(sub_n_segment_next)*M], sizeof(double)*M,
                                            hipMemcpyHostToDevice, streams[0]));
                        }

                        // STREAM 1: compute linear combination
                        lin_comb<<<N_BLOCKS, N_THREADS, 0, streams[1]>>>(
                                d_q[0], 1.0, d_q[0],
                                sub_n_repeated, d_propagator_sub_dep[prev], M);

                        std::swap(prev, next);
                        hipDeviceSynchronize();
                    }

                    #ifndef NDEBUG
                    propagator_finished[key][0] = true;
                    #endif
                }
                else
                { 
                    // initialize to one
                    gpu_error_check(hipMemcpy(d_q[0], q_uniform,
                        sizeof(double)*M, hipMemcpyHostToDevice));

                    int prev, next;
                    prev = 0;
                    next = 1;

                    // copy memory from host to device
                    std::string sub_dep = std::get<0>(deps[0]);
                    int sub_n_segment   = std::get<1>(deps[0]);
                    gpu_error_check(hipMemcpy(d_propagator_sub_dep[prev], &propagator[sub_dep][(sub_n_segment)*M], sizeof(double)*M, hipMemcpyHostToDevice));

                    for(size_t d=0; d<deps.size(); d++)
                    {
                        std::string sub_dep = std::get<0>(deps[d]);
                        int sub_n_segment   = std::get<1>(deps[d]);

                        // check sub key
                        #ifndef NDEBUG
                        if (propagator.find(sub_dep) == propagator.end())
                            throw_with_line_number("Could not find sub key '" + sub_dep + "'. ");
                        if (!propagator_finished[sub_dep][sub_n_segment])
                            throw_with_line_number("Could not compute '" + key +  "', since '"+ sub_dep + std::to_string(sub_n_segment) + "' is not prepared.");
                        #endif

                        // STREAM 0: copy memory from host to device
                        if (d < deps.size()-1)
                        {
                            std::string sub_dep_next = std::get<0>(deps[d+1]);
                            int sub_n_segment_next   = std::get<1>(deps[d+1]);

                            gpu_error_check(hipMemcpyAsync(d_propagator_sub_dep[next],
                                            &propagator[sub_dep_next][(sub_n_segment_next)*M], sizeof(double)*M,
                                            hipMemcpyHostToDevice, streams[0]));
                        }

                        // STREAM 1: multiply 
                        multi_real<<<N_BLOCKS, N_THREADS, 0, streams[1]>>>(
                            d_q[0], d_q[0], d_propagator_sub_dep[prev], 1.0, M);

                        std::swap(prev, next);
                        hipDeviceSynchronize();
                    }
                    
                    #ifndef NDEBUG
                    propagator_finished[key][0] = true;
                    #endif
                }
            }
            hipDeviceSynchronize();

            // if there is no segment to be computed
            if (n_segment == 0)
            {
                gpu_error_check(hipMemcpy(&_propagator[0], d_q[0], sizeof(double)*M, hipMemcpyDeviceToHost));
                continue;
            }

            // advance propagator successively
            int prev, next;
            prev = 0;
            next = 1;

            for(int n=1; n<=n_segment; n++)
            {
                #ifndef NDEBUG
                if (!propagator_finished[key][n-1])
                    throw_with_line_number("unfinished, key: " + key + ", " + std::to_string(n-1));
                #endif

                // STREAM 0: copy memory from host to device
                gpu_error_check(hipMemcpyAsync(&_propagator[(n-1)*M], d_q[prev], sizeof(double)*M,
                    hipMemcpyDeviceToHost, streams[0]));

                // STREAM 1: calculate propagator
                one_step(
                    d_q[prev],
                    d_q[next],
                    d_boltz_bond[monomer_type],
                    d_boltz_bond_half[monomer_type],
                    d_exp_dw[monomer_type],
                    d_exp_dw_half[monomer_type]);

                std::swap(prev, next);
                hipDeviceSynchronize();

                #ifndef NDEBUG
                propagator_finished[key][n] = true;
                #endif
            }
            gpu_error_check(hipMemcpy(&_propagator[(n_segment)*M], d_q[prev], sizeof(double)*M,
                hipMemcpyDeviceToHost));
        }

        // compute total partition function of each distinct polymers
        int current_p = 0;
        for(const auto& block: block_phi)
        {
            int p                = std::get<0>(block.first);
            std::string dep_v    = std::get<1>(block.first);
            std::string dep_u    = std::get<2>(block.first);

            // already computed
            if (p != current_p)
                continue;

            int n_superposed;
            // int n_segment_allocated = mx->get_essential_block(block.first).n_segment_allocated;
            int n_segment_offset    = mx->get_essential_block(block.first).n_segment_offset;
            int n_segment_original  = mx->get_essential_block(block.first).n_segment_original;

            // contains no '['
            if (dep_u.find('[') == std::string::npos)
                n_superposed = 1;
            else
                n_superposed = mx->get_essential_block(block.first).v_u.size();

            // check keys
            #ifndef NDEBUG
            if (propagator.find(dep_v) == propagator.end())
                std::cout << "Could not find dep_v key'" + dep_v + "'. " << std::endl;
            if (propagator.find(dep_u) == propagator.end())
                std::cout << "Could not find dep_u key'" + dep_u + "'. " << std::endl;
            #endif

            single_partitions[p]= cb->inner_product(
                &propagator[dep_v][(n_segment_original-n_segment_offset)*M], // q
                &propagator[dep_u][0])/n_superposed/cb->get_volume();        // q^dagger

            // std::cout << p << ", " << single_partitions[p] << std::endl;
            // std::cout << p << ", "<< dep_v << ", "<< dep_u << ", "<< single_partitions[p] << std::endl;
            // std::cout << p <<", "<< n_segment <<", "<< n_segment_offset <<", "<< single_partitions[p] << std::endl;
            current_p++;
        }

        // calculate segment concentrations
        for(size_t b=0; b<block_phi.size();b++)
        {
            auto block = block_phi.begin();
            advance(block, b);
            const auto& key = block->first;

            int p                = std::get<0>(key);
            std::string dep_v    = std::get<1>(key);
            std::string dep_u    = std::get<2>(key);

            int n_repeated;
            int n_segment_allocated = mx->get_essential_block(key).n_segment_allocated;
            int n_segment_offset    = mx->get_essential_block(key).n_segment_offset;
            int n_segment_original  = mx->get_essential_block(key).n_segment_original;

            // if there is no segment
            if(n_segment_allocated == 0)
            {
                for(int i=0; i<M;i++)
                    block->second[i] = 0.0;
                continue;
            }

            // contains no '['
            if (dep_u.find('[') == std::string::npos)
                n_repeated = mx->get_essential_block(key).v_u.size();
            else
                n_repeated = 1;

            // check keys
            #ifndef NDEBUG
            if (propagator.find(dep_v) == propagator.end())
                std::cout << "Could not find dep_v key'" + dep_v + "'. " << std::endl;
            if (propagator.find(dep_u) == propagator.end())
                std::cout << "Could not find dep_u key'" + dep_u + "'. " << std::endl;
            #endif

            // calculate phi of one block (possibly multiple blocks when using superposition)
            calculate_phi_one_block(
                block->second,             // phi
                propagator[dep_v],  // dependency v
                propagator[dep_u],  // dependency u
                n_segment_allocated,
                n_segment_offset,
                n_segment_original);

            // normalize concentration
            PolymerChain& pc = mx->get_polymer(p);
            double norm = mx->get_ds()*pc.get_volume_fraction()/pc.get_alpha()/single_partitions[p]*n_repeated;
            for(int i=0; i<M; i++)
                block->second[i] *= norm;
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}

// Advance propagator using Richardson extrapolation.
void CudaPseudoReduceMemoryContinuous::one_step(
    double *d_q_in, double *d_q_out,
    double *d_boltz_bond, double *d_boltz_bond_half,
    double *d_exp_dw, double *d_exp_dw_half)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        //-------------- step 1 ----------
        // Evaluate e^(-w*ds/2) in real space
        multi_real<<<N_BLOCKS, N_THREADS, 0, streams[1]>>>(d_q_step1, d_q_in, d_exp_dw, 1.0, M);

        // Execute a Forw_ard FFT
        hipfftExecD2Z(plan_for, d_q_step1, d_qk_in);

        // Multiply e^(-k^2 ds/6) in fourier space
        multi_complex_real<<<N_BLOCKS, N_THREADS, 0, streams[1]>>>(d_qk_in, d_boltz_bond, M_COMPLEX);

        // Execute a backw_ard FFT
        hipfftExecZ2D(plan_bak, d_qk_in, d_q_step1);

        // Evaluate e^(-w*ds/2) in real space
        multi_real<<<N_BLOCKS, N_THREADS, 0, streams[1]>>>(d_q_step1, d_q_step1, d_exp_dw, 1.0/((double)M), M);

        //-------------- step 2 ----------
        // Evaluate e^(-w*ds/4) in real space
        multi_real<<<N_BLOCKS, N_THREADS, 0, streams[1]>>>(d_q_step2, d_q_in, d_exp_dw_half, 1.0, M);

        // Execute a Forw_ard FFT
        hipfftExecD2Z(plan_for, d_q_step2, d_qk_in);

        // Multiply e^(-k^2 ds/12) in fourier space
        multi_complex_real<<<N_BLOCKS, N_THREADS, 0, streams[1]>>>(d_qk_in, d_boltz_bond_half, M_COMPLEX);

        // Execute a backw_ard FFT
        hipfftExecZ2D(plan_bak, d_qk_in, d_q_step2);

        // Evaluate e^(-w*ds/2) in real space
        multi_real<<<N_BLOCKS, N_THREADS, 0, streams[1]>>>(d_q_step2, d_q_step2, d_exp_dw, 1.0/((double)M), M);
        // Execute a Forw_ard FFT
        hipfftExecD2Z(plan_for, d_q_step2, d_qk_in);

        // Multiply e^(-k^2 ds/12) in fourier space
        multi_complex_real<<<N_BLOCKS, N_THREADS, 0, streams[1]>>>(d_qk_in, d_boltz_bond_half, M_COMPLEX);

        // Execute a backw_ard FFT
        hipfftExecZ2D(plan_bak, d_qk_in, d_q_step2);

        // Evaluate e^(-w*ds/4) in real space.
        multi_real<<<N_BLOCKS, N_THREADS, 0, streams[1]>>>(d_q_step2, d_q_step2, d_exp_dw_half, 1.0/((double)M), M);
        //-------------- step 3 ----------
        lin_comb<<<N_BLOCKS, N_THREADS, 0, streams[1]>>>(d_q_out, 4.0/3.0, d_q_step2, -1.0/3.0, d_q_step1, M);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoReduceMemoryContinuous::calculate_phi_one_block(
    double *phi, double *q_1, double *q_2, const int N, const int N_OFFSET, const int N_ORIGINAL)
{

    try
    {
        const int M = cb->get_n_grid();
        std::vector<double> simpson_rule_coeff = SimpsonRule::get_coeff(N);

        // Compute segment concentration
        for(int i=0; i<M; i++)
            phi[i] = simpson_rule_coeff[0]*q_1[i+(N_ORIGINAL-N_OFFSET)*M]*q_2[i];
        for(int n=1; n<=N; n++)
        {
            for(int i=0; i<M; i++)
                phi[i] += simpson_rule_coeff[n]*q_1[i+(N_ORIGINAL-N_OFFSET-n)*M]*q_2[i+n*M];
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
double CudaPseudoReduceMemoryContinuous::get_total_partition(int polymer)
{
    try
    {
        return single_partitions[polymer];
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoReduceMemoryContinuous::get_monomer_concentration(std::string monomer_type, double *phi)
{
    try
    {
        const int M = cb->get_n_grid();
        // initialize array
        for(int i=0; i<M; i++)
            phi[i] = 0.0;

        // for each block
        for(const auto& block: block_phi)
        {
            std::string dep_v = std::get<1>(block.first);
            int n_segment_allocated = mx->get_essential_block(block.first).n_segment_allocated;
            if (Mixture::get_monomer_type_from_key(dep_v) == monomer_type && n_segment_allocated != 0)
            {
                for(int i=0; i<M; i++)
                    phi[i] += block.second[i]; 
            }
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoReduceMemoryContinuous::get_polymer_concentration(int p, double *phi)
{
    try
    {
        const int M = cb->get_n_grid();
        const int P = mx->get_n_polymers();

        if (p < 0 || p > P-1)
            throw_with_line_number("Index (" + std::to_string(p) + ") must be in range [0, " + std::to_string(P-1) + "]");

        if (mx->is_using_superposition())
            throw_with_line_number("Disable 'superposition' option to invoke 'get_polymer_concentration'.");

        PolymerChain& pc = mx->get_polymer(p);
        std::vector<PolymerChainBlock>& blocks = pc.get_blocks();

        for(size_t b=0; b<blocks.size(); b++)
        {
            std::string dep_v = pc.get_propagator_key(blocks[b].v, blocks[b].u);
            std::string dep_u = pc.get_propagator_key(blocks[b].u, blocks[b].v);
            if (dep_v < dep_u)
                dep_v.swap(dep_u);

            double* _essential_block_phi = block_phi[std::make_tuple(p, dep_v, dep_u)];
            for(int i=0; i<M; i++)
                phi[i+b*M] = _essential_block_phi[i]; 
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
std::vector<double> CudaPseudoReduceMemoryContinuous::compute_stress()
{
    // This method should be invoked after invoking compute_statistics().

    // To calculate stress, we multiply weighted fourier basis to q(k)*q^dagger(-k).
    // We only need the real part of stress calculation.

    try{
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int DIM  = cb->get_dim();
        const int M    = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        auto bond_lengths = mx->get_bond_lengths();
        std::vector<double> stress(cb->get_dim());
        std::map<std::tuple<int, std::string, std::string>, std::array<double,3>> block_dq_dl;
        thrust::device_ptr<double> temp_gpu_ptr(d_stress_sum);

        // reset stress map
        for(const auto& item: block_phi)
        {
            for(int d=0; d<3; d++)
                block_dq_dl[item.first][d] = 0.0;
        }

        // compute stress for each block
        for(const auto& block: block_phi)
        {
            const auto& key      = block.first;
            int p                = std::get<0>(key);
            std::string dep_v    = std::get<1>(key);
            std::string dep_u    = std::get<2>(key);

            const int N           = mx->get_essential_block(block.first).n_segment_allocated;
            const int N_OFFSET    = mx->get_essential_block(block.first).n_segment_offset;
            const int N_ORIGINAL  = mx->get_essential_block(block.first).n_segment_original;
            std::string monomer_type = mx->get_essential_block(key).monomer_type;

            // if there is no segment
            if(N == 0)
                continue;

            // contains no '['
            int n_repeated;
            if (dep_u.find('[') == std::string::npos)
                n_repeated = mx->get_essential_block(block.first).v_u.size();
            else
                n_repeated = 1;

            std::vector<double> s_coeff = SimpsonRule::get_coeff(N);
            double bond_length_sq = bond_lengths[monomer_type]*bond_lengths[monomer_type];
            double* q_1 = propagator[dep_v];    // dependency v
            double* q_2 = propagator[dep_u];    // dependency u

            std::array<double,3> _block_dq_dl = block_dq_dl[key];

            int prev, next;
            prev = 0;
            next = 1;

            // copy memory from host to device
            gpu_error_check(hipMemcpy(&d_q_two_partition[prev][0], &q_1[(N_ORIGINAL-N_OFFSET)*M], sizeof(double)*M,hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(&d_q_two_partition[prev][M], &q_2[0],                       sizeof(double)*M,hipMemcpyHostToDevice));

            // compute
            for(int n=0; n<=N; n++)
            {
                // STREAM 0: copy memory from host to device
                if (n < N)
                {
                    gpu_error_check(hipMemcpyAsync(&d_q_two_partition[next][0], &q_1[(N_ORIGINAL-N_OFFSET-(n+1))*M], sizeof(double)*M,hipMemcpyHostToDevice, streams[0]));
                    gpu_error_check(hipMemcpyAsync(&d_q_two_partition[next][M], &q_2[(n+1)*M],                       sizeof(double)*M,hipMemcpyHostToDevice, streams[0]));
                }

                // STREAM 2: execute a Forward FFT
                hipfftExecD2Z(plan_for_two, d_q_two_partition[prev], d_two_qk_in);

                // multiply two propagators in the fourier spaces
                multi_complex_conjugate<<<N_BLOCKS, N_THREADS, 0, streams[2]>>>(d_q_multi, &d_two_qk_in[0], &d_two_qk_in[M_COMPLEX], M_COMPLEX);

                if ( DIM == 3 )
                {
                    multi_real<<<N_BLOCKS, N_THREADS, 0, streams[2]>>>(d_stress_sum, d_q_multi, d_fourier_basis_x, bond_length_sq, M_COMPLEX);
                    _block_dq_dl[0] += s_coeff[n]*thrust::reduce(thrust::cuda::par.on(streams[2]), temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX)*n_repeated;

                    multi_real<<<N_BLOCKS, N_THREADS, 0, streams[2]>>>(d_stress_sum, d_q_multi, d_fourier_basis_y, bond_length_sq, M_COMPLEX);
                    _block_dq_dl[1] += s_coeff[n]*thrust::reduce(thrust::cuda::par.on(streams[2]), temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX)*n_repeated;

                    multi_real<<<N_BLOCKS, N_THREADS, 0, streams[2]>>>(d_stress_sum, d_q_multi, d_fourier_basis_z, bond_length_sq, M_COMPLEX);
                    _block_dq_dl[2] += s_coeff[n]*thrust::reduce(thrust::cuda::par.on(streams[2]), temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX)*n_repeated;

                }
                if ( DIM == 2 )
                {
                    multi_real<<<N_BLOCKS, N_THREADS, 0, streams[2]>>>(d_stress_sum, d_q_multi, d_fourier_basis_y, bond_length_sq, M_COMPLEX);
                    _block_dq_dl[0] += s_coeff[n]*thrust::reduce(thrust::cuda::par.on(streams[2]), temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX)*n_repeated;
                    
                    multi_real<<<N_BLOCKS, N_THREADS, 0, streams[2]>>>(d_stress_sum, d_q_multi, d_fourier_basis_z, bond_length_sq, M_COMPLEX);
                    _block_dq_dl[1] += s_coeff[n]*thrust::reduce(thrust::cuda::par.on(streams[2]), temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX)*n_repeated;
                }
                if ( DIM == 1 )
                {
                    multi_real<<<N_BLOCKS, N_THREADS, 0, streams[2]>>>(d_stress_sum, d_q_multi, d_fourier_basis_z, bond_length_sq, M_COMPLEX);
                    _block_dq_dl[0] += s_coeff[n]*thrust::reduce(thrust::cuda::par.on(streams[2]), temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX)*n_repeated;
                }
                std::swap(prev, next);
                hipDeviceSynchronize();
            }
            block_dq_dl[key] = _block_dq_dl;
        }

        // compute total stress
        for(int d=0; d<cb->get_dim(); d++)
            stress[d] = 0.0;
        for(const auto& block: block_phi)
        {
            const auto& key   = block.first;
            int p             = std::get<0>(key);
            std::string dep_v = std::get<1>(key);
            std::string dep_u = std::get<2>(key);
            PolymerChain& pc  = mx->get_polymer(p);

            for(int d=0; d<cb->get_dim(); d++)
                stress[d] += block_dq_dl[key][d]*pc.get_volume_fraction()/pc.get_alpha()/single_partitions[p];
        }
        for(int d=0; d<cb->get_dim(); d++)
            stress[d] /= -3.0*cb->get_lx(d)*M*M/mx->get_ds();
            
        return stress;
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoReduceMemoryContinuous::get_chain_propagator(double *q_out, int polymer, int v, int u, int n)
{
    // This method should be invoked after invoking compute_statistics()

    // Get chain propagator for a selected polymer, block and direction.
    // This is made for debugging and testing.
    try
    {
        const int M = cb->get_n_grid();
        PolymerChain& pc = mx->get_polymer(polymer);
        std::string dep = pc.get_propagator_key(v,u);

        if (mx->get_essential_propagator_codes().find(dep) == mx->get_essential_propagator_codes().end())
            throw_with_line_number("Could not find the propagator code '" + dep + "'. Disable 'superposition' option to obtain propagators.");

        const int N = mx->get_essential_propagator_codes()[dep].max_n_segment;
        if (n < 0 || n > N)
            throw_with_line_number("n (" + std::to_string(n) + ") must be in range [0, " + std::to_string(N) + "]");

        double* _partition = propagator[dep];
        for(int i=0; i<M; i++)
            q_out[i] = _partition[n*M+i];
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}