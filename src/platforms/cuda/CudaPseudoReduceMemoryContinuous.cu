#include "hip/hip_runtime.h"
#include <complex>
#include <thrust/reduce.h>
#include "CudaPseudoReduceMemoryContinuous.h"
#include "CudaComputationBox.h"
#include "SimpsonRule.h"

CudaPseudoReduceMemoryContinuous::CudaPseudoReduceMemoryContinuous(
    ComputationBox *cb,
    Molecules *molecules)
    : Pseudo(cb, molecules)
{
    try{
        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;
        const int N_GPUS = CudaCommon::get_instance().get_n_gpus();

        // Allocate memory for propagators
        gpu_error_check(hipSetDevice(0));
        if( molecules->get_essential_propagator_codes().size() == 0)
            throw_with_line_number("There is no propagator code. Add polymers first.");
        for(const auto& item: molecules->get_essential_propagator_codes())
        {
            std::string key = item.first;
            int max_n_segment = item.second.max_n_segment;

            propagator_size[key] = max_n_segment+1;
            propagator[key] = new double*[max_n_segment+1];
            // Allocate pinned memory for device overlapping
            for(int i=0; i<propagator_size[key]; i++)
                gpu_error_check(hipHostMalloc((void**)&propagator[key][i], sizeof(double)*M));

            #ifndef NDEBUG
            propagator_finished[key] = new bool[max_n_segment+1];
            for(int i=0; i<=max_n_segment;i++)
                propagator_finished[key][i] = false;
            #endif
        }

        // Allocate memory for concentrations
        if( molecules->get_essential_blocks().size() == 0)
            throw_with_line_number("There is no block. Add polymers first.");
        for(const auto& item: molecules->get_essential_blocks())
        {
            block_phi[item.first] = nullptr;
            // Allocate pinned memory
            gpu_error_check(hipHostMalloc((void**)&block_phi[item.first], sizeof(double)*M));
        }

        // Create boltz_bond, boltz_bond_half, exp_dw, and exp_dw_half
        for(const auto& item: molecules->get_bond_lengths())
        {
            std::string monomer_type = item.first;
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                d_boltz_bond     [gpu][monomer_type] = nullptr;
                d_boltz_bond_half[gpu][monomer_type] = nullptr;
                d_exp_dw         [gpu][monomer_type] = nullptr;
                d_exp_dw_half    [gpu][monomer_type] = nullptr;

                gpu_error_check(hipMalloc((void**)&d_exp_dw         [gpu][monomer_type], sizeof(double)*M));
                gpu_error_check(hipMalloc((void**)&d_exp_dw_half    [gpu][monomer_type], sizeof(double)*M));
                gpu_error_check(hipMalloc((void**)&d_boltz_bond     [gpu][monomer_type], sizeof(double)*M_COMPLEX));
                gpu_error_check(hipMalloc((void**)&d_boltz_bond_half[gpu][monomer_type], sizeof(double)*M_COMPLEX));
            }
        }

        // Total partition functions for each polymer
        single_partitions = new double[molecules->get_n_polymer_types()];

        // Remember one segment for each polymer chain to compute total partition function
        int current_p = 0;
        for(const auto& block: block_phi)
        {
            const auto& key = block.first;
            int p                = std::get<0>(key);
            std::string dep_v    = std::get<1>(key);
            std::string dep_u    = std::get<2>(key);

            // Skip if already found one segment
            if (p != current_p)
                continue;

            int n_superposed;
            int n_segment_offset    = molecules->get_essential_block(key).n_segment_offset;
            int n_segment_original  = molecules->get_essential_block(key).n_segment_original;

            // Contains no '['
            if (dep_u.find('[') == std::string::npos)
                n_superposed = 1;
            else
                n_superposed = molecules->get_essential_block(key).v_u.size();

            single_partition_segment.push_back(std::make_tuple(
                p,
                propagator[dep_v][n_segment_original-n_segment_offset],   // q
                propagator[dep_u][0],                                   // Q_dagger
                n_superposed                    // How many propagators are aggregated
                ));
            current_p++;
        }

        // Create scheduler for computation of propagator
        sc = new Scheduler(molecules->get_essential_propagator_codes(), N_SCHEDULER_STREAMS); 

        // Create streams
        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            gpu_error_check(hipStreamCreate(&streams[gpu][0])); // for kernel execution
            gpu_error_check(hipStreamCreate(&streams[gpu][1])); // for memcpy
        }

        // Create FFT plan
        const int NRANK{cb->get_dim()};
        int n_grid[NRANK];

        if(cb->get_dim() == 3)
        {
            n_grid[0] = cb->get_nx(0);
            n_grid[1] = cb->get_nx(1);
            n_grid[2] = cb->get_nx(2);
        }
        else if(cb->get_dim() == 2)
        {
            n_grid[0] = cb->get_nx(0);
            n_grid[1] = cb->get_nx(1);
        }
        else if(cb->get_dim() == 1)
        {
            n_grid[0] = cb->get_nx(0);
        }

        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            hipfftPlanMany(&plan_for_one[gpu], NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z,1);
            hipfftPlanMany(&plan_for_two[gpu], NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z,2);
            hipfftPlanMany(&plan_bak_one[gpu], NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D,1);
            hipfftPlanMany(&plan_bak_two[gpu], NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D,2);
            hipfftSetStream(plan_for_one[gpu], streams[gpu][0]);
            hipfftSetStream(plan_for_two[gpu], streams[gpu][0]);
            hipfftSetStream(plan_bak_one[gpu], streams[gpu][0]);
            hipfftSetStream(plan_bak_two[gpu], streams[gpu][0]);
        }

        gpu_error_check(hipSetDevice(0));
        // Allocate memory for pseudo-spectral: advance_one_propagator()
        gpu_error_check(hipMalloc((void**)&d_propagator_sub_dep[0], sizeof(double)*M)); // for prev
        gpu_error_check(hipMalloc((void**)&d_propagator_sub_dep[1], sizeof(double)*M)); // for next

        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            // Allocate memory for propagator computation
            gpu_error_check(hipMalloc((void**)&d_q_one[gpu][0], sizeof(double)*M)); // for prev
            gpu_error_check(hipMalloc((void**)&d_q_one[gpu][1], sizeof(double)*M)); // for next
            // Allocate memory for pseudo-spectral: advance_one_propagator()
            gpu_error_check(hipMalloc((void**)&d_q_step_1_one[gpu], sizeof(double)*M));
            gpu_error_check(hipMalloc((void**)&d_q_step_2_one[gpu], sizeof(double)*M));
            gpu_error_check(hipMalloc((void**)&d_q_step_1_two[gpu], sizeof(double)*2*M));
            gpu_error_check(hipMalloc((void**)&d_qk_in_2_one[gpu], sizeof(ftsComplex)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_qk_in_1_two[gpu], sizeof(ftsComplex)*2*M_COMPLEX));
        }

        gpu_error_check(hipSetDevice(0));
        double q_unity[M];
        for(int i=0; i<M; i++)
            q_unity[i] = 1.0;
        gpu_error_check(hipMalloc((void**)&d_q_unity, sizeof(double)*M));
        gpu_error_check(hipMemcpy(d_q_unity, q_unity, sizeof(double)*M, hipMemcpyHostToDevice));

        // For concentration computation
        gpu_error_check(hipMalloc((void**)&d_q_block_v[0], sizeof(double)*M)); // for prev
        gpu_error_check(hipMalloc((void**)&d_q_block_v[1], sizeof(double)*M)); // for next
        gpu_error_check(hipMalloc((void**)&d_q_block_u[0], sizeof(double)*M)); // for prev
        gpu_error_check(hipMalloc((void**)&d_q_block_u[1], sizeof(double)*M)); // for next
        gpu_error_check(hipMalloc((void**)&d_phi,          sizeof(double)*M));

        // Allocate memory for stress calculation: compute_stress()
        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            gpu_error_check(hipMalloc((void**)&d_fourier_basis_x[gpu], sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_fourier_basis_y[gpu], sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_fourier_basis_z[gpu], sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_stress_sum[gpu],      sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_stress_sum_out[gpu],  sizeof(double)*1));
            gpu_error_check(hipMalloc((void**)&d_stress_q[gpu][0],     sizeof(double)*2*M)); // prev
            gpu_error_check(hipMalloc((void**)&d_stress_q[gpu][1],     sizeof(double)*2*M)); // next
            gpu_error_check(hipMalloc((void**)&d_q_multi[gpu],         sizeof(double)*M_COMPLEX));
        }

        // Allocate memory for cub reduction sum
        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            d_temp_storage[gpu] = nullptr;
            temp_storage_bytes[gpu] = 0;
            hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
            gpu_error_check(hipMalloc(&d_temp_storage[gpu], temp_storage_bytes[gpu]));
        }
        update_bond_function();

        gpu_error_check(hipSetDevice(0));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
CudaPseudoReduceMemoryContinuous::~CudaPseudoReduceMemoryContinuous()
{
    const int N_GPUS = CudaCommon::get_instance().get_n_gpus();
    
    for(int gpu=0; gpu<N_GPUS; gpu++)
    {
        hipfftDestroy(plan_for_one[gpu]);
        hipfftDestroy(plan_for_two[gpu]);
        hipfftDestroy(plan_bak_one[gpu]);
        hipfftDestroy(plan_bak_two[gpu]);
    }

    delete sc;

    delete[] single_partitions;

    for(int gpu=0; gpu<N_GPUS; gpu++)
    {
        for(const auto& item: d_boltz_bond[gpu])
            hipFree(item.second);
        for(const auto& item: d_boltz_bond_half[gpu])
            hipFree(item.second);
        for(const auto& item: d_exp_dw[gpu])
            hipFree(item.second);
        for(const auto& item: d_exp_dw_half[gpu])
            hipFree(item.second);
    }

    for(const auto& item: propagator)
    {
        for(int i=0; i<propagator_size[item.first]; i++)
            hipHostFree(item.second[i]);
        delete[] item.second;
    }
    for(const auto& item: block_phi)
        hipHostFree(item.second);

    #ifndef NDEBUG
    for(const auto& item: propagator_finished)
        delete[] item.second;
    #endif

    for(int gpu=0; gpu<N_GPUS; gpu++)
    {
        hipFree(d_q_one[gpu][0]);
        hipFree(d_q_one[gpu][1]);
        hipFree(d_q_step_1_one[gpu]);
        hipFree(d_q_step_2_one[gpu]);
        hipFree(d_q_step_1_two[gpu]);
        hipFree(d_qk_in_2_one[gpu]);
        hipFree(d_qk_in_1_two[gpu]);
    }
    hipFree(d_q_unity);

    // For pseudo-spectral: advance_one_propagator()
    hipFree(d_propagator_sub_dep[0]);
    hipFree(d_propagator_sub_dep[1]);

    // For stress calculation: compute_stress()
    for(int gpu=0; gpu<N_GPUS; gpu++)
    {
        hipFree(d_fourier_basis_x[gpu]);
        hipFree(d_fourier_basis_y[gpu]);
        hipFree(d_fourier_basis_z[gpu]);
        hipFree(d_stress_q[gpu][0]);
        hipFree(d_stress_q[gpu][1]);
        hipFree(d_stress_sum[gpu]);
        hipFree(d_stress_sum_out[gpu]);
        hipFree(d_q_multi[gpu]);
        hipFree(d_temp_storage[gpu]);
    }

    // For concentration computation
    hipFree(d_q_block_v[0]);
    hipFree(d_q_block_v[1]);
    hipFree(d_q_block_u[0]);
    hipFree(d_q_block_u[1]);
    hipFree(d_phi);

    // Destroy streams
    for(int gpu=0; gpu<N_GPUS; gpu++)
    {
        hipStreamDestroy(streams[gpu][0]);
        hipStreamDestroy(streams[gpu][1]);
    }
}

void CudaPseudoReduceMemoryContinuous::update_bond_function()
{
    try{
        // For pseudo-spectral: advance_one_propagator()
        const int M_COMPLEX = this->n_complex_grid;
        const int N_GPUS = CudaCommon::get_instance().get_n_gpus();
        double boltz_bond[M_COMPLEX], boltz_bond_half[M_COMPLEX];

        for(const auto& item: molecules->get_bond_lengths())
        {
            std::string monomer_type = item.first;
            double bond_length_sq = item.second*item.second;
            
            get_boltz_bond(boltz_bond     , bond_length_sq,   cb->get_nx(), cb->get_dx(), molecules->get_ds());
            get_boltz_bond(boltz_bond_half, bond_length_sq/2, cb->get_nx(), cb->get_dx(), molecules->get_ds());
        
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                gpu_error_check(hipMemcpy(d_boltz_bond     [gpu][monomer_type], boltz_bond,      sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
                gpu_error_check(hipMemcpy(d_boltz_bond_half[gpu][monomer_type], boltz_bond_half, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
            }
        }

        // For stress calculation: compute_stress()
        double fourier_basis_x[M_COMPLEX];
        double fourier_basis_y[M_COMPLEX];
        double fourier_basis_z[M_COMPLEX];
        get_weighted_fourier_basis(fourier_basis_x, fourier_basis_y, fourier_basis_z, cb->get_nx(), cb->get_dx());
        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            gpu_error_check(hipMemcpy(d_fourier_basis_x[gpu], fourier_basis_x, sizeof(double)*M_COMPLEX, hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_fourier_basis_y[gpu], fourier_basis_y, sizeof(double)*M_COMPLEX, hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_fourier_basis_z[gpu], fourier_basis_z, sizeof(double)*M_COMPLEX, hipMemcpyHostToDevice));
        }
        gpu_error_check(hipSetDevice(0));
    }
    catch(std::exception& exc)
    {
        throw_with_line_number(exc.what());
    }
}
void CudaPseudoReduceMemoryContinuous::compute_statistics(
    std::map<std::string, const double*> w_input,
    std::map<std::string, const double*> q_init, std::string device)
{
    try{
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();
        const int N_GPUS = CudaCommon::get_instance().get_n_gpus();

        const int M = cb->get_n_grid();
        const double ds = molecules->get_ds();

        hipMemcpyKind cudaMemcpyInputToDevice;
        if (device == "gpu")
            cudaMemcpyInputToDevice = hipMemcpyDeviceToDevice;
        else if(device == "cpu")
            cudaMemcpyInputToDevice = hipMemcpyHostToDevice;
        else
        {
            throw_with_line_number("Invalid device \"" + device + "\".");
        }

        for(const auto& item: molecules->get_essential_propagator_codes())
        {
            if( w_input.find(item.second.monomer_type) == w_input.end())
                throw_with_line_number("monomer_type \"" + item.second.monomer_type + "\" is not in w_input.");
        }

        for(const auto& item: w_input)
        {
            if( d_exp_dw[0].find(item.first) == d_exp_dw[0].end())
                throw_with_line_number("monomer_type \"" + item.first + "\" is not in d_exp_dw.");     
        }

        // Exp_dw and exp_dw_half
        for(const auto& item: w_input)
        {
            std::string monomer_type = item.first;
            const double *w = item.second;

            // Copy field configurations from host to device
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                gpu_error_check(hipMemcpyAsync(
                    d_exp_dw     [gpu][monomer_type], w,      
                    sizeof(double)*M, cudaMemcpyInputToDevice, streams[gpu][1]));
                gpu_error_check(hipMemcpyAsync(
                    d_exp_dw_half[gpu][monomer_type], w,
                    sizeof(double)*M, cudaMemcpyInputToDevice, streams[gpu][1]));
            }

            // Compute exp_dw and exp_dw_half
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                exp_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][1]>>>
                    (d_exp_dw[gpu][monomer_type],      d_exp_dw[gpu][monomer_type],      1.0, -0.50*ds, M);
                exp_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][1]>>>
                    (d_exp_dw_half[gpu][monomer_type], d_exp_dw_half[gpu][monomer_type], 1.0, -0.25*ds, M);
                
            }
            // Synchronize all GPUs
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                gpu_error_check(hipDeviceSynchronize());
            }
        }

        auto& branch_schedule = sc->get_schedule();
        // For each time span
        for (auto parallel_job = branch_schedule.begin(); parallel_job != branch_schedule.end(); parallel_job++)
        {
            gpu_error_check(hipSetDevice(0));
            // For each propagator
            for(size_t job=0; job<parallel_job->size(); job++)
            {
                auto& key = std::get<0>((*parallel_job)[job]);
                int n_segment_from = std::get<1>((*parallel_job)[job]);
                int n_segment_to = std::get<2>((*parallel_job)[job]);
                auto& deps = molecules->get_essential_propagator_code(key).deps;
                auto monomer_type = molecules->get_essential_propagator_code(key).monomer_type;

                // Check key
                #ifndef NDEBUG
                if (propagator.find(key) == propagator.end())
                    throw_with_line_number("Could not find key '" + key + "'. ");
                #endif
                double *_propagator_0 = propagator[key][0];

                // If it is leaf node
                if(deps.size() == 0)
                {
                    // Q_init
                    if (key[0] == '{')
                    {
                        std::string g = Molecules::get_q_input_idx_from_key(key);
                        if (q_init.find(g) == q_init.end())
                            throw_with_line_number( "Could not find q_init[\"" + g + "\"].");
                        gpu_error_check(hipMemcpy(d_q_one[0][0], q_init[g], sizeof(double)*M, cudaMemcpyInputToDevice));
                    }
                    else
                    {
                        gpu_error_check(hipMemcpy(d_q_one[0][0], d_q_unity, sizeof(double)*M, hipMemcpyDeviceToDevice));
                    }
                    gpu_error_check(hipMemcpy(_propagator_0, d_q_one[0][0], sizeof(double)*M, hipMemcpyDeviceToHost));

                    #ifndef NDEBUG
                    propagator_finished[key][0] = true;
                    #endif
                }
                // If it is not leaf node
                else if (n_segment_from == 1 && deps.size() > 0)
                {
                    // If it is superposed
                    if (key[0] == '[')
                    {
                        // Initialize to zero
                        gpu_error_check(hipMemset(d_q_one[0][0], 0, sizeof(double)*M));

                        int prev, next;
                        prev = 0;
                        next = 1;

                        // Copy memory from host to device
                        std::string sub_dep = std::get<0>(deps[0]);
                        int sub_n_segment   = std::get<1>(deps[0]);
                        int sub_n_repeated;
                        gpu_error_check(hipMemcpy(d_propagator_sub_dep[prev], propagator[sub_dep][sub_n_segment], sizeof(double)*M, hipMemcpyHostToDevice));

                        for(size_t d=0; d<deps.size(); d++)
                        {
                            sub_dep         = std::get<0>(deps[d]);
                            sub_n_segment   = std::get<1>(deps[d]);
                            sub_n_repeated  = std::get<2>(deps[d]);

                            // Check sub key
                            #ifndef NDEBUG
                            if (propagator.find(sub_dep) == propagator.end())
                                throw_with_line_number("Could not find sub key '" + sub_dep + "'. ");
                            if (!propagator_finished[sub_dep][sub_n_segment])
                                throw_with_line_number("Could not compute '" + key +  "', since '"+ sub_dep + std::to_string(sub_n_segment) + "' is not prepared.");
                            #endif

                            // STREAM 1: copy memory from host to device
                            if (d < deps.size()-1)
                            {
                                std::string sub_dep_next = std::get<0>(deps[d+1]);
                                int sub_n_segment_next   = std::get<1>(deps[d+1]);

                                gpu_error_check(hipMemcpyAsync(d_propagator_sub_dep[next],
                                                propagator[sub_dep_next][sub_n_segment_next], sizeof(double)*M,
                                                hipMemcpyHostToDevice, streams[0][1]));
                            }

                            // STREAM 0: compute linear combination
                            lin_comb<<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(
                                    d_q_one[0][0], 1.0, d_q_one[0][0],
                                    sub_n_repeated, d_propagator_sub_dep[prev], M);

                            std::swap(prev, next);
                            hipDeviceSynchronize();
                        }
                        gpu_error_check(hipMemcpy(_propagator_0, d_q_one[0][0], sizeof(double)*M, hipMemcpyDeviceToHost));
                        
                        #ifndef NDEBUG
                        propagator_finished[key][0] = true;
                        #endif
                    }
                    else
                    {
                        // Initialize to one
                        gpu_error_check(hipMemcpy(d_q_one[0][0], d_q_unity, sizeof(double)*M, hipMemcpyDeviceToDevice));

                        int prev, next;
                        prev = 0;
                        next = 1;

                        // Copy memory from host to device
                        std::string sub_dep = std::get<0>(deps[0]);
                        int sub_n_segment   = std::get<1>(deps[0]);
                        gpu_error_check(hipMemcpy(d_propagator_sub_dep[prev], propagator[sub_dep][sub_n_segment], sizeof(double)*M, hipMemcpyHostToDevice));

                        for(size_t d=0; d<deps.size(); d++)
                        {
                            std::string sub_dep = std::get<0>(deps[d]);
                            int sub_n_segment   = std::get<1>(deps[d]);

                            // Check sub key
                            #ifndef NDEBUG
                            if (propagator.find(sub_dep) == propagator.end())
                                throw_with_line_number("Could not find sub key '" + sub_dep + "'. ");
                            if (!propagator_finished[sub_dep][sub_n_segment])
                                throw_with_line_number("Could not compute '" + key +  "', since '"+ sub_dep + std::to_string(sub_n_segment) + "' is not prepared.");
                            #endif

                            // STREAM 1: copy memory from host to device
                            if (d < deps.size()-1)
                            {
                                std::string sub_dep_next = std::get<0>(deps[d+1]);
                                int sub_n_segment_next   = std::get<1>(deps[d+1]);

                                gpu_error_check(hipMemcpyAsync(d_propagator_sub_dep[next],
                                                propagator[sub_dep_next][sub_n_segment_next], sizeof(double)*M,
                                                hipMemcpyHostToDevice, streams[0][1]));
                            }

                            // STREAM 0: multiply 
                            multi_real<<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(
                                d_q_one[0][0], d_q_one[0][0], d_propagator_sub_dep[prev], 1.0, M);

                            std::swap(prev, next);
                            hipDeviceSynchronize();
                        }
                        gpu_error_check(hipMemcpy(_propagator_0, d_q_one[0][0], sizeof(double)*M, hipMemcpyDeviceToHost));
                        #ifndef NDEBUG
                        propagator_finished[key][0] = true;
                        #endif
                    }
                }
                hipDeviceSynchronize();
            }
            // Synchronize all GPUs
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                gpu_error_check(hipDeviceSynchronize());
            }

            // Copy jobs that have non-zero segments
            std::vector<std::tuple<std::string, int, int>> non_zero_segment_jobs;
            for (auto it = parallel_job->begin(); it != parallel_job->end(); it++)
            {
                int n_segment_from = std::get<1>(*it);
                int n_segment_to = std::get<2>(*it);
                if(n_segment_to-n_segment_from >= 0)
                    non_zero_segment_jobs.push_back(*it);
            }

            // Advance propagator successively
            if(N_GPUS > 1 && non_zero_segment_jobs.size() == 2)
            {
                const int N_JOBS = non_zero_segment_jobs.size();
                std::string keys[N_JOBS];
                int n_segment_froms[N_JOBS];
                int n_segment_tos[N_JOBS];
                std::string monomer_types[N_JOBS];
                double **_propagator_keys[N_JOBS];
                
                for(int j=0; j<N_JOBS; j++)
                {
                    keys[j] = std::get<0>(non_zero_segment_jobs[j]);
                    n_segment_froms[j] = std::get<1>(non_zero_segment_jobs[j]);
                    n_segment_tos[j] = std::get<2>(non_zero_segment_jobs[j]);
                    monomer_types[j] = molecules->get_essential_propagator_code(keys[j]).monomer_type;
                    _propagator_keys[j] = propagator[keys[j]];
                }

                int prev, next;
                prev = 0;
                next = 1;

                // Copy propagators from host to device
                for(int gpu=0; gpu<N_GPUS; gpu++)
                    gpu_error_check(hipMemcpy(d_q_one[gpu][prev], _propagator_keys[gpu][n_segment_froms[gpu]-1], sizeof(double)*M,
                        hipMemcpyHostToDevice));

                for(int n=0; n<=n_segment_tos[0]-n_segment_froms[0]; n++)
                {
                    #ifndef NDEBUG
                    for(int gpu=0; gpu<N_GPUS; gpu++)
                    {
                        if (!propagator_finished[keys[gpu]][n-1+n_segment_froms[gpu]])
                            throw_with_line_number("unfinished, key: " + keys[gpu] + ", " + std::to_string(n-1+n_segment_froms[gpu]));
                    }
                    #endif

                    // DEVICE 0,1, STREAM 0: calculate propagators 
                    advance_two_propagators_two_gpus(
                        d_q_one[0][prev],
                        d_q_one[1][prev],
                        d_q_one[0][next],
                        d_q_one[1][next],
                        d_boltz_bond[0][monomer_types[0]],
                        d_boltz_bond[1][monomer_types[1]],
                        d_boltz_bond_half[0][monomer_types[0]],
                        d_boltz_bond_half[1][monomer_types[1]],
                        d_exp_dw[0][monomer_types[0]],
                        d_exp_dw[1][monomer_types[1]],
                        d_exp_dw_half[0][monomer_types[0]],
                        d_exp_dw_half[1][monomer_types[1]]);

                    // STREAM 1: copy propagators from device to host
                    for(int gpu=0; gpu<N_GPUS; gpu++)
                    {
                        gpu_error_check(hipSetDevice(gpu));
                        if (n > 0)
                        {
                            gpu_error_check(hipMemcpyAsync(_propagator_keys[gpu][n-1+n_segment_froms[gpu]], d_q_one[gpu][prev], sizeof(double)*M,
                                hipMemcpyDeviceToHost, streams[gpu][1]));
                        }
                    }

                    // Synchronize all GPUs
                    for(int gpu=0; gpu<N_GPUS; gpu++)
                    {
                        gpu_error_check(hipSetDevice(gpu));
                        gpu_error_check(hipDeviceSynchronize());
                    }
                    std::swap(prev, next);

                    #ifndef NDEBUG
                    for(int gpu=0; gpu<N_GPUS; gpu++)
                        propagator_finished[keys[gpu]][n+n_segment_froms[gpu]] = true;
                    #endif
                }
                for(int gpu=0; gpu<N_GPUS; gpu++)
                {
                    // Copy propagators from device to host
                    gpu_error_check(hipMemcpy(_propagator_keys[gpu][n_segment_tos[gpu]], d_q_one[gpu][prev], sizeof(double)*M,
                        hipMemcpyDeviceToHost));
                }
            }
            else if(non_zero_segment_jobs.size() > 0)
            {
                const int N_JOBS = non_zero_segment_jobs.size();
                std::string keys[N_JOBS];
                int n_segment_froms[N_JOBS];
                int n_segment_tos[N_JOBS];
                std::string monomer_types[N_JOBS];
                double **_propagator_keys[N_JOBS];
                
                for(int j=0; j<N_JOBS; j++)
                {
                    keys[j] = std::get<0>(non_zero_segment_jobs[j]);
                    n_segment_froms[j] = std::get<1>(non_zero_segment_jobs[j]);
                    n_segment_tos[j] = std::get<2>(non_zero_segment_jobs[j]);
                    monomer_types[j] = molecules->get_essential_propagator_code(keys[j]).monomer_type;
                    _propagator_keys[j] = propagator[keys[j]];
                }
                for(int j=0; j<N_JOBS; j++)
                {
                    int prev, next;
                    prev = 0;
                    next = 1;

                    gpu_error_check(hipSetDevice(0));
                    // Copy propagators from host to device
                    gpu_error_check(hipMemcpy(d_q_one[0][prev], _propagator_keys[j][n_segment_froms[j]-1], sizeof(double)*M,
                        hipMemcpyHostToDevice));

                    for(int n=n_segment_froms[j]; n<=n_segment_tos[j]; n++)
                    {
                        #ifndef NDEBUG
                        if (!propagator_finished[keys[j]][n-1])
                            throw_with_line_number("unfinished, key: " + keys[j] + ", " + std::to_string(n-1));
                        #endif

                        // STREAM 0: compute propagator
                        advance_one_propagator(0, 
                            d_q_one[0][prev],
                            d_q_one[0][next],
                            d_boltz_bond[0][monomer_types[j]],
                            d_boltz_bond_half[0][monomer_types[j]],
                            d_exp_dw[0][monomer_types[j]],
                            d_exp_dw_half[0][monomer_types[j]]);

                        // STREAM 1: copy propagators from device to host
                        if (n > n_segment_froms[j])
                        {
                            gpu_error_check(hipMemcpyAsync(_propagator_keys[j][n-1], d_q_one[0][prev], sizeof(double)*M,
                                hipMemcpyDeviceToHost, streams[0][1]));
                        }

                        std::swap(prev, next);
                        hipDeviceSynchronize();

                        #ifndef NDEBUG
                        propagator_finished[keys[j]][n] = true;
                        #endif
                    }
                    // Copy propagators from device to host
                    gpu_error_check(hipMemcpy(_propagator_keys[j][n_segment_tos[j]], d_q_one[0][prev], sizeof(double)*M,
                        hipMemcpyDeviceToHost));
                }
            }
            // Synchronize all GPUs
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                gpu_error_check(hipDeviceSynchronize());
            }
        }
        gpu_error_check(hipSetDevice(0));

        // Compute total partition function of each distinct polymers
        for(const auto& segment_info: single_partition_segment)
        {
            int p                = std::get<0>(segment_info);
            double *propagator_v = std::get<1>(segment_info);
            double *propagator_u = std::get<2>(segment_info);
            int n_superposed     = std::get<3>(segment_info);

            single_partitions[p]= cb->inner_product(
                propagator_v, propagator_u)/n_superposed/cb->get_volume();
        }

        // Calculate segment concentrations
        for(const auto& block: block_phi)
        {
            const auto& key = block.first;
            int p                = std::get<0>(key);
            std::string dep_v    = std::get<1>(key);
            std::string dep_u    = std::get<2>(key);

            int n_repeated;
            int n_segment_allocated = molecules->get_essential_block(key).n_segment_allocated;
            int n_segment_offset    = molecules->get_essential_block(key).n_segment_offset;
            int n_segment_original  = molecules->get_essential_block(key).n_segment_original;

            // If there is no segment
            if(n_segment_allocated == 0)
            {
                gpu_error_check(hipMemset(block.second, 0, sizeof(double)*M));
                continue;
            }

            // Check keys
            #ifndef NDEBUG
            if (propagator.find(dep_v) == propagator.end())
                std::cout << "Could not find dep_v key'" + dep_v + "'. " << std::endl;
            if (propagator.find(dep_u) == propagator.end())
                std::cout << "Could not find dep_u key'" + dep_u + "'. " << std::endl;
            #endif

            // Contains no '['
            if (dep_u.find('[') == std::string::npos)
                n_repeated = molecules->get_essential_block(key).v_u.size();
            else
                n_repeated = 1;

            // Normalization constant
            Polymer& pc = molecules->get_polymer(p);
            double norm = molecules->get_ds()*pc.get_volume_fraction()/pc.get_alpha()/single_partitions[p]*n_repeated;

            // Calculate phi of one block (possibly multiple blocks when using superposition)
            calculate_phi_one_block(
                block.second,       // Phi
                propagator[dep_v],  // dependency v
                propagator[dep_u],  // dependency u
                n_segment_allocated,
                n_segment_offset,
                n_segment_original,
                norm);
        }
        gpu_error_check(hipSetDevice(0));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}

// Advance propagator using Richardson extrapolation
void CudaPseudoReduceMemoryContinuous::advance_one_propagator(const int GPU,
    double *d_q_in, double *d_q_out,
    double *d_boltz_bond, double *d_boltz_bond_half,
    double *d_exp_dw, double *d_exp_dw_half)
{
    // Overlapping computations for 1/2 step and 1/4 step
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        // step 1/2: Evaluate exp(-w*ds/2) in real space
        // step 1/4: Evaluate exp(-w*ds/4) in real space
        real_multi_exp_dw_two<<<N_BLOCKS, N_THREADS, 0, streams[GPU][0]>>>(
            &d_q_step_1_two[GPU][0], d_q_in, d_exp_dw,
            &d_q_step_1_two[GPU][M], d_q_in, d_exp_dw_half, 1.0, M);

        // step 1/2: Execute a Forward FFT
        // step 1/4: Execute a Forward FFT
        hipfftExecD2Z(plan_for_two[GPU], d_q_step_1_two[GPU], d_qk_in_1_two[GPU]);

        // step 1/2: Multiply exp(-k^2 ds/6)  in fourier space
        // step 1/4: Multiply exp(-k^2 ds/12) in fourier space
        complex_real_multi_bond_two<<<N_BLOCKS, N_THREADS, 0, streams[GPU][0]>>>(
            &d_qk_in_1_two[GPU][0],         d_boltz_bond,
            &d_qk_in_1_two[GPU][M_COMPLEX], d_boltz_bond_half, M_COMPLEX);

        // step 1/2: Execute a backward FFT
        // step 1/4: Execute a backward FFT
        hipfftExecZ2D(plan_bak_two[GPU], d_qk_in_1_two[GPU], d_q_step_1_two[GPU]);

        // step 1/2: Evaluate exp(-w*ds/2) in real space
        // step 1/4: Evaluate exp(-w*ds/2) in real space
        real_multi_exp_dw_two<<<N_BLOCKS, N_THREADS, 0, streams[GPU][0]>>>(
            d_q_step_1_one[GPU], &d_q_step_1_two[GPU][0], d_exp_dw,
            d_q_step_2_one[GPU], &d_q_step_1_two[GPU][M], d_exp_dw, 1.0/((double)M), M);

        // step 1/4: Execute a Forward FFT
        hipfftExecD2Z(plan_for_one[GPU], d_q_step_2_one[GPU], d_qk_in_2_one[GPU]);

        // step 1/4: Multiply exp(-k^2 ds/12) in fourier space
        multi_complex_real<<<N_BLOCKS, N_THREADS, 0, streams[GPU][0]>>>(d_qk_in_2_one[GPU], d_boltz_bond_half, M_COMPLEX);

        // step 1/4: Execute a backward FFT
        hipfftExecZ2D(plan_bak_one[GPU], d_qk_in_2_one[GPU], d_q_step_2_one[GPU]);

        // step 1/4: Evaluate exp(-w*ds/4) in real space.
        multi_real<<<N_BLOCKS, N_THREADS, 0, streams[GPU][0]>>>(d_q_step_2_one[GPU], d_q_step_2_one[GPU], d_exp_dw_half, 1.0/((double)M), M);

        // Compute linear combination with 4/3 and -1/3 ratio
        lin_comb<<<N_BLOCKS, N_THREADS, 0, streams[GPU][0]>>>(d_q_out, 4.0/3.0, d_q_step_2_one[GPU], -1.0/3.0, d_q_step_1_one[GPU], M);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoReduceMemoryContinuous::advance_two_propagators_two_gpus(
    double *d_q_in_1, double *d_q_in_2,
    double *d_q_out_1, double *d_q_out_2,
    double *d_boltz_bond_1, double *d_boltz_bond_2, 
    double *d_boltz_bond_half_1, double *d_boltz_bond_half_2,         
    double *d_exp_dw_1, double *d_exp_dw_2,
    double *d_exp_dw_half_1, double *d_exp_dw_half_2)
{
    // Overlapping computations for 1/2 step and 1/4 step
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();
        const int N_GPUS = CudaCommon::get_instance().get_n_gpus();

        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        // step 1/2: Evaluate exp(-w*ds/2) in real space
        // step 1/4: Evaluate exp(-w*ds/4) in real space
        gpu_error_check(hipSetDevice(0));
        real_multi_exp_dw_two<<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(
            &d_q_step_1_two[0][0], d_q_in_1, d_exp_dw_1,
            &d_q_step_1_two[0][M], d_q_in_1, d_exp_dw_half_1, 1.0, M);
        gpu_error_check(hipSetDevice(1));
        real_multi_exp_dw_two<<<N_BLOCKS, N_THREADS, 0, streams[1][0]>>>(
            &d_q_step_1_two[1][0], d_q_in_2, d_exp_dw_2,
            &d_q_step_1_two[1][M], d_q_in_2, d_exp_dw_half_2, 1.0, M);

        // step 1/2: Execute a Forward FFT
        // step 1/4: Execute a Forward FFT
        gpu_error_check(hipSetDevice(0));
        hipfftExecD2Z(plan_for_two[0], d_q_step_1_two[0], d_qk_in_1_two[0]);
        gpu_error_check(hipSetDevice(1));
        hipfftExecD2Z(plan_for_two[1], d_q_step_1_two[1], d_qk_in_1_two[1]);

        // step 1/2: Multiply exp(-k^2 ds/6)  in fourier space
        // step 1/4: Multiply exp(-k^2 ds/12) in fourier space
        gpu_error_check(hipSetDevice(0));
        complex_real_multi_bond_two<<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(
            &d_qk_in_1_two[0][0],         d_boltz_bond_1,
            &d_qk_in_1_two[0][M_COMPLEX], d_boltz_bond_half_1, M_COMPLEX);
        gpu_error_check(hipSetDevice(1));
        complex_real_multi_bond_two<<<N_BLOCKS, N_THREADS, 0, streams[1][0]>>>(
            &d_qk_in_1_two[1][0],         d_boltz_bond_2,
            &d_qk_in_1_two[1][M_COMPLEX], d_boltz_bond_half_2, M_COMPLEX);

        // step 1/2: Execute a backward FFT
        // step 1/4: Execute a backward FFT
        gpu_error_check(hipSetDevice(0));
        hipfftExecZ2D(plan_bak_two[0], d_qk_in_1_two[0], d_q_step_1_two[0]);
        gpu_error_check(hipSetDevice(1));
        hipfftExecZ2D(plan_bak_two[1], d_qk_in_1_two[1], d_q_step_1_two[1]);

        // step 1/2: Evaluate exp(-w*ds/2) in real space
        // step 1/4: Evaluate exp(-w*ds/2) in real space
        gpu_error_check(hipSetDevice(0));
        real_multi_exp_dw_two<<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(
            d_q_step_1_one[0], &d_q_step_1_two[0][0], d_exp_dw_1,
            d_q_step_2_one[0], &d_q_step_1_two[0][M], d_exp_dw_1, 1.0/((double)M), M);
        gpu_error_check(hipSetDevice(1));
        real_multi_exp_dw_two<<<N_BLOCKS, N_THREADS, 0, streams[1][0]>>>(
            d_q_step_1_one[1], &d_q_step_1_two[1][0], d_exp_dw_2,
            d_q_step_2_one[1], &d_q_step_1_two[1][M], d_exp_dw_2, 1.0/((double)M), M);

        // step 1/4: Execute a Forward FFT
        gpu_error_check(hipSetDevice(0));
        hipfftExecD2Z(plan_for_one[0], d_q_step_2_one[0], d_qk_in_2_one[0]);
        gpu_error_check(hipSetDevice(1));
        hipfftExecD2Z(plan_for_one[1], d_q_step_2_one[1], d_qk_in_2_one[1]);

        // step 1/4: Multiply exp(-k^2 ds/12) in fourier space
        gpu_error_check(hipSetDevice(0));
        multi_complex_real<<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(d_qk_in_2_one[0], d_boltz_bond_half_1, M_COMPLEX);
        gpu_error_check(hipSetDevice(1));
        multi_complex_real<<<N_BLOCKS, N_THREADS, 0, streams[1][0]>>>(d_qk_in_2_one[1], d_boltz_bond_half_2, M_COMPLEX);

        // step 1/4: Execute a backward FFT
        gpu_error_check(hipSetDevice(0));
        hipfftExecZ2D(plan_bak_one[0], d_qk_in_2_one[0], d_q_step_2_one[0]);
        gpu_error_check(hipSetDevice(1));
        hipfftExecZ2D(plan_bak_one[1], d_qk_in_2_one[1], d_q_step_2_one[1]);

        // step 1/4: Evaluate exp(-w*ds/4) in real space.
        gpu_error_check(hipSetDevice(0));
        multi_real<<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(d_q_step_2_one[0], d_q_step_2_one[0], d_exp_dw_half_1, 1.0/((double)M), M);
        gpu_error_check(hipSetDevice(1));
        multi_real<<<N_BLOCKS, N_THREADS, 0, streams[1][0]>>>(d_q_step_2_one[1], d_q_step_2_one[1], d_exp_dw_half_2, 1.0/((double)M), M);

        // Compute linear combination with 4/3 and -1/3 ratio
        gpu_error_check(hipSetDevice(0));
        lin_comb<<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(d_q_out_1, 4.0/3.0, d_q_step_2_one[0], -1.0/3.0, d_q_step_1_one[0], M);
        gpu_error_check(hipSetDevice(1));
        lin_comb<<<N_BLOCKS, N_THREADS, 0, streams[1][0]>>>(d_q_out_2, 4.0/3.0, d_q_step_2_one[1], -1.0/3.0, d_q_step_1_one[1], M);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoReduceMemoryContinuous::calculate_phi_one_block(
    double *phi, double **q_1, double **q_2, const int N, const int N_OFFSET, const int N_ORIGINAL, const double NORM)
{
    try
    {
        gpu_error_check(hipSetDevice(0));

        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();
        const int M = cb->get_n_grid();
        std::vector<double> simpson_rule_coeff = SimpsonRule::get_coeff(N);

        int prev, next;
        prev = 0;
        next = 1;

        // Copy propagators from host to device
        gpu_error_check(hipMemcpy(d_q_block_v[prev], q_1[N_ORIGINAL-N_OFFSET], sizeof(double)*M, hipMemcpyHostToDevice));
        gpu_error_check(hipMemcpy(d_q_block_u[prev], q_2[0],                   sizeof(double)*M, hipMemcpyHostToDevice));

        // Initialize to zero
        gpu_error_check(hipMemset(d_phi, 0, sizeof(double)*M));
 
        for(int n=0; n<=N; n++)
        {
            // STREAM 1: copy propagators from host to device
            if (n+1 <= N)
            {
                gpu_error_check(hipMemcpyAsync(d_q_block_v[next], q_1[N_ORIGINAL-N_OFFSET-(n+1)],
                    sizeof(double)*M, hipMemcpyHostToDevice, streams[0][1]));
                gpu_error_check(hipMemcpyAsync(d_q_block_u[next], q_2[n+1],
                    sizeof(double)*M, hipMemcpyHostToDevice, streams[0][1]));
            }

            // STREAM 0: multiply two propagators
            add_multi_real<<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(d_phi, d_q_block_v[prev], d_q_block_u[prev], NORM*simpson_rule_coeff[n], M);
            std::swap(prev, next);
            hipDeviceSynchronize();
        }
        // Copy propagators from device to host
        gpu_error_check(hipMemcpy(phi, d_phi, sizeof(double)*M, hipMemcpyDeviceToHost));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
double CudaPseudoReduceMemoryContinuous::get_total_partition(int polymer)
{
    try
    {
        return single_partitions[polymer];
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoReduceMemoryContinuous::get_total_concentration(std::string monomer_type, double *phi)
{
    try
    {
        const int M = cb->get_n_grid();
        // Initialize array
        for(int i=0; i<M; i++)
            phi[i] = 0.0;

        // For each block
        for(const auto& block: block_phi)
        {
            std::string dep_v = std::get<1>(block.first);
            int n_segment_allocated = molecules->get_essential_block(block.first).n_segment_allocated;
            if (Molecules::get_monomer_type_from_key(dep_v) == monomer_type && n_segment_allocated != 0)
            {
                for(int i=0; i<M; i++)
                    phi[i] += block.second[i]; 
            }
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoReduceMemoryContinuous::get_total_concentration(int p, std::string monomer_type, double *phi)
{
    try
    {
        const int M = cb->get_n_grid();
        const int P = molecules->get_n_polymer_types();

        if (p < 0 || p > P-1)
            throw_with_line_number("Index (" + std::to_string(p) + ") must be in range [0, " + std::to_string(P-1) + "]");

        // Initialize array
        for(int i=0; i<M; i++)
            phi[i] = 0.0;

        // For each block
        for(const auto& block: block_phi)
        {
            int polymer_idx = std::get<0>(block.first);
            std::string dep_v = std::get<1>(block.first);
            int n_segment_allocated = molecules->get_essential_block(block.first).n_segment_allocated;
            if (polymer_idx == p && Molecules::get_monomer_type_from_key(dep_v) == monomer_type && n_segment_allocated != 0)
            {
                for(int i=0; i<M; i++)
                    phi[i] += block.second[i]; 
            }
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoReduceMemoryContinuous::get_block_concentration(int p, double *phi)
{
    try
    {
        const int M = cb->get_n_grid();
        const int P = molecules->get_n_polymer_types();

        if (p < 0 || p > P-1)
            throw_with_line_number("Index (" + std::to_string(p) + ") must be in range [0, " + std::to_string(P-1) + "]");

        if (molecules->is_using_superposition())
            throw_with_line_number("Disable 'superposition' option to invoke 'get_block_concentration'.");

        Polymer& pc = molecules->get_polymer(p);
        std::vector<Block>& blocks = pc.get_blocks();

        for(size_t b=0; b<blocks.size(); b++)
        {
            std::string dep_v = pc.get_propagator_key(blocks[b].v, blocks[b].u);
            std::string dep_u = pc.get_propagator_key(blocks[b].u, blocks[b].v);
            if (dep_v < dep_u)
                dep_v.swap(dep_u);

            double* _essential_block_phi = block_phi[std::make_tuple(p, dep_v, dep_u)];
            for(int i=0; i<M; i++)
                phi[i+b*M] = _essential_block_phi[i]; 
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
std::vector<double> CudaPseudoReduceMemoryContinuous::compute_stress()
{
    // This method should be invoked after invoking compute_statistics().

    // To calculate stress, we multiply weighted fourier basis to q(k)*q^dagger(-k).
    // We only need the real part of stress calculation.

    try{
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();
        const int N_GPUS = CudaCommon::get_instance().get_n_gpus();

        const int DIM  = cb->get_dim();
        const int M    = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        auto bond_lengths = molecules->get_bond_lengths();
        std::vector<double> stress(DIM);
        std::map<std::tuple<int, std::string, std::string>, std::array<double,3>> block_dq_dl[MAX_GPUS];
        double stress_sum_out[MAX_GPUS][3];

        // Compute stress for each block
        for(const auto& block: block_phi)
        {
            const auto& key = block.first;
            int p                = std::get<0>(key);
            std::string dep_v    = std::get<1>(key);
            std::string dep_u    = std::get<2>(key);

            const int N           = molecules->get_essential_block(key).n_segment_allocated;
            const int N_OFFSET    = molecules->get_essential_block(key).n_segment_offset;
            const int N_ORIGINAL  = molecules->get_essential_block(key).n_segment_original;
            std::string monomer_type = molecules->get_essential_block(key).monomer_type;

            // If there is no segment
            if(N == 0)
                continue;

            // Contains no '['
            int n_repeated;
            if (dep_u.find('[') == std::string::npos)
                n_repeated = molecules->get_essential_block(key).v_u.size();
            else
                n_repeated = 1;

            std::vector<double> s_coeff = SimpsonRule::get_coeff(N);
            double bond_length_sq = bond_lengths[monomer_type]*bond_lengths[monomer_type];
            double** q_1 = propagator[dep_v];    // dependency v
            double** q_2 = propagator[dep_u];    // dependency u

            std::array<double,3> _block_dq_dl[MAX_GPUS];
            for(int gpu=0; gpu<N_GPUS; gpu++)
                for(int d=0; d<3; d++)
                    _block_dq_dl[gpu][d] = 0.0;

            int prev, next;
            prev = 0;
            next = 1;

            // Copy memory from host to device
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                // Index
                int idx = gpu;
                if (idx <= N)
                {
                    gpu_error_check(hipSetDevice(gpu));
                    gpu_error_check(hipMemcpy(&d_stress_q[gpu][prev][0], q_1[N_ORIGINAL-N_OFFSET-idx],
                            sizeof(double)*M,hipMemcpyHostToDevice));
                    gpu_error_check(hipMemcpy(&d_stress_q[gpu][prev][M], q_2[idx],
                            sizeof(double)*M,hipMemcpyHostToDevice));
                }
            }

            // Compute
            for(int n=0; n<=N; n+=N_GPUS)
            {
                // STREAM 1: copy memory from host to device
                for(int gpu=0; gpu<N_GPUS; gpu++)
                {
                    // Index
                    const int idx = n + gpu;
                    const int idx_next = idx + N_GPUS;

                    gpu_error_check(hipSetDevice(gpu));
                    if (idx_next <= N)
                    {
                        gpu_error_check(hipMemcpyAsync(&d_stress_q[gpu][next][0], q_1[N_ORIGINAL-N_OFFSET-idx_next],
                                sizeof(double)*M,hipMemcpyHostToDevice, streams[gpu][1]));
                        gpu_error_check(hipMemcpyAsync(&d_stress_q[gpu][next][M], q_2[idx_next],
                                sizeof(double)*M,hipMemcpyHostToDevice, streams[gpu][1]));
                    }
                }
                // STREAM 0: execute kernels
                // Execute a Forward FFT
                for(int gpu=0; gpu<N_GPUS; gpu++)
                {
                    const int idx = n + gpu;
                    gpu_error_check(hipSetDevice(gpu));
                    if (idx <= N)
                        hipfftExecD2Z(plan_for_two[gpu], d_stress_q[gpu][prev], d_qk_in_1_two[gpu]);
                }
                // Multiply two propagators in the fourier spaces
                for(int gpu=0; gpu<N_GPUS; gpu++)
                {
                    const int idx = n + gpu;
                    gpu_error_check(hipSetDevice(gpu));
                    if (idx <= N)
                    {
                        multi_complex_conjugate<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_q_multi[gpu], &d_qk_in_1_two[gpu][0], &d_qk_in_1_two[gpu][M_COMPLEX], M_COMPLEX);
                        if ( DIM == 3 )
                        {
                            // x direction
                            multi_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_stress_sum[gpu], d_q_multi[gpu], d_fourier_basis_x[gpu], bond_length_sq, M_COMPLEX);
                            hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
                            gpu_error_check(hipMemcpyAsync(&stress_sum_out[gpu][0],d_stress_sum_out[gpu],sizeof(double),hipMemcpyDeviceToHost, streams[gpu][0]));

                            // y direction
                            multi_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_stress_sum[gpu], d_q_multi[gpu], d_fourier_basis_y[gpu], bond_length_sq, M_COMPLEX);
                            hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
                            gpu_error_check(hipMemcpyAsync(&stress_sum_out[gpu][1],d_stress_sum_out[gpu],sizeof(double),hipMemcpyDeviceToHost, streams[gpu][0]));

                            // z direction
                            multi_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_stress_sum[gpu], d_q_multi[gpu], d_fourier_basis_z[gpu], bond_length_sq, M_COMPLEX);
                            hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
                            gpu_error_check(hipMemcpyAsync(&stress_sum_out[gpu][2],d_stress_sum_out[gpu],sizeof(double),hipMemcpyDeviceToHost, streams[gpu][0]));
                        }
                        if ( DIM == 2 )
                        {
                            // y direction
                            multi_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_stress_sum[gpu], d_q_multi[gpu], d_fourier_basis_y[gpu], bond_length_sq, M_COMPLEX);
                            hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
                            gpu_error_check(hipMemcpyAsync(&stress_sum_out[gpu][0],d_stress_sum_out[gpu],sizeof(double),hipMemcpyDeviceToHost, streams[gpu][0]));

                            // z direction
                            multi_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_stress_sum[gpu], d_q_multi[gpu], d_fourier_basis_z[gpu], bond_length_sq, M_COMPLEX);
                            hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
                            gpu_error_check(hipMemcpyAsync(&stress_sum_out[gpu][1],d_stress_sum_out[gpu],sizeof(double),hipMemcpyDeviceToHost, streams[gpu][0]));
                        }
                        if ( DIM == 1 )
                        {
                            // z direction
                            multi_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_stress_sum[gpu], d_q_multi[gpu], d_fourier_basis_z[gpu], bond_length_sq, M_COMPLEX);
                            hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
                            gpu_error_check(hipMemcpyAsync(&stress_sum_out[gpu][0],d_stress_sum_out[gpu],sizeof(double),hipMemcpyDeviceToHost, streams[gpu][0]));
                        }
                        // Synchronize streams and add results
                        gpu_error_check(hipStreamSynchronize(streams[gpu][0]));
                        for(int d=0; d<DIM; d++)
                            _block_dq_dl[gpu][d] += s_coeff[idx]*stress_sum_out[gpu][d]*n_repeated;
                    }
                }
                // Synchronize all GPUs
                for(int gpu=0; gpu<N_GPUS; gpu++)
                {
                    gpu_error_check(hipSetDevice(gpu));
                    gpu_error_check(hipDeviceSynchronize());
                }
                std::swap(prev, next);
            }
            // Copy stress data
            for(int gpu=0; gpu<N_GPUS; gpu++)
                block_dq_dl[gpu][key] = _block_dq_dl[gpu];
        }
        gpu_error_check(hipSetDevice(0));

        // Compute total stress
        for(int d=0; d<DIM; d++)
            stress[d] = 0.0;
        for(const auto& block: block_phi)
        {
            const auto& key = block.first;
            int p             = std::get<0>(key);
            std::string dep_v = std::get<1>(key);
            std::string dep_u = std::get<2>(key);
            Polymer& pc  = molecules->get_polymer(p);

            for(int gpu=0; gpu<N_GPUS; gpu++)
                for(int d=0; d<DIM; d++)
                    stress[d] += block_dq_dl[gpu][key][d]*pc.get_volume_fraction()/pc.get_alpha()/single_partitions[p];
        }
        for(int d=0; d<DIM; d++)
            stress[d] /= -3.0*cb->get_lx(d)*M*M/molecules->get_ds();
            
        return stress;
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoReduceMemoryContinuous::get_chain_propagator(double *q_out, int polymer, int v, int u, int n)
{
    // This method should be invoked after invoking compute_statistics()

    // Get chain propagator for a selected polymer, block and direction.
    // This is made for debugging and testing.
    try
    {
        const int M = cb->get_n_grid();
        Polymer& pc = molecules->get_polymer(polymer);
        std::string dep = pc.get_propagator_key(v,u);

        if (molecules->get_essential_propagator_codes().find(dep) == molecules->get_essential_propagator_codes().end())
            throw_with_line_number("Could not find the propagator code '" + dep + "'. Disable 'superposition' option to obtain propagators.");

        const int N = molecules->get_essential_propagator_codes()[dep].max_n_segment;
        if (n < 0 || n > N)
            throw_with_line_number("n (" + std::to_string(n) + ") must be in range [0, " + std::to_string(N) + "]");

        double* _propagator = propagator[dep][n];
        for(int i=0; i<M; i++)
            q_out[i] = _propagator[i];
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}