#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include "CudaSolverReal.h"

CudaSolverReal::CudaSolverReal(
    ComputationBox *cb,
    Molecules *molecules,
    hipStream_t streams[MAX_GPUS][2],
    bool reduce_gpu_memory_usage)
{
    try{
        this->cb = cb;
        this->molecules = molecules;

        if(molecules->get_model_name() != "continuous")
            throw_with_line_number("Real-space method only support 'continuous' chain model.");     
        const int M = cb->get_n_grid();

        // // Create boltz_bond, boltz_bond_half, exp_dw, and exp_dw_half
        // for(const auto& item: molecules->get_bond_lengths())
        // {
        //     std::string monomer_type = item.first;
        //     exp_dw     [monomer_type] = new double[M];
        //     exp_dw_half[monomer_type] = new double[M];

        //     xl[monomer_type] = new double[M];
        //     xd[monomer_type] = new double[M];
        //     xh[monomer_type] = new double[M];

        //     yl[monomer_type] = new double[M];
        //     yd[monomer_type] = new double[M];
        //     yh[monomer_type] = new double[M];

        //     zl[monomer_type] = new double[M];
        //     zd[monomer_type] = new double[M];
        //     zh[monomer_type] = new double[M];
        // }

        update_laplacian_operator();
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
CudaSolverReal::~CudaSolverReal()
{
    const int N_GPUS = CudaCommon::get_instance().get_n_gpus();

    for(int gpu=0; gpu<N_GPUS; gpu++)
    {
        for(const auto& item: d_exp_dw[gpu])
            hipFree(item.second);
        for(const auto& item: d_exp_dw_half[gpu])
            hipFree(item.second);
    }
}
int CudaSolverReal::max_of_two(int x, int y)
{
   return (x > y) ? x : y;
}
int CudaSolverReal::min_of_two(int x, int y)
{
   return (x < y) ? x : y;
}
void CudaSolverReal::update_laplacian_operator()
{
    try
    {
        for(const auto& item: molecules->get_bond_lengths())
        {
            std::string monomer_type = item.first;
            double bond_length_sq = item.second*item.second;

            FiniteDifference::get_laplacian_matrix(
                cb->get_boundary_conditions(),
                cb->get_nx(), cb->get_dx(),
                xl[monomer_type], xd[monomer_type], xh[monomer_type],
                yl[monomer_type], yd[monomer_type], yh[monomer_type],
                zl[monomer_type], zd[monomer_type], zh[monomer_type],
                bond_length_sq, molecules->get_ds());
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaSolverReal::update_dw(std::string device, std::map<std::string, const double*> w_input)
{
    try{
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();
        const int N_GPUS = CudaCommon::get_instance().get_n_gpus();

        const int M = cb->get_n_grid();
        const double ds = molecules->get_ds();

        for(const auto& item: w_input)
        {
            if( d_exp_dw[0].find(item.first) == d_exp_dw[0].end())
                throw_with_line_number("monomer_type \"" + item.first + "\" is not in d_exp_dw.");     
        }

        hipMemcpyKind cudaMemcpyInputToDevice;
        if (device == "gpu")
            cudaMemcpyInputToDevice = hipMemcpyDeviceToDevice;
        else if(device == "cpu")
            cudaMemcpyInputToDevice = hipMemcpyHostToDevice;
        else
        {
            throw_with_line_number("Invalid device \"" + device + "\".");
        }

        // Compute exp_dw and exp_dw_half
        for(const auto& item: w_input)
        {
            std::string monomer_type = item.first;
            const double *w = item.second;

            // Copy field configurations from host to device
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                gpu_error_check(hipMemcpyAsync(
                    d_exp_dw     [gpu][monomer_type], w,      
                    sizeof(double)*M, cudaMemcpyInputToDevice, streams[gpu][1]));
                gpu_error_check(hipMemcpyAsync(
                    d_exp_dw_half[gpu][monomer_type], w,
                    sizeof(double)*M, cudaMemcpyInputToDevice, streams[gpu][1]));
            }

            // Compute d_exp_dw and d_exp_dw_half
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                exp_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][1]>>>
                    (d_exp_dw[gpu][monomer_type],      d_exp_dw[gpu][monomer_type],      1.0, -0.50*ds, M);
                exp_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][1]>>>
                    (d_exp_dw_half[gpu][monomer_type], d_exp_dw_half[gpu][monomer_type], 1.0, -0.25*ds, M);
            }

            // Synchronize all GPUs
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                gpu_error_check(hipDeviceSynchronize());
            }
        }
        gpu_error_check(hipSetDevice(0));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaSolverReal::advance_one_propagator_continuous(
    const int GPU,
    double *d_q_in, double *d_q_out,
    std::string monomer_type, double *d_q_mask) 
{
    try
    {
        const int M = cb->get_n_grid();
        const int DIM = cb->get_dim();

        // double *_exp_dw = exp_dw[monomer_type];
        // double *_exp_dw_half = exp_dw_half[monomer_type];

        // // Evaluate exp(-w*ds/2) in real space
        // for(int i=0; i<M; i++)
        //     q_out[i] = _exp_dw[i]*q_in[i];

        // if(DIM == 3)           // input, output
        //     advance_propagator_3d(q_out, q_out, monomer_type);
        // else if(DIM == 2)
        //     advance_propagator_2d(q_out, q_out, monomer_type);
        // else if(DIM ==1 )
        //     advance_propagator_1d(q_out, q_out, monomer_type);

        // // Evaluate exp(-w*ds/2) in real space
        // for(int i=0; i<M; i++)
        //     q_out[i] *= _exp_dw[i];

        // // Multiply mask
        // if(q_mask != nullptr)
        // {
        //     for(int i=0; i<M; i++)
        //         q_out[i] *= q_mask[i];
        // }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}

void CudaSolverReal::advance_propagator_3d(
    double *q_in, double *q_out, std::string monomer_type)
{
    try
    {
        const int M = cb->get_n_grid();
        const std::vector<int> nx = cb->get_nx();
        double q_star[M];
        double q_dstar[M];
        double temp1[nx[0]];
        double temp2[nx[1]];
        double temp3[nx[2]];

        double *_xl = xl[monomer_type];
        double *_xd = xd[monomer_type];
        double *_xh = xh[monomer_type];

        double *_yl = yl[monomer_type];
        double *_yd = yd[monomer_type];
        double *_yh = yh[monomer_type];

        double *_zl = zl[monomer_type];
        double *_zd = zd[monomer_type];
        double *_zh = zh[monomer_type];

        int im, ip, jm, jp, km, kp;

        // Calculate q_star
        for(int j=0;j<nx[1];j++)
        {
            jm = max_of_two(0,j-1);
            jp = min_of_two(nx[1]-1,j+1);

            for(int k=0;k<nx[2];k++)
            {
                km = max_of_two(0,k-1);
                kp = min_of_two(nx[2]-1,k+1);

                // B part of Ax=B matrix equation
                for(int i=0;i<nx[0];i++)
                {
                    im = max_of_two(0,i-1);
                    ip = min_of_two(nx[0]-1,i+1);

                    int i_j_k  = i*nx[1]*nx[2] + j*nx[2] + k;
                    int im_j_k = im*nx[1]*nx[2] + j*nx[2] + k;
                    int ip_j_k = ip*nx[1]*nx[2] + j*nx[2] + k;
                    int i_jm_k = i*nx[1]*nx[2] + jm*nx[2] + k;
                    int i_jp_k = i*nx[1]*nx[2] + jp*nx[2] + k;
                    int i_j_km = i*nx[1]*nx[2] + j*nx[2] + km;
                    int i_j_kp = i*nx[1]*nx[2] + j*nx[2] + kp;

                    temp1[i] = 2.0*((3.0-0.5*_xd[i]-_yd[j]-_zd[k])*q_in[i_j_k]
                            - _zl[k]*q_in[i_j_km] - _zh[k]*q_in[i_j_kp]
                            - _yl[j]*q_in[i_jm_k] - _yh[j]*q_in[i_jp_k])
                            - _xl[i]*q_in[im_j_k] - _xh[i]*q_in[ip_j_k];

                }
                int j_k = j*nx[2] + k;
                tridiagonal(_xl, _xd, _xh, &q_star[j_k], nx[1]*nx[2], temp1, nx[0]);
            }
        }
        // Calculate q_dstar
        for(int i=0;i<nx[0];i++)
        {
            for(int k=0;k<nx[2];k++)
            {
                for(int j=0;j<nx[1];j++)
                {
                    jm = max_of_two(0,j-1);
                    jp = min_of_two(nx[1]-1,j+1);

                    int i_j_k  = i*nx[1]*nx[2] + j*nx[2] + k;
                    int i_jm_k = i*nx[1]*nx[2] + jm*nx[2] + k;
                    int i_jp_k = i*nx[1]*nx[2] + jp*nx[2] + k;

                    temp2[j] = q_star[i_j_k] + (_yd[j]-1.0)*q_out[i_j_k]
                        + _yl[j]*q_out[i_jm_k] + _yh[j]*q_out[i_jp_k];
                }
                int i_k = i*nx[1]*nx[2] + k;
                tridiagonal(_yl, _yd, _yh, &q_dstar[i_k], nx[2], temp2, nx[1]);
            }
        }

        // Calculate q^(n+1)
        for(int i=0;i<nx[0];i++)
        {
            for(int j=0;j<nx[1];j++)
            {
                for(int k=0;k<nx[2];k++)
                {
                    km = max_of_two(0,k-1);
                    kp = min_of_two(nx[2]-1,k+1);

                    int i_j_k  = i*nx[1]*nx[2] + j*nx[2] + k;
                    int i_j_km = i*nx[1]*nx[2] + j*nx[2] + km;
                    int i_j_kp = i*nx[1]*nx[2] + j*nx[2] + kp;

                    temp3[k] = q_dstar[i_j_k] + (_zd[k]-1.0)*q_out[i_j_k]
                        + _zl[k]*q_out[i_j_km] + _zh[k]*q_out[i_j_kp];
                }
                int i_j = i*nx[1]*nx[2] + j*nx[2];
                tridiagonal(_zl, _zd, _zh, &q_out[i_j], 1, temp3, nx[2]);
            }
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaSolverReal::advance_propagator_2d(
    double *q_in, double *q_out, std::string monomer_type)
{
    try
    {
        const int M = cb->get_n_grid();
        const std::vector<int> nx = cb->get_nx();
        double q_star[M];
        double temp1[nx[0]];
        double temp2[nx[1]];

        double *_xl = xl[monomer_type];
        double *_xd = xd[monomer_type];
        double *_xh = xh[monomer_type];

        double *_yl = yl[monomer_type];
        double *_yd = yd[monomer_type];
        double *_yh = yh[monomer_type];

        int im, ip, jm, jp;

        // Calculate q_star
        for(int j=0;j<nx[1];j++)
        {
            jm = max_of_two(0,j-1);
            jp = min_of_two(nx[1]-1,j+1);

            // B part of Ax=B matrix equation
            for(int i=0;i<nx[0];i++)
            {
                im = max_of_two(0,i-1);
                ip = min_of_two(nx[0]-1,i+1);

                int i_j = i*nx[1] + j;
                int i_jm = i*nx[1] + jm;
                int i_jp = i*nx[1] + jp;
                int im_j = im*nx[1] + j;
                int ip_j = ip*nx[1] + j;

                temp1[i] = 2.0*((2.0-0.5*_xd[i]-_yd[j])*q_in[i_j]
                          - _yl[j]*q_in[i_jm] - _yh[j]*q_in[i_jp])
                          - _xl[i]*q_in[im_j] - _xh[i]*q_in[ip_j];
            }
            tridiagonal(_xl, _xd, _xh, &q_star[j], nx[1], temp1, nx[0]);
        }

        // Calculate q_dstar
        for(int i=0;i<nx[0];i++)
        {
            for(int j=0;j<nx[1];j++)
            {
                jm = max_of_two(0,j-1);
                jp = min_of_two(nx[1]-1,j+1);

                int i_j = i*nx[1] + j;
                int i_jm = i*nx[1] + jm;
                int i_jp = i*nx[1] + jp;

                temp2[j] = q_star[i_j] + (_yd[j]-1.0)*q_out[i_j]
                    + _yl[j]*q_out[i_jm] + _yh[j]*q_out[i_jp];
            }
            tridiagonal(_yl, _yd, _yh, &q_out[i*nx[1]], 1, temp2, nx[1]);
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaSolverReal::advance_propagator_1d(
    double *q_in, double *q_out, std::string monomer_type)
{
    try
    {
        const int M = cb->get_n_grid();
        const std::vector<int> nx = cb->get_nx();
        double q_star[nx[0]];

        double *_xl = xl[monomer_type];
        double *_xd = xd[monomer_type];
        double *_xh = xh[monomer_type];

        int im, ip;

        for(int i=0;i<nx[0];i++)
        {
            im = max_of_two(0,i-1);
            ip = min_of_two(nx[0]-1,i+1);

            // B part of Ax=B matrix equation
            q_star[i] = (2.0-_xd[i])*q_in[i] - _xl[i]*q_in[im] - _xh[i]*q_in[ip];
        }
        tridiagonal(_xl, _xd, _xh, q_out, 1, q_star, nx[0]);

    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaSolverReal::compute_single_segment_stress_fourier(const int GPU, double *d_q)
{
    try
    {
        throw_with_line_number("Currently, real-space does not support stress computation.");   

    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
std::vector<double> CudaSolverReal::compute_single_segment_stress_continuous(
                const int GPU, std::string monomer_type)
{
    try
    {
        const int DIM  = cb->get_dim();
        const int M    = cb->get_n_grid();
        std::vector<double> stress(DIM);

        throw_with_line_number("Currently, real-space does not support stress computation.");   

        return stress;
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}

// This method solves CX=Y, where C is a tridiagonal matrix 
void CudaSolverReal::tridiagonal(
    const double *xl, const double *xd, const double *xh,
    double *x, const int OFFSET, const double *d, const int M)
{
    // xl: a
    // xd: b
    // xh: c

    double c_star[M-1];
    double temp;

    // Forward sweep
    temp = xd[0];
    c_star[0] = xh[0]/xd[0];
    x[0] = d[0]/xd[0];

    for(int i=1; i<M; i++)
    {
        c_star[i-1] = xh[i-1]/temp;
        temp = xd[i]-xl[i]*c_star[i-1];
        x[i*OFFSET] = (d[i]-xl[i]*x[(i-1)*OFFSET])/temp;
    }

    // Backward substitution
    for(int i=M-2;i>=0; i--)
        x[i*OFFSET] = x[i*OFFSET] - c_star[i]*x[(i+1)*OFFSET];
}

// This method solves CX=Y, where C is a near-tridiagonal matrix with periodic boundary condition
void CudaSolverReal::tridiagonal_periodic(
    const double *xl, const double *xd, const double *xh,
    double *x, const int OFFSET, const double *d, const int M)
{
    // xl: a
    // xd: b
    // xh: c
    // gamma = 1.0

    double c_star[M-1];
    double q[M];
    double temp, value;

    // Forward sweep
    temp = xd[0] - 1.0 ; 
    c_star[0] = xh[0]/temp;
    x[0] = d[0]/temp;
    q[0] =  1.0/temp;

    for(int i=1; i<M-1; i++)
    {
        c_star[i-1] = xh[i-1]/temp;
        temp = xd[i]-xl[i]*c_star[i-1];
        x[i*OFFSET] = (d[i]-xl[i]*x[(i-1)*OFFSET])/temp;
        q[i]        =     (-xl[i]*q[i-1])         /temp;
    }
    c_star[M-2] = xh[M-2]/temp;
    temp = xd[M-1]-xh[M-1]*xl[0] - xl[M-1]*c_star[M-2];
    x[(M-1)*OFFSET] = ( d[M-1]-xl[M-1]*x[(M-2)*OFFSET])/temp;
    q[M-1]          = (xh[M-1]-xl[M-1]*q[M-2])         /temp;

    // Backward substitution
    for(int i=M-2;i>=0; i--)
    {
        x[i*OFFSET] = x[i*OFFSET] - c_star[i]*x[(i+1)*OFFSET];
        q[i]        = q[i]        - c_star[i]*q[i+1];
    }

    value = (x[0]+xl[0]*x[(M-1)*OFFSET])/(1.0+q[0]+xl[0]*q[M-1]);
    for(int i=0; i<M; i++)
        x[i*OFFSET] = x[i*OFFSET] - q[i]*value;
}