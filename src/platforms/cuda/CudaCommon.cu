#include "hip/hip_runtime.h"
#define THRUST_IGNORE_DEPRECATED_CPP_DIALECT
#define CUB_IGNORE_DEPRECATED_CPP_DIALECT

#include <iostream>
#include <cstdlib>
#include <string>

#include "CudaCommon.h"

void throw_on_cuda_error(hipError_t code, const char *file, int line, const char *func)
{
    if (code != hipSuccess){
        std::string file_and_line("File: \"" + std::string(file) + "\", line: " + std::to_string(line) + ", function <" + std::string(func) + ">");
        throw thrust::system_error(code, thrust::cuda_category(), file_and_line);
    }
}

CudaCommon::CudaCommon()
{
    try{
        // intialize NUM_BLOCKS and NUM_THREADS
        const char *ENV_N_BLOCKS  = getenv("LFTS_GPU_NUM_BLOCKS");
        const char *ENV_N_THREADS = getenv("LFTS_GPU_NUM_THREADS");

        std::string env_var_n_blocks (ENV_N_BLOCKS  ? ENV_N_BLOCKS  : "");
        std::string env_var_n_threads(ENV_N_THREADS ? ENV_N_THREADS : "");

        if (env_var_n_blocks.empty())
            this->n_blocks = 256;
        else
            this->n_blocks = std::stoi(env_var_n_blocks);

        if (env_var_n_threads.empty())
            this->n_threads = 256;
        else
            this->n_threads = std::stoi(env_var_n_threads);

        // the number of GPUs
        int devices_count;
        gpu_error_check(hipGetDeviceCount(&devices_count));
        const char *ENV_N_GPUS = getenv("LFTS_NUM_GPUS");
        std::string env_var_n_gpus (ENV_N_GPUS  ? ENV_N_GPUS  : "");

        if (env_var_n_gpus.empty())
            n_gpus = 1;
        else
            n_gpus = std::min(std::min(std::stoi(env_var_n_gpus), devices_count), MAX_GPUS);

        // check if can access peer GPUs
        if (n_gpus > 1)
        {
            int can_access_from_0_to_1;
            int can_access_from_1_to_0;
            gpu_error_check(hipDeviceCanAccessPeer(&can_access_from_0_to_1, 0, 1));
            gpu_error_check(hipDeviceCanAccessPeer(&can_access_from_1_to_0, 1, 0));

            if (can_access_from_0_to_1 == 1 && can_access_from_1_to_0 == 1)
            {
                gpu_error_check(hipSetDevice(0));
                gpu_error_check(hipDeviceEnablePeerAccess(1, 0));
                gpu_error_check(hipSetDevice(1));
                gpu_error_check(hipDeviceEnablePeerAccess(0, 0));
            }
            else
            {
                std::cout << "Could not establish peer access between GPUs." << std::endl;
                std::cout << "Only one GPU will be used." << std::endl;
                n_gpus = 1;
            }
        }
        gpu_error_check(hipSetDevice(0));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaCommon::set(int n_blocks, int n_threads, int process_idx)
{
    int devices_count;

    this->set_n_blocks(n_blocks);
    this->set_n_threads(n_threads);

    // change GPU setting
    gpu_error_check(hipGetDeviceCount(&devices_count));
    gpu_error_check(hipSetDevice(process_idx%devices_count));
}
int CudaCommon::get_n_blocks()
{
    return n_blocks;
}
int CudaCommon::get_n_threads()
{
    return n_threads;
}
int CudaCommon::get_n_gpus()
{
    return n_gpus;
}
void CudaCommon::set_n_blocks(int n_blocks)
{
    this->n_blocks = n_blocks;
}
void CudaCommon::set_n_threads(int n_threads)
{
    this->n_threads = n_threads;
}
void CudaCommon::set_idx(int process_idx)
{
    int devices_count;

    // change GPU setting
    gpu_error_check(hipGetDeviceCount(&devices_count));
    gpu_error_check(hipSetDevice(process_idx%devices_count));
}
__global__ void multi_real(double* dst,
                          double* src1,
                          double* src2,
                          double  a, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = a * src1[i] * src2[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void mutiple_multi_real(int n_comp,
                          double* dst,
                          double* src1,
                          double* src2,
                          double  a, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {  
        dst[i] = a * src1[i] * src2[i];
        for(int n = 1; n < n_comp; n++)
            dst[i] += a * src1[i+n*M] * src2[i+n*M];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void divide_real(double* dst,
                          double* src1,
                          double* src2,
                          double  a, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = a * src1[i]/src2[i];
        i += blockDim.x * gridDim.x;
    }
}
__global__ void add_multi_real(double* dst,
                             double* src1,
                             double* src2,
                             double  a, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = dst[i] + a * src1[i] * src2[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void lin_comb(double* dst,
                        double a,
                        double* src1,
                        double b,
                        double* src2,
                        const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = a*src1[i] + b*src2[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void add_lin_comb(double* dst,
                           double a,
                           double* src1,
                           double b,
                           double* src2,
                           const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = dst[i] + a*src1[i] + b*src2[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void multi_complex_real(ftsComplex* dst,
                                 double* src, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i].x = dst[i].x * src[i];
        dst[i].y = dst[i].y * src[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void multi_complex_real(ftsComplex* dst,
                                 double* src, double a, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i].x = a * dst[i].x * src[i];
        dst[i].y = a * dst[i].y * src[i];
        i += blockDim.x * gridDim.x;
    }
}

__global__ void multi_complex_conjugate(double* dst,
                                 ftsComplex* src1,
                                 ftsComplex* src2, const int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < M)
    {
        dst[i] = src1[i].x * src2[i].x + src1[i].y * src2[i].y;
        i += blockDim.x * gridDim.x;
    }
}
