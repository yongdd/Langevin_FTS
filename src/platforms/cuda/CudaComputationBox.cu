/*-------------------------------------------------------------
* This class defines Simulation Grids and Lengths parameters and provide
* methods that compute inner product in a given geometry.
*--------------------------------------------------------------*/
#include <iostream>
#include <thrust/reduce.h>
#include "CudaComputationBox.h"
#include "CudaCommon.h"

//----------------- Constructor -----------------------------
CudaComputationBox::CudaComputationBox(
    std::vector<int> nx, std::vector<double> lx)
    : ComputationBox(nx, lx)
{
    initialize();
}
void CudaComputationBox::initialize()
{
    gpu_error_check(hipMalloc((void**)&d_dv, sizeof(double)*n_grid));
    gpu_error_check(hipMemcpy(d_dv, dv,      sizeof(double)*n_grid,hipMemcpyHostToDevice));

    // temporal storage
    gpu_error_check(hipMalloc((void**)&d_multiple, sizeof(double)*n_grid));

    gpu_error_check(hipMalloc((void**)&d_g, sizeof(double)*n_grid));
    gpu_error_check(hipMalloc((void**)&d_h, sizeof(double)*n_grid));
    gpu_error_check(hipMalloc((void**)&d_w, sizeof(double)*n_grid));

    // allocate memory for cub reduction sum
    gpu_error_check(hipMalloc((void**)&d_sum, sizeof(double)*n_grid));
    gpu_error_check(hipMalloc((void**)&d_sum_out, sizeof(double)));
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, n_grid);
    gpu_error_check(hipMalloc(&d_temp_storage, temp_storage_bytes));
}
//----------------- Destructor -----------------------------
CudaComputationBox::~CudaComputationBox()
{
    hipFree(d_dv);

    hipFree(d_multiple);
    hipFree(d_g);
    hipFree(d_h);
    hipFree(d_w);

    hipFree(d_sum);
    hipFree(d_sum_out);
    hipFree(d_temp_storage);
}
//-----------------------------------------------------------
void CudaComputationBox::set_lx(std::vector<double> new_lx)
{
    ComputationBox::set_lx(new_lx);
    gpu_error_check(hipMemcpy(d_dv, dv,  sizeof(double)*n_grid,hipMemcpyHostToDevice));
}
//-----------------------------------------------------------
double CudaComputationBox::integral(double *g)
{
    double sum{0.0};
    for(int i=0; i<n_grid; i++)
        sum += dv[i]*g[i];
    return sum;
}
double CudaComputationBox::integral(Array& g)
{
    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    double *d_g = g.get_ptr();
    double sum{0.0};

    multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_dv, d_g, 1.0, n_grid);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, n_grid);
    gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(double), hipMemcpyDeviceToHost));
    return sum;
}
//-----------------------------------------------------------
double CudaComputationBox::inner_product(double *g, double *h)
{
    double sum{0.0};
    for(int i=0; i<n_grid; i++)
        sum += dv[i]*g[i]*h[i];
    return sum;
}
double CudaComputationBox::inner_product(Array& g, Array& h)
{
    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    double *d_g = g.get_ptr();
    double *d_h = h.get_ptr();
    double sum{0.0};

    multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_g, d_h, 1.0, n_grid);
    multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_dv, d_sum, 1.0, n_grid);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, n_grid);
    gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(double), hipMemcpyDeviceToHost));
    return sum;
}
//-----------------------------------------------------------
double CudaComputationBox::inner_product_inverse_weight(double *g, double *h, double *w)
{
    double sum{0.0};
    for(int i=0; i<n_grid; i++)
        sum += dv[i]*g[i]*h[i]/w[i];
    return sum;
}
double CudaComputationBox::inner_product_inverse_weight(Array& g, Array& h, Array& w)
{
    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    double *d_g = g.get_ptr();
    double *d_h = h.get_ptr();
    double *w_h = w.get_ptr();
    double sum{0.0};

    multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_g, d_h, 1.0, n_grid);
    multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_dv, d_sum, 1.0, n_grid);
    divide_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_sum, d_w, 1.0, n_grid);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, n_grid);
    gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(double), hipMemcpyDeviceToHost));
    return sum;
}
//-----------------------------------------------------------
double CudaComputationBox::multi_inner_product(int n_comp, double *g, double *h)
{
    double sum{0.0};
    for(int n=0; n < n_comp; n++)
    {
        for(int i=0; i<n_grid; i++)
            sum += dv[i]*g[i+n*n_grid]*h[i+n*n_grid];
    }
    return sum;
}
double CudaComputationBox::multi_inner_product(int n_comp, Array& g, Array& h)
{
    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    double *d_g = g.get_ptr();
    double *d_h = h.get_ptr();
    double sum{0.0};

    mutiple_multi_real<<<N_BLOCKS, N_THREADS>>>(n_comp, d_sum, d_g, d_h, 1.0, n_grid);
    multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_dv, d_sum, 1.0, n_grid);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, n_grid);
    gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(double), hipMemcpyDeviceToHost));
    return sum;
}
//-----------------------------------------------------------
void CudaComputationBox::zero_mean(double *g)
{
    double sum{0.0};
    for(int i=0; i<n_grid; i++)
        sum += dv[i]*g[i];
    for(int i=0; i<n_grid; i++)
        g[i] -= sum/volume;
}
void CudaComputationBox::zero_mean(Array& g)
{
    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    double *d_g = g.get_ptr();
    double sum{0.0};

    multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_dv, d_g, 1.0, n_grid);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, n_grid);
    gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(double), hipMemcpyDeviceToHost));
    linear_scaling_real<<<N_BLOCKS, N_THREADS>>>(d_g, d_g, 1.0, -sum/volume, n_grid);
}
//-----------------------------------------------------------
double CudaComputationBox::integral_gpu(double *d_g)
{
    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    double sum{0};

    multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_g, d_dv, 1.0, n_grid);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, n_grid);
    gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(double),hipMemcpyDeviceToHost));
    return sum;
}
//-----------------------------------------------------------
double CudaComputationBox::inner_product_gpu(double *d_g, double *d_h)
{
    const int N_BLOCKS = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();

    multi_real<<<N_BLOCKS, N_THREADS>>>(d_multiple, d_g, d_h, 1.0, n_grid);
    return CudaComputationBox::integral_gpu(d_multiple);
}
//-----------------------------------------------------------
double CudaComputationBox::inner_product_inverse_weight_gpu(double *d_g, double *d_h, double *d_w)
{
    const int N_BLOCKS = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();

    multi_real <<<N_BLOCKS, N_THREADS>>>(d_multiple, d_g,        d_h, 1.0, n_grid);
    divide_real<<<N_BLOCKS, N_THREADS>>>(d_multiple, d_multiple, d_w, 1.0, n_grid);
    return CudaComputationBox::integral_gpu(d_multiple);
}
//-----------------------------------------------------------
double CudaComputationBox::mutiple_inner_product_gpu(int n_comp, double *d_g, double *d_h)
{
    const int N_BLOCKS = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();

    mutiple_multi_real<<<N_BLOCKS, N_THREADS>>>(n_comp, d_multiple, d_g, d_h, 1.0, n_grid);
    return CudaComputationBox::integral_gpu(d_multiple);
}
