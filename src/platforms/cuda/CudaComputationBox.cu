/*-------------------------------------------------------------
* This class defines Simulation Grids and Lengths parameters and provide
* methods that compute inner product in a given geometry.
*--------------------------------------------------------------*/
#include <iostream>
#include <thrust/reduce.h>
#include "CudaComputationBox.h"
#include "CudaCommon.h"

//----------------- Constructor -----------------------------
CudaComputationBox::CudaComputationBox(
    std::vector<int> nx, std::vector<double> lx)
    : ComputationBox(nx, lx)
{
    initialize();
}
void CudaComputationBox::initialize()
{
    gpu_error_check(hipMalloc((void**)&d_dv, sizeof(double)*n_grid));
    gpu_error_check(hipMemcpy(d_dv, dv,      sizeof(double)*n_grid,hipMemcpyHostToDevice));

    // temporal storage
    gpu_error_check(hipMalloc((void**)&d_multiple, sizeof(double)*n_grid));

    gpu_error_check(hipMalloc((void**)&d_g, sizeof(double)*n_grid));
    gpu_error_check(hipMalloc((void**)&d_h, sizeof(double)*n_grid));
    gpu_error_check(hipMalloc((void**)&d_w, sizeof(double)*n_grid));

    // allocate memory for cub reduction sum
    gpu_error_check(hipMalloc((void**)&d_sum, sizeof(double)*n_grid));
    gpu_error_check(hipMalloc((void**)&d_sum_out, sizeof(double)));
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, n_grid);
    gpu_error_check(hipMalloc(&d_temp_storage, temp_storage_bytes));
}
//----------------- Destructor -----------------------------
CudaComputationBox::~CudaComputationBox()
{
    hipFree(d_dv);

    hipFree(d_multiple);
    hipFree(d_g);
    hipFree(d_h);
    hipFree(d_w);

    hipFree(d_sum);
    hipFree(d_sum_out);
    hipFree(d_temp_storage);
}
//-----------------------------------------------------------
void CudaComputationBox::set_lx(std::vector<double> new_lx)
{
    ComputationBox::set_lx(new_lx);
    gpu_error_check(hipMemcpy(d_dv, dv,  sizeof(double)*n_grid,hipMemcpyHostToDevice));
}
// //-----------------------------------------------------------
// double CudaComputationBox::integral(double *g)
// {
//     const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
//     const int N_THREADS = CudaCommon::get_instance().get_n_threads();
//     double sum{0.0};

//     gpu_error_check(hipMemcpy(d_g, g,  sizeof(double)*n_grid, hipMemcpyHostToDevice));
//     multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_dv, d_g, 1.0, n_grid);
//     hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, n_grid);
//     gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(double), hipMemcpyDeviceToHost));
//     return sum;
// }
// //-----------------------------------------------------------
// double CudaComputationBox::inner_product(double *g, double *h)
// {
//     const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
//     const int N_THREADS = CudaCommon::get_instance().get_n_threads();
//     double sum{0.0};

//     gpu_error_check(hipMemcpy(d_g, g,  sizeof(double)*n_grid, hipMemcpyHostToDevice));
//     gpu_error_check(hipMemcpy(d_h, h,  sizeof(double)*n_grid, hipMemcpyHostToDevice));
//     multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_g, d_h, 1.0, n_grid);
//     multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_dv, d_sum, 1.0, n_grid);
//     hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, n_grid);
//     gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(double), hipMemcpyDeviceToHost));
//     return sum;
// }
// //-----------------------------------------------------------
// double CudaComputationBox::inner_product_inverse_weight(double *g, double *h, double *w)
// {
//     const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
//     const int N_THREADS = CudaCommon::get_instance().get_n_threads();
//     double sum{0.0};

//     gpu_error_check(hipMemcpy(d_g, g,  sizeof(double)*n_grid, hipMemcpyHostToDevice));
//     gpu_error_check(hipMemcpy(d_h, h,  sizeof(double)*n_grid, hipMemcpyHostToDevice));
//     gpu_error_check(hipMemcpy(d_w, w,  sizeof(double)*n_grid, hipMemcpyHostToDevice));
//     multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_g, d_h, 1.0, n_grid);
//     multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_dv, d_sum, 1.0, n_grid);
//     divide_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_sum, d_w, 1.0, n_grid);
//     hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, n_grid);
//     gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(double), hipMemcpyDeviceToHost));
//     return sum;
// }
// //-----------------------------------------------------------
// double CudaComputationBox::multi_inner_product(int n_comp, double *g, double *h)
// {
//     double sum{0.0};
//     for(int n=0; n < n_comp; n++)
//     {
//         for(int i=0; i<n_grid; i++)
//             sum += dv[i]*g[i+n*n_grid]*h[i+n*n_grid];
//     }
//     return sum;
// }
// //-----------------------------------------------------------
// void CudaComputationBox::zero_mean(double *g)
// {
//     const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
//     const int N_THREADS = CudaCommon::get_instance().get_n_threads();
//     double sum{0.0};

//     gpu_error_check(hipMemcpy(d_g, g,  sizeof(double)*n_grid, hipMemcpyHostToDevice));
//     multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_dv, d_g, 1.0, n_grid);
//     hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, n_grid);
//     gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(double), hipMemcpyDeviceToHost));

//     add_scalar_constant<<<N_BLOCKS, N_THREADS>>>(d_g, -sum/volume, n_grid);
//     gpu_error_check(hipMemcpy(g, d_g, sizeof(double)*n_grid, hipMemcpyDeviceToHost));
// }
//-----------------------------------------------------------
double CudaComputationBox::integral_gpu(double *d_g)
{
    const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();
    double sum{0};

    multi_real<<<N_BLOCKS, N_THREADS>>>(d_sum, d_g, d_dv, 1.0, n_grid);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_sum, d_sum_out, n_grid);
    gpu_error_check(hipMemcpy(&sum, d_sum_out, sizeof(double),hipMemcpyDeviceToHost));
    return sum;
}
//-----------------------------------------------------------
double CudaComputationBox::inner_product_gpu(double *d_g, double *d_h)
{
    const int N_BLOCKS = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();

    multi_real<<<N_BLOCKS, N_THREADS>>>(d_multiple, d_g, d_h, 1.0, n_grid);
    return CudaComputationBox::integral_gpu(d_multiple);
}
//-----------------------------------------------------------
double CudaComputationBox::inner_product_inverse_weight_gpu(double *d_g, double *d_h, double *d_w)
{
    const int N_BLOCKS = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();

    multi_real <<<N_BLOCKS, N_THREADS>>>(d_multiple, d_g,        d_h, 1.0, n_grid);
    divide_real<<<N_BLOCKS, N_THREADS>>>(d_multiple, d_multiple, d_w, 1.0, n_grid);
    return CudaComputationBox::integral_gpu(d_multiple);
}
//-----------------------------------------------------------
double CudaComputationBox::mutiple_inner_product_gpu(int n_comp, double *d_g, double *d_h)
{
    const int N_BLOCKS = CudaCommon::get_instance().get_n_blocks();
    const int N_THREADS = CudaCommon::get_instance().get_n_threads();

    mutiple_multi_real<<<N_BLOCKS, N_THREADS>>>(n_comp, d_multiple, d_g, d_h, 1.0, n_grid);
    return CudaComputationBox::integral_gpu(d_multiple);
}
