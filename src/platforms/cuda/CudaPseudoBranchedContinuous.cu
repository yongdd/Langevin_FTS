#include "hip/hip_runtime.h"
#define THRUST_IGNORE_DEPRECATED_CPP_DIALECT
#define CUB_IGNORE_DEPRECATED_CPP_DIALECT

#include <complex>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>

#include "CudaPseudoBranchedContinuous.h"
#include "CudaComputationBox.h"
#include "SimpsonQuadrature.h"

CudaPseudoBranchedContinuous::CudaPseudoBranchedContinuous(
    ComputationBox *cb,
    PolymerChain *pc)
    : Pseudo(cb, pc)
{
    try{
        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        // create reduced_edges, which contains partition function
        for(const auto& item: pc->get_reduced_branches_max_segment()){
            std::string dep = item.first;
            d_reduced_edges[dep].max_n_segment = item.second;
            d_reduced_edges[dep].species       = pc->key_to_species(dep);
            d_reduced_edges[dep].deps          = pc->key_to_deps(dep);
            d_reduced_edges[dep].partition     = nullptr;
            gpu_error_check(hipMalloc((void**)&d_reduced_edges[dep].partition, sizeof(double)*M*(item.second+1)));
        }

        // create reduced_blocks, which contains concentration
        std::vector<polymer_chain_block>& blocks = pc->get_blocks();
        for(int i=0; i<blocks.size(); i++){
            std::string dep_v = pc->get_dep(blocks[i].v, blocks[i].u);
            std::string dep_u = pc->get_dep(blocks[i].u, blocks[i].v);
            if (dep_v > dep_u)
                dep_v.swap(dep_u);
            std::pair<std::string, std::string> key = std::make_pair(dep_v, dep_u);
            d_reduced_blocks[key].n_segment = blocks[i].n_segment;
            d_reduced_blocks[key].species   = blocks[i].species;
        }
        for(const auto& item: d_reduced_blocks){
            d_reduced_blocks[item.first].phi = nullptr;
            gpu_error_check(hipMalloc((void**)&d_reduced_blocks[item.first].phi, sizeof(double)*M));
        }

        // create boltz_bond, boltz_bond_half, exp_dw, and exp_dw_half
        for(const auto& item: pc->get_dict_bond_lengths()){
            std::string species = item.first;
            d_boltz_bond     [species] = nullptr;
            d_boltz_bond_half[species] = nullptr;
            d_exp_dw         [species] = nullptr;
            d_exp_dw_half    [species] = nullptr;

            gpu_error_check(hipMalloc((void**)&d_exp_dw         [species], sizeof(double)*M));
            gpu_error_check(hipMalloc((void**)&d_exp_dw_half    [species], sizeof(double)*M));
            gpu_error_check(hipMalloc((void**)&d_boltz_bond     [species], sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_boltz_bond_half[species], sizeof(double)*M_COMPLEX));
        }

        // create FFT plan
        const int BATCH{1};
        const int NRANK{cb->get_dim()};
        int n_grid[NRANK];

        if(cb->get_dim() == 3)
        {
            n_grid[0] = cb->get_nx(0);
            n_grid[1] = cb->get_nx(1);
            n_grid[2] = cb->get_nx(2);
        }
        else if(cb->get_dim() == 2)
        {
            n_grid[0] = cb->get_nx(1);
            n_grid[1] = cb->get_nx(2);
        }
        else if(cb->get_dim() == 1)
        {
            n_grid[0] = cb->get_nx(2);
        }
        hipfftPlanMany(&plan_for, NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z,BATCH);
        hipfftPlanMany(&plan_bak, NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D,BATCH);

        // allocate memory for pseudo-spectral: one-step()
        gpu_error_check(hipMalloc((void**)&d_q_step1, sizeof(double)*M));
        gpu_error_check(hipMalloc((void**)&d_q_step2, sizeof(double)*M));
        gpu_error_check(hipMalloc((void**)&d_qk_in,  sizeof(ftsComplex)*M_COMPLEX));
        
        // allocate memory for stress calculation: dq_dl()
        gpu_error_check(hipMalloc((void**)&d_fourier_basis_x, sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_fourier_basis_y, sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_fourier_basis_z, sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_qk_1,        sizeof(ftsComplex)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_qk_2,        sizeof(ftsComplex)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_q_multi,         sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_stress_sum,      sizeof(double)*M_COMPLEX));

        update();
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
CudaPseudoBranchedContinuous::~CudaPseudoBranchedContinuous()
{
    hipfftDestroy(plan_for);
    hipfftDestroy(plan_bak);

    for(const auto& item: d_boltz_bond)
        hipFree(item.second);
    for(const auto& item: d_boltz_bond_half)
        hipFree(item.second);
    for(const auto& item: d_exp_dw)
        hipFree(item.second);
    for(const auto& item: d_exp_dw_half)
        hipFree(item.second);
    for(const auto& item: d_reduced_edges)
        hipFree(item.second.partition);
    for(const auto& item: d_reduced_blocks)
        hipFree(item.second.phi);

    // for pseudo-spectral: one-step()
    hipFree(d_q_step1);
    hipFree(d_q_step2);
    hipFree(d_qk_in);

    // for stress calculation: dq_dl()
    hipFree(d_fourier_basis_x);
    hipFree(d_fourier_basis_y);
    hipFree(d_fourier_basis_z);
    hipFree(d_qk_1);
    hipFree(d_qk_2);
    hipFree(d_q_multi);
    hipFree(d_stress_sum);
}

void CudaPseudoBranchedContinuous::update()
{
    try{
        // for pseudo-spectral: one-step()
        const int M_COMPLEX = this->n_complex_grid;
        double boltz_bond[M_COMPLEX], boltz_bond_half[M_COMPLEX];

        for(const auto& item: pc->get_dict_bond_lengths())
        {
            std::string species = item.first;
            double bond_length_sq = item.second*item.second;
            get_boltz_bond(boltz_bond     , bond_length_sq,   cb->get_nx(), cb->get_dx(), pc->get_ds());
            get_boltz_bond(boltz_bond_half, bond_length_sq/2, cb->get_nx(), cb->get_dx(), pc->get_ds());
        
            gpu_error_check(hipMemcpy(d_boltz_bond[species],      boltz_bond,      sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_boltz_bond_half[species], boltz_bond_half, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
        }

        // for stress calculation: dq_dl()
        double fourier_basis_x[M_COMPLEX];
        double fourier_basis_y[M_COMPLEX];
        double fourier_basis_z[M_COMPLEX];
        get_weighted_fourier_basis(fourier_basis_x, fourier_basis_y, fourier_basis_z, cb->get_nx(), cb->get_dx());
        gpu_error_check(hipMemcpy(d_fourier_basis_x, fourier_basis_x, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
        gpu_error_check(hipMemcpy(d_fourier_basis_y, fourier_basis_y, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
        gpu_error_check(hipMemcpy(d_fourier_basis_z, fourier_basis_z, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
    }
    catch(std::exception& exc)
    {
        throw_with_line_number(exc.what());
    }
}
std::vector<int> CudaPseudoBranchedContinuous::get_block_start()
{
    std::vector<int> seg_start;
    seg_start.push_back(0);
    int seg_start_temp = 0;
    for(int i=0; i<pc->get_n_block(); i++){
        seg_start_temp += pc->get_n_segment(i);
        seg_start.push_back(seg_start_temp);
    }
    return seg_start;
}
void CudaPseudoBranchedContinuous::compute_statistics(
    std::map<std::string, double*> q_init,
    std::map<std::string, double*> w_block,
    double *phi, double &single_partition)
{
    try{
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M     = cb->get_n_grid();
        const int N = pc->get_n_segment_total();
        const double ds = pc->get_ds();
        auto seg_start = get_block_start();

        for(const auto& item: d_reduced_edges)
        {
            if( w_block.count(item.second.species) == 0)
                throw_with_line_number("\"" + item.second.species + "\" species is not in w_block.");
        }

        if( q_init.size() > 0)
            throw_with_line_number("Currently, \'q_init\' is not supported for branched polymers.");

        // exp_dw and exp_dw_half
        double exp_dw[M];
        double exp_dw_half[M];
        for(const auto& item: w_block)
        {
            std::string species = item.first;
            double *w = item.second;
            for(int i=0; i<M; i++)
            { 
                exp_dw     [i] = exp(-w[i]*ds*0.5);
                exp_dw_half[i] = exp(-w[i]*ds*0.25);
            }
            gpu_error_check(hipMemcpy(d_exp_dw     [species], exp_dw,      sizeof(double)*M,hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_exp_dw_half[species], exp_dw_half, sizeof(double)*M,hipMemcpyHostToDevice));
        }

        double q_uniform[M];
        for(int i=0; i<M; i++)
            q_uniform[i] = 1.0;
        for(const auto& item: d_reduced_edges)
        {
            // calculate one block end
            if (item.second.deps.size() > 0) // if it is not leaf node
            {
                gpu_error_check(hipMemcpy(item.second.partition, q_uniform,
                    sizeof(double)*M, hipMemcpyHostToDevice));

                for(int p=0; p<item.second.deps.size(); p++)
                {
                    std::string sub_dep = item.second.deps[p].first;
                    int sub_n_segment   = item.second.deps[p].second;
                    multi_real<<<N_BLOCKS, N_THREADS>>>(
                        item.second.partition, item.second.partition,
                        &d_reduced_edges[sub_dep].partition[sub_n_segment*M], 1.0, M);
                }
            }
            else // if it is leaf node
            {
                gpu_error_check(hipMemcpy(item.second.partition, q_uniform,
                    sizeof(double)*M, hipMemcpyHostToDevice)); //* q_init
            }

            // apply the propagator successively
            for(int n=1; n<=item.second.max_n_segment; n++)
            {
                one_step(&item.second.partition[(n-1)*M],
                         &item.second.partition[n*M],
                         d_boltz_bond[item.second.species],
                         d_boltz_bond_half[item.second.species],
                         d_exp_dw[item.second.species],
                         d_exp_dw_half[item.second.species]);
            }
        }

        // calculate the single chain partition function
        std::string dep_v = d_reduced_blocks.begin()->first.first;
        std::string dep_u = d_reduced_blocks.begin()->first.second;
        int n_segment = d_reduced_blocks.begin()->second.n_segment;
        single_partition = ((CudaComputationBox *)cb)->inner_product_gpu(
            &d_reduced_edges[dep_v].partition[n_segment*M],  // q
            &d_reduced_edges[dep_u].partition[0]);           // q^dagger

        // segment concentrations
        for(const auto& item: d_reduced_blocks)
        {
            calculate_phi_one_type(
                item.second.phi,                                // phi
                d_reduced_edges[item.first.first].partition,    // dependency v
                d_reduced_edges[item.first.second].partition,   // dependency u
                item.second.n_segment);                         // n_segment
        }

        // normalize the concentration
        for(const auto& item: d_reduced_blocks)
            lin_comb<<<N_BLOCKS, N_THREADS>>>(item.second.phi, cb->get_volume()*pc->get_ds()/single_partition, item.second.phi, 0.0, item.second.phi, M);

        // copy phi
        std::vector<polymer_chain_block>& blocks = pc->get_blocks();
        for(int n=0; n<blocks.size(); n++)
        {
            std::string dep_v = pc->get_dep(blocks[n].v, blocks[n].u);
            std::string dep_u = pc->get_dep(blocks[n].u, blocks[n].v);
            if (dep_v > dep_u)
                dep_v.swap(dep_u);
            gpu_error_check(hipMemcpy(
                &phi[n*M], d_reduced_blocks[std::make_pair(dep_v, dep_u)].phi,
                sizeof(double)*M, hipMemcpyDeviceToHost));
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
// Advance two partial partition functions simultaneously using Richardson extrapolation.

void CudaPseudoBranchedContinuous::one_step(
    double *d_q_in, double *d_q_out,
    double *d_boltz_bond, double *d_boltz_bond_half,
    double *d_exp_dw, double *d_exp_dw_half)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        //-------------- step 1 ----------
        // Evaluate e^(-w*ds/2) in real space
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_step1, d_q_in, d_exp_dw, 1.0, M);

        // Execute a Forw_ard FFT
        hipfftExecD2Z(plan_for, d_q_step1, d_qk_in);

        // Multiply e^(-k^2 ds/6) in fourier space
        multi_complex_real<<<N_BLOCKS, N_THREADS>>>(d_qk_in, d_boltz_bond, M_COMPLEX);

        // Execute a backw_ard FFT
        hipfftExecZ2D(plan_bak, d_qk_in, d_q_step1);

        // Evaluate e^(-w*ds/2) in real space
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_step1, d_q_step1, d_exp_dw, 1.0/((double)M), M);

        //-------------- step 2 ----------
        // Evaluate e^(-w*ds/4) in real space
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_step2, d_q_in, d_exp_dw_half, 1.0, M);

        // Execute a Forw_ard FFT
        hipfftExecD2Z(plan_for, d_q_step2, d_qk_in);

        // Multiply e^(-k^2 ds/12) in fourier space
        multi_complex_real<<<N_BLOCKS, N_THREADS>>>(d_qk_in, d_boltz_bond_half, M_COMPLEX);

        // Execute a backw_ard FFT
        hipfftExecZ2D(plan_bak, d_qk_in, d_q_step2);

        // Evaluate e^(-w*ds/2) in real space
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_step2, d_q_step2, d_exp_dw, 1.0/((double)M), M);
        // Execute a Forw_ard FFT
        hipfftExecD2Z(plan_for, d_q_step2, d_qk_in);

        // Multiply e^(-k^2 ds/12) in fourier space
        multi_complex_real<<<N_BLOCKS, N_THREADS>>>(d_qk_in, d_boltz_bond_half, M_COMPLEX);

        // Execute a backw_ard FFT
        hipfftExecZ2D(plan_bak, d_qk_in, d_q_step2);

        // Evaluate e^(-w*ds/4) in real space.
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_step2, d_q_step2, d_exp_dw_half, 1.0/((double)M), M);
        //-------------- step 3 ----------
        lin_comb<<<N_BLOCKS, N_THREADS>>>(d_q_out, 4.0/3.0, d_q_step2, -1.0/3.0, d_q_step1, M);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoBranchedContinuous::calculate_phi_one_type(
    double *d_phi, double *d_q_1, double *d_q_2, const int N)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        std::vector<double> simpson_rule_coeff = SimpsonQuadrature::get_coeff(N);

        // Compute segment concentration
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_phi, &d_q_1[0], &d_q_2[N*M], simpson_rule_coeff[0], M);
        for(int n=1; n<=N; n++)
        {
            add_multi_real<<<N_BLOCKS, N_THREADS>>>(d_phi, &d_q_1[n*M], &d_q_2[(N-n)*M], simpson_rule_coeff[n], M);
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
std::array<double,3> CudaPseudoBranchedContinuous::dq_dl()
{
    // This method should be invoked after invoking compute_statistics().

    // To calculate stress, we multiply weighted fourier basis to q(k)*q^dagger(-k).
    // We only need the real part of stress calculation.

    try{
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int DIM  = cb->get_dim();
        const int M    = cb->get_n_grid();
        const int N    = pc->get_n_segment_total();
        const int N_B  = pc->get_n_block();
        const int M_COMPLEX = this->n_complex_grid;

        std::array<double,3> dq_dl;
        std::map<std::pair<std::string, std::string>, std::array<double,3>> reduced_dq_dl;
        std::map<std::string, double>& dict_bond_lengths = pc->get_dict_bond_lengths();
        thrust::device_ptr<double> temp_gpu_ptr(d_stress_sum);

        // compute stress for reduced key pairs
        for(const auto& item: d_reduced_blocks)
        {
            const int N = item.second.n_segment;
            auto key = item.first;
            std::string dep_v = key.first;
            std::string dep_u = key.second; 
            std::string species = item.second.species;

            std::vector<double> s_coeff = SimpsonQuadrature::get_coeff(N);
            double bond_length_sq = dict_bond_lengths[species]*dict_bond_lengths[species];
            double* d_q_1 = d_reduced_edges[dep_v].partition;    // dependency v
            double* d_q_2 = d_reduced_edges[dep_u].partition;    // dependency u

            // reset
            for(int d=0; d<3; d++)
                reduced_dq_dl[key][d] = 0.0;

            // compute
            for(int n=0; n<=N; n++)
            {
                hipfftExecD2Z(plan_for, &d_q_1[n*M],     d_qk_1);
                hipfftExecD2Z(plan_for, &d_q_2[(N-n)*M], d_qk_2);
                multi_complex_conjugate<<<N_BLOCKS, N_THREADS>>>(d_q_multi, d_qk_1, d_qk_2, M_COMPLEX);
                if ( DIM >= 3 )
                {
                    multi_real<<<N_BLOCKS, N_THREADS>>>(d_stress_sum, d_q_multi, d_fourier_basis_x, bond_length_sq, M_COMPLEX);
                    reduced_dq_dl[key][0] += s_coeff[n]*thrust::reduce(temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX);
                }
                if ( DIM >= 2 )
                {
                    multi_real<<<N_BLOCKS, N_THREADS>>>(d_stress_sum, d_q_multi, d_fourier_basis_y, bond_length_sq, M_COMPLEX);
                    reduced_dq_dl[key][1] += s_coeff[n]*thrust::reduce(temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX);
                }
                if ( DIM >= 1 )
                {
                    multi_real<<<N_BLOCKS, N_THREADS>>>(d_stress_sum, d_q_multi, d_fourier_basis_z, bond_length_sq, M_COMPLEX);
                    reduced_dq_dl[key][2] += s_coeff[n]*thrust::reduce(temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX);
                }
            }
        }

        // compute total stress
        std::vector<polymer_chain_block>& blocks = pc->get_blocks();
        for(int d=0; d<3; d++)
            dq_dl[d] = 0.0;
        for(int n=0; n<blocks.size(); n++)
        {
            std::string dep_v = pc->get_dep(blocks[n].v, blocks[n].u);
            std::string dep_u = pc->get_dep(blocks[n].u, blocks[n].v);
            if (dep_v > dep_u)
                dep_v.swap(dep_u);
            for(int d=0; d<3; d++)
                dq_dl[d] += reduced_dq_dl[std::make_pair(dep_v, dep_u)][d];
        }
        for(int d=0; d<3; d++)
            dq_dl[d] /= 3.0*cb->get_lx(d)*M*M/pc->get_ds()/cb->get_volume();

        return dq_dl;
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoBranchedContinuous::get_partition(double *q_out, int v, int u, int n)
{
    // This method should be invoked after invoking compute_statistics()

    // Get partial partition functions
    // This is made for debugging and testing
    try
    {
        const int M = cb->get_n_grid();
        std::string dep = pc->get_dep(v,u);
        const int N = d_reduced_edges[dep].max_n_segment;
        if (n < 0 || n > N)
            throw_with_line_number("n (" + std::to_string(n) + ") must be in range [0, " + std::to_string(N) + "]");

        double* partition = d_reduced_edges[dep].partition;
        gpu_error_check(hipMemcpy(q_out, &partition[n*M], sizeof(double)*M,hipMemcpyDeviceToHost));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}