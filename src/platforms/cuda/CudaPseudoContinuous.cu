#include "hip/hip_runtime.h"
#include <complex>
#include <thrust/reduce.h>
#include "CudaPseudoContinuous.h"
#include "CudaComputationBox.h"
#include "SimpsonRule.h"

CudaPseudoContinuous::CudaPseudoContinuous(
    ComputationBox *cb,
    Mixture *mx)
    : Pseudo(cb, mx)
{
    try{
        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;
        const int N_GPUS = CudaCommon::get_instance().get_n_gpus();

        // allocate memory for propagators
        gpu_error_check(hipSetDevice(0));
        if( mx->get_essential_propagator_codes().size() == 0)
            throw_with_line_number("There is no propagator code. Add polymers first.");
        for(const auto& item: mx->get_essential_propagator_codes())
        {
            std::string key = item.first;
            int max_n_segment = item.second.max_n_segment;
            
            propagator_size[key] = max_n_segment+1;
            d_propagator[key] = new double*[max_n_segment+1];
            for(int i=0; i<propagator_size[key]; i++)
                gpu_error_check(hipMalloc((void**)&d_propagator[key][i], sizeof(double)*M));

            #ifndef NDEBUG
            propagator_finished[key] = new bool[max_n_segment+1];
            for(int i=0; i<=max_n_segment;i++)
                propagator_finished[key][i] = false;
            #endif
        }

        // allocate memory for concentrations
        if( mx->get_essential_blocks().size() == 0)
            throw_with_line_number("There is no block. Add polymers first.");
        for(const auto& item: mx->get_essential_blocks())
        {
            d_block_phi[item.first] = nullptr;
            gpu_error_check(hipMalloc((void**)&d_block_phi[item.first], sizeof(double)*M));
        }

        // create boltz_bond, boltz_bond_half, exp_dw, and exp_dw_half
        for(const auto& item: mx->get_bond_lengths())
        {
            std::string monomer_type = item.first;
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                d_boltz_bond     [gpu][monomer_type] = nullptr;
                d_boltz_bond_half[gpu][monomer_type] = nullptr;
                d_exp_dw         [gpu][monomer_type] = nullptr;
                d_exp_dw_half    [gpu][monomer_type] = nullptr;

                gpu_error_check(hipMalloc((void**)&d_exp_dw         [gpu][monomer_type], sizeof(double)*M));
                gpu_error_check(hipMalloc((void**)&d_exp_dw_half    [gpu][monomer_type], sizeof(double)*M));
                gpu_error_check(hipMalloc((void**)&d_boltz_bond     [gpu][monomer_type], sizeof(double)*M_COMPLEX));
                gpu_error_check(hipMalloc((void**)&d_boltz_bond_half[gpu][monomer_type], sizeof(double)*M_COMPLEX));
            }
        }
        
        // total partition functions for each polymer
        single_partitions = new double[mx->get_n_polymers()];

        // remember one segment for each polymer chain to compute total partition function
        int current_p = 0;
        for(const auto& block: d_block_phi)
        {
            const auto& key = block.first;
            int p                = std::get<0>(key);
            std::string dep_v    = std::get<1>(key);
            std::string dep_u    = std::get<2>(key);

            // skip if already found one segment
            if (p != current_p)
                continue;

            int n_superposed;
            int n_segment_offset    = mx->get_essential_block(key).n_segment_offset;
            int n_segment_original  = mx->get_essential_block(key).n_segment_original;

            // contains no '['
            if (dep_u.find('[') == std::string::npos)
                n_superposed = 1;
            else
                n_superposed = mx->get_essential_block(key).v_u.size();

            single_partition_segment.push_back(std::make_tuple(
                p,
                d_propagator[dep_v][n_segment_original-n_segment_offset], // q
                d_propagator[dep_u][0],                                   // q_dagger
                n_superposed                    // how many propagators are aggregated
                ));
            current_p++;
        }

        // create scheduler for computation of propagator
        sc = new Scheduler(mx->get_essential_propagator_codes(), N_SCHEDULER_STREAMS); 

        // create streams
        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            gpu_error_check(hipStreamCreate(&streams[gpu][0])); // for kernel execution
            gpu_error_check(hipStreamCreate(&streams[gpu][1])); // for memcpy
        }

        // create FFT plan
        const int NRANK{cb->get_dim()};
        int n_grid[NRANK];

        if(cb->get_dim() == 3)
        {
            n_grid[0] = cb->get_nx(0);
            n_grid[1] = cb->get_nx(1);
            n_grid[2] = cb->get_nx(2);
        }
        else if(cb->get_dim() == 2)
        {
            n_grid[0] = cb->get_nx(0);
            n_grid[1] = cb->get_nx(1);
        }
        else if(cb->get_dim() == 1)
        {
            n_grid[0] = cb->get_nx(0);
        }

        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            hipfftPlanMany(&plan_for_one[gpu], NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z,1);
            hipfftPlanMany(&plan_for_two[gpu], NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z,2);
            hipfftPlanMany(&plan_bak_one[gpu], NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D,1);
            hipfftPlanMany(&plan_bak_two[gpu], NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D,2);
            hipfftSetStream(plan_for_one[gpu], streams[gpu][0]);
            hipfftSetStream(plan_for_two[gpu], streams[gpu][0]);
            hipfftSetStream(plan_bak_one[gpu], streams[gpu][0]);
            hipfftSetStream(plan_bak_two[gpu], streams[gpu][0]);
        }
        gpu_error_check(hipSetDevice(0));
        hipfftPlanMany(&plan_for_four, NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z,4);
        hipfftPlanMany(&plan_bak_four, NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D,4);
        hipfftSetStream(plan_for_four, streams[0][0]);
        hipfftSetStream(plan_bak_four, streams[0][0]);

        // allocate memory for pseudo-spectral: advance_propagator()
        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            gpu_error_check(hipMalloc((void**)&d_q_step_1_one[gpu], sizeof(double)*M));
            gpu_error_check(hipMalloc((void**)&d_q_step_2_one[gpu], sizeof(double)*M));
            gpu_error_check(hipMalloc((void**)&d_q_step_1_two[gpu], sizeof(double)*2*M));
            gpu_error_check(hipMalloc((void**)&d_q_step_2_two[gpu], sizeof(double)*2*M));

            gpu_error_check(hipMalloc((void**)&d_qk_in_2_one[gpu], sizeof(ftsComplex)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_qk_in_1_two[gpu], sizeof(ftsComplex)*2*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_qk_in_2_two[gpu], sizeof(ftsComplex)*2*M_COMPLEX));
        }
        if (N_GPUS > 1)
        {
            gpu_error_check(hipSetDevice(1));
            gpu_error_check(hipMalloc((void**)&d_propagator_device_1[0], sizeof(double)*M));  // prev
            gpu_error_check(hipMalloc((void**)&d_propagator_device_1[1], sizeof(double)*M));  // next
        }

        gpu_error_check(hipSetDevice(0));
        gpu_error_check(hipMalloc((void**)&d_q_step_1_four, sizeof(double)*4*M));
        gpu_error_check(hipMalloc((void**)&d_qk_in_1_four,  sizeof(ftsComplex)*4*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_phi,           sizeof(double)*M));
        
        double q_unity[M];
        for(int i=0; i<M; i++)
            q_unity[i] = 1.0;
        gpu_error_check(hipMalloc((void**)&d_q_unity, sizeof(double)*M));
        gpu_error_check(hipMemcpy(d_q_unity, q_unity, sizeof(double)*M, hipMemcpyHostToDevice));

        // allocate memory for stress calculation: compute_stress()
        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            gpu_error_check(hipMalloc((void**)&d_fourier_basis_x[gpu], sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_fourier_basis_y[gpu], sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_fourier_basis_z[gpu], sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_stress_sum[gpu],      sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_stress_sum_out[gpu],  sizeof(double)*1));
            gpu_error_check(hipMalloc((void**)&d_stress_q[gpu][0],     sizeof(double)*2*M)); // prev
            gpu_error_check(hipMalloc((void**)&d_stress_q[gpu][1],     sizeof(double)*2*M)); // next
            gpu_error_check(hipMalloc((void**)&d_q_multi[gpu],         sizeof(double)*M_COMPLEX));
        }

        // allocate memory for cub reduction sum
        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            d_temp_storage[gpu] = nullptr; // it seems that hipcub::DeviceReduce::Sum changes temp_storage_bytes[gpu] if d_temp_storage[gpu] is nullptr
            temp_storage_bytes[gpu] = 0;
            hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
            gpu_error_check(hipMalloc(&d_temp_storage[gpu], temp_storage_bytes[gpu]));
        }
        update_bond_function();
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
CudaPseudoContinuous::~CudaPseudoContinuous()
{
    const int N_GPUS = CudaCommon::get_instance().get_n_gpus();
    
    for(int gpu=0; gpu<N_GPUS; gpu++)
    {
        hipfftDestroy(plan_for_one[gpu]);
        hipfftDestroy(plan_for_two[gpu]);
        hipfftDestroy(plan_bak_one[gpu]);
        hipfftDestroy(plan_bak_two[gpu]);
    }
    hipfftDestroy(plan_for_four);
    hipfftDestroy(plan_bak_four);

    delete sc;

    delete[] single_partitions;

    for(int gpu=0; gpu<N_GPUS; gpu++)
    {
        for(const auto& item: d_boltz_bond[gpu])
            hipFree(item.second);
        for(const auto& item: d_boltz_bond_half[gpu])
            hipFree(item.second);
        for(const auto& item: d_exp_dw[gpu])
            hipFree(item.second);
        for(const auto& item: d_exp_dw_half[gpu])
            hipFree(item.second);
    }

    for(const auto& item: d_propagator)
    {
        for(int i=0; i<propagator_size[item.first]; i++)
            hipFree(item.second[i]);
        delete[] item.second;
    }
    for(const auto& item: d_block_phi)
        hipFree(item.second);

    #ifndef NDEBUG
    for(const auto& item: propagator_finished)
        delete[] item.second;
    #endif

    hipFree(d_phi);
    hipFree(d_q_unity);

    // for pseudo-spectral: advance_propagator()
    for(int gpu=0; gpu<N_GPUS; gpu++)
    {
        hipFree(d_q_step_1_one[gpu]);
        hipFree(d_q_step_2_one[gpu]);
        hipFree(d_q_step_1_two[gpu]);
        hipFree(d_q_step_2_two[gpu]);
        hipFree(d_qk_in_2_one[gpu]);
        hipFree(d_qk_in_1_two[gpu]);
        hipFree(d_qk_in_2_two[gpu]);
    }
    hipFree(d_qk_in_1_four);

    if (N_GPUS > 1)
    {
        hipFree(d_propagator_device_1[0]);
        hipFree(d_propagator_device_1[1]);
    }

    // for stress calculation: compute_stress()
    for(int gpu=0; gpu<N_GPUS; gpu++)
    {
        hipFree(d_fourier_basis_x[gpu]);
        hipFree(d_fourier_basis_y[gpu]);
        hipFree(d_fourier_basis_z[gpu]);
        hipFree(d_stress_q[gpu][0]);
        hipFree(d_stress_q[gpu][1]);
        hipFree(d_stress_sum[gpu]);
        hipFree(d_stress_sum_out[gpu]);
        hipFree(d_q_multi[gpu]);
        hipFree(d_temp_storage[gpu]);
    }

    // destroy streams
    for(int gpu=0; gpu<N_GPUS; gpu++)
    {
        hipStreamDestroy(streams[gpu][0]);
        hipStreamDestroy(streams[gpu][1]);
    }
}

void CudaPseudoContinuous::update_bond_function()
{
    try{
        // for pseudo-spectral: advance_propagator()
        const int M_COMPLEX = this->n_complex_grid;
        const int N_GPUS = CudaCommon::get_instance().get_n_gpus();
        double boltz_bond[M_COMPLEX], boltz_bond_half[M_COMPLEX];

        for(const auto& item: mx->get_bond_lengths())
        {
            std::string monomer_type = item.first;
            double bond_length_sq = item.second*item.second;
            
            get_boltz_bond(boltz_bond     , bond_length_sq,   cb->get_nx(), cb->get_dx(), mx->get_ds());
            get_boltz_bond(boltz_bond_half, bond_length_sq/2, cb->get_nx(), cb->get_dx(), mx->get_ds());
        
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                gpu_error_check(hipMemcpy(d_boltz_bond     [gpu][monomer_type], boltz_bond,      sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
                gpu_error_check(hipMemcpy(d_boltz_bond_half[gpu][monomer_type], boltz_bond_half, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
            }
        }

        // for stress calculation: compute_stress()
        double fourier_basis_x[M_COMPLEX];
        double fourier_basis_y[M_COMPLEX];
        double fourier_basis_z[M_COMPLEX];
        get_weighted_fourier_basis(fourier_basis_x, fourier_basis_y, fourier_basis_z, cb->get_nx(), cb->get_dx());
        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            gpu_error_check(hipMemcpy(d_fourier_basis_x[gpu], fourier_basis_x, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_fourier_basis_y[gpu], fourier_basis_y, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_fourier_basis_z[gpu], fourier_basis_z, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
        }
    }
    catch(std::exception& exc)
    {
        throw_with_line_number(exc.what());
    }
}
void CudaPseudoContinuous::compute_statistics(
    std::map<std::string, double*> w_input,
    std::map<std::string, double*> q_init)
{
    try{
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();
        const int N_GPUS = CudaCommon::get_instance().get_n_gpus();

        const int M = cb->get_n_grid();
        const double ds = mx->get_ds();

        for(const auto& item: mx->get_essential_propagator_codes())
        {
            if( w_input.find(item.second.monomer_type) == w_input.end())
                throw_with_line_number("monomer_type \"" + item.second.monomer_type + "\" is not in w_input.");
        }

        for(const auto& item: w_input)
        {
            if( d_exp_dw[0].find(item.first) == d_exp_dw[0].end())
                throw_with_line_number("monomer_type \"" + item.first + "\" is not in d_exp_dw.");     
        }

        // exp_dw and exp_dw_half
        for(const auto& item: w_input)
        {
            std::string monomer_type = item.first;
            double *w = item.second;

            // copy field configurations from host to device
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                gpu_error_check(hipMemcpyAsync(
                    d_exp_dw     [gpu][monomer_type], w,      
                    sizeof(double)*M, hipMemcpyHostToDevice, streams[gpu][1]));
                gpu_error_check(hipMemcpyAsync(
                    d_exp_dw_half[gpu][monomer_type], w,
                    sizeof(double)*M, hipMemcpyHostToDevice, streams[gpu][1]));
            }

            // compute exp_dw and exp_dw_half
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                exp_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][1]>>>
                    (d_exp_dw[gpu][monomer_type],      d_exp_dw[gpu][monomer_type],      1.0, -0.50*ds, M);
                exp_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][1]>>>
                    (d_exp_dw_half[gpu][monomer_type], d_exp_dw_half[gpu][monomer_type], 1.0, -0.25*ds, M);
                
            }
            // synchronize all GPUs
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                gpu_error_check(hipDeviceSynchronize());
            }
        }

        auto& branch_schedule = sc->get_schedule();
        // // display all jobs
        // int time_span_count=0;
        // for (auto parallel_job = branch_schedule.begin(); parallel_job != branch_schedule.end(); parallel_job++)
        // {
        //     std::cout << "jobs: " << time_span_count << std::endl;
        //     for(int job=0; job<parallel_job->size(); job++)
        //     {
        //         auto& key = std::get<0>((*parallel_job)[job]);
        //         int n_segment_from = std::get<1>((*parallel_job)[job]);
        //         int n_segment_to = std::get<2>((*parallel_job)[job]);
        //         std::cout << "key, n_segment_from, n_segment_to: " + key + ", " + std::to_string(n_segment_from) + ", " + std::to_string(n_segment_to) + ". " << std::endl;
        //     }
        //     time_span_count++;
        // }
        // time_span_count=0;

        // for each time span
        for (auto parallel_job = branch_schedule.begin(); parallel_job != branch_schedule.end(); parallel_job++)
        {
            gpu_error_check(hipSetDevice(0));
            // for each propagator
            for(size_t job=0; job<parallel_job->size(); job++)
            {
                auto& key = std::get<0>((*parallel_job)[job]);
                int n_segment_from = std::get<1>((*parallel_job)[job]);
                int n_segment_to = std::get<2>((*parallel_job)[job]);
                auto& deps = mx->get_essential_propagator_code(key).deps;
                auto monomer_type = mx->get_essential_propagator_code(key).monomer_type;

                // check key
                #ifndef NDEBUG
                if (d_propagator.find(key) == d_propagator.end())
                    throw_with_line_number("Could not find key '" + key + "'. ");
                #endif

                double **_d_propagator = d_propagator[key];

                // if it is leaf node
                if(n_segment_from == 1 && deps.size() == 0)
                {
                     // q_init
                    if (key[0] == '{')
                    {
                        std::string g = Mixture::get_q_input_idx_from_key(key);
                        if (q_init.find(g) == q_init.end())
                            throw_with_line_number("Could not find q_init[\"" + g + "\"].");
                        gpu_error_check(hipMemcpy(_d_propagator[0], q_init[g],
                            sizeof(double)*M, hipMemcpyHostToDevice));
                    }
                    else
                    {
                        gpu_error_check(hipMemcpy(_d_propagator[0], d_q_unity,
                            sizeof(double)*M, hipMemcpyDeviceToDevice));
                    }

                    #ifndef NDEBUG
                    propagator_finished[key][0] = true;
                    #endif
                }
                // if it is not leaf node
                else if (n_segment_from == 1 && deps.size() > 0)
                {
                    // if it is superposed
                    if (key[0] == '[')
                    {
                        // initialize to zero
                        gpu_error_check(hipMemset(_d_propagator[0], 0, sizeof(double)*M));

                        // add all propagators at junction if necessary 
                        for(size_t d=0; d<deps.size(); d++)
                        {
                            std::string sub_dep = std::get<0>(deps[d]);
                            int sub_n_segment   = std::get<1>(deps[d]);
                            int sub_n_repeated  = std::get<2>(deps[d]);

                            // check sub key
                            #ifndef NDEBUG
                            if (d_propagator.find(sub_dep) == d_propagator.end())
                                throw_with_line_number("Could not find sub key '" + sub_dep + "'. ");
                            if (!propagator_finished[sub_dep][sub_n_segment])
                                throw_with_line_number("Could not compute '" + key +  "', since '"+ sub_dep + std::to_string(sub_n_segment) + "' is not prepared.");
                            #endif

                            lin_comb<<<N_BLOCKS, N_THREADS>>>(
                                _d_propagator[0], 1.0, _d_propagator[0],
                                sub_n_repeated, d_propagator[sub_dep][sub_n_segment], M);
                        }

                        #ifndef NDEBUG
                        propagator_finished[key][0] = true;
                        #endif
                    }
                    else
                    {
                        // initialize to one
                        gpu_error_check(hipMemcpy(_d_propagator[0], d_q_unity,
                            sizeof(double)*M, hipMemcpyDeviceToDevice));

                        // multiply all propagators at junction if necessary 
                        for(size_t d=0; d<deps.size(); d++)
                        {
                            std::string sub_dep = std::get<0>(deps[d]);
                            int sub_n_segment   = std::get<1>(deps[d]);

                            // check sub key
                            #ifndef NDEBUG
                            if (d_propagator.find(sub_dep) == d_propagator.end())
                                throw_with_line_number("Could not find sub key '" + sub_dep + "'. ");
                            if (!propagator_finished[sub_dep][sub_n_segment])
                                throw_with_line_number("Could not compute '" + key +  "', since '"+ sub_dep + std::to_string(sub_n_segment) + "' is not prepared.");
                            #endif

                            multi_real<<<N_BLOCKS, N_THREADS>>>(
                                _d_propagator[0], _d_propagator[0],
                                d_propagator[sub_dep][sub_n_segment], 1.0, M);
                        }
                        
                        #ifndef NDEBUG
                        propagator_finished[key][0] = true;
                        #endif
                    }
                }
                hipDeviceSynchronize();
            }
            // synchronize all GPUs
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                gpu_error_check(hipDeviceSynchronize());
            }

            // copy jobs that have non-zero segments
            std::vector<std::tuple<std::string, int, int>> non_zero_segment_jobs;
            for (auto it = parallel_job->begin(); it != parallel_job->end(); it++)
            {
                int n_segment_from = std::get<1>(*it);
                int n_segment_to = std::get<2>(*it);
                if(n_segment_to-n_segment_from >= 0)
                    non_zero_segment_jobs.push_back(*it);
            }

            // advance propagator successively
            if(non_zero_segment_jobs.size()==1)
            {
                gpu_error_check(hipSetDevice(0));
                auto& key = std::get<0>(non_zero_segment_jobs[0]);
                int n_segment_from = std::get<1>(non_zero_segment_jobs[0]);
                int n_segment_to = std::get<2>(non_zero_segment_jobs[0]);
                auto monomer_type = mx->get_essential_propagator_code(key).monomer_type;
                double **_d_propagator_key = d_propagator[key];

                for(int n=n_segment_from; n<=n_segment_to; n++)
                {
                    #ifndef NDEBUG
                    if (!propagator_finished[key][n-1])
                        throw_with_line_number("unfinished, key: " + key + ", " + std::to_string(n-1));
                    #endif

                    advance_one_propagator(0, 
                        _d_propagator_key[n-1],
                        _d_propagator_key[n],
                        d_boltz_bond[0][monomer_type],
                        d_boltz_bond_half[0][monomer_type],
                        d_exp_dw[0][monomer_type],
                        d_exp_dw_half[0][monomer_type]);

                    #ifndef NDEBUG
                    propagator_finished[key][n] = true;
                    #endif
                }
            }
            else if(non_zero_segment_jobs.size()==2)
            {
                const int N_JOBS = 2;
                std::string keys[N_JOBS];
                int n_segment_froms[N_JOBS];
                int n_segment_tos[N_JOBS];
                std::string monomer_types[N_JOBS];
                double **_d_propagator_keys[N_JOBS];
                
                for(int j=0; j<N_JOBS; j++)
                {
                    keys[j] = std::get<0>(non_zero_segment_jobs[j]);
                    n_segment_froms[j] = std::get<1>(non_zero_segment_jobs[j]);
                    n_segment_tos[j] = std::get<2>(non_zero_segment_jobs[j]);
                    monomer_types[j] = mx->get_essential_propagator_code(keys[j]).monomer_type;
                    _d_propagator_keys[j] = d_propagator[keys[j]];
                }

                if (N_GPUS > 1)
                {
                    int prev, next;
                    prev = 0;
                    next = 1;

                    // copy propagator of key1 from device0 to device1
                    gpu_error_check(hipMemcpy(
                        d_propagator_device_1[prev],
                        _d_propagator_keys[1][n_segment_froms[1]-1],
                        sizeof(double)*M, hipMemcpyDeviceToDevice));

                    for(int n=0; n<=n_segment_tos[0]-n_segment_froms[0]; n++)
                    {
                        #ifndef NDEBUG
                        if (!propagator_finished[keys[0]][n-1+n_segment_froms[0]])
                            throw_with_line_number("unfinished, key: " + keys[0] + ", " + std::to_string(n-1+n_segment_froms[0]));
                        if (!propagator_finished[keys[1]][n-1+n_segment_froms[1]])
                            throw_with_line_number("unfinished, key: " + keys[1] + ", " + std::to_string(n-1+n_segment_froms[1]));
                        #endif

                        // DEVICE 0, STREAM 0: calculate propagator of key0
                        gpu_error_check(hipSetDevice(0));
                        advance_one_propagator(0,
                            _d_propagator_keys[0][n-1+n_segment_froms[0]],
                            _d_propagator_keys[0][n+n_segment_froms[0]],
                            d_boltz_bond[0][monomer_types[0]],
                            d_boltz_bond_half[0][monomer_types[0]],
                            d_exp_dw[0][monomer_types[0]],
                            d_exp_dw_half[0][monomer_types[0]]);

                        // DEVICE 1, STREAM 0: calculate propagator of key1
                        gpu_error_check(hipSetDevice(1));
                        advance_one_propagator(1,
                            d_propagator_device_1[prev],
                            d_propagator_device_1[next],
                            d_boltz_bond[1][monomer_types[1]],
                            d_boltz_bond_half[1][monomer_types[1]],
                            d_exp_dw[1][monomer_types[1]],
                            d_exp_dw_half[1][monomer_types[1]]);

                        // DEVICE 1, STREAM 1: copy memory from device 1 to device 0
                        if (n > 0)
                        {
                            gpu_error_check(hipMemcpyAsync(
                                _d_propagator_keys[1][n-1+n_segment_froms[1]],
                                d_propagator_device_1[prev],
                                sizeof(double)*M, hipMemcpyDeviceToDevice, streams[1][1]));
                        }
                        gpu_error_check(hipStreamSynchronize(streams[1][0]));
                        gpu_error_check(hipStreamSynchronize(streams[1][1]));

                        std::swap(prev, next);

                        #ifndef NDEBUG
                        propagator_finished[keys[0]][n+n_segment_froms[0]] = true;
                        propagator_finished[keys[1]][n+n_segment_froms[1]] = true;
                        #endif
                    }
                    gpu_error_check(hipMemcpy(
                        _d_propagator_keys[1][n_segment_tos[1]],
                        d_propagator_device_1[prev],
                        sizeof(double)*M, hipMemcpyDeviceToDevice));
                }
                else
                {
                    gpu_error_check(hipSetDevice(0));
                    for(int n=0; n<=n_segment_tos[0]-n_segment_froms[0]; n++)
                    {
                        #ifndef NDEBUG
                        if (!propagator_finished[keys[0]][n-1+n_segment_froms[0]])
                            throw_with_line_number("unfinished, key: " + keys[0] + ", " + std::to_string(n-1+n_segment_froms[0]));
                        if (!propagator_finished[keys[1]][n-1+n_segment_froms[1]])
                            throw_with_line_number("unfinished, key: " + keys[1] + ", " + std::to_string(n-1+n_segment_froms[1]));
                        #endif

                        advance_two_propagators(
                            _d_propagator_keys[0][n-1+n_segment_froms[0]],
                            _d_propagator_keys[1][n-1+n_segment_froms[1]],
                            _d_propagator_keys[0][n+n_segment_froms[0]],
                            _d_propagator_keys[1][n+n_segment_froms[1]],
                            d_boltz_bond[0][monomer_types[0]],
                            d_boltz_bond[0][monomer_types[1]],
                            d_boltz_bond_half[0][monomer_types[0]],
                            d_boltz_bond_half[0][monomer_types[1]],
                            d_exp_dw[0][monomer_types[0]],
                            d_exp_dw[0][monomer_types[1]],
                            d_exp_dw_half[0][monomer_types[0]],
                            d_exp_dw_half[0][monomer_types[1]]);

                        #ifndef NDEBUG
                        propagator_finished[keys[0]][n+n_segment_froms[0]] = true;
                        propagator_finished[keys[1]][n+n_segment_froms[1]] = true;
                        #endif
                    }
                }
            }
            // synchronize all GPUs
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                gpu_error_check(hipDeviceSynchronize());
            }
        }
        gpu_error_check(hipSetDevice(0));

        // compute total partition function of each distinct polymers
        for(const auto& segment_info: single_partition_segment)
        {
            int p                  = std::get<0>(segment_info);
            double *d_propagator_v = std::get<1>(segment_info);
            double *d_propagator_u = std::get<2>(segment_info);
            int n_superposed       = std::get<3>(segment_info);

            single_partitions[p] = ((CudaComputationBox *)cb)->inner_product_gpu(
                d_propagator_v, d_propagator_u)/n_superposed/cb->get_volume();
        }

        // calculate segment concentrations
        for(const auto& d_block: d_block_phi)
        {
            const auto& key = d_block.first;
            int p                = std::get<0>(key);
            std::string dep_v    = std::get<1>(key);
            std::string dep_u    = std::get<2>(key);

            int n_repeated;
            int n_segment_allocated = mx->get_essential_block(key).n_segment_allocated;
            int n_segment_offset    = mx->get_essential_block(key).n_segment_offset;
            int n_segment_original  = mx->get_essential_block(key).n_segment_original;

            // if there is no segment
            if(n_segment_allocated == 0)
            {
                gpu_error_check(hipMemset(d_block.second, 0, sizeof(double)*M));
                continue;
            }

            // check keys
            #ifndef NDEBUG
            if (d_propagator.find(dep_v) == d_propagator.end())
                throw_with_line_number("Could not find dep_v key'" + dep_v + "'. ");
            if (d_propagator.find(dep_u) == d_propagator.end())
                throw_with_line_number("Could not find dep_u key'" + dep_u + "'. ");
            #endif

            // contains no '['
            if (dep_u.find('[') == std::string::npos)
                n_repeated = mx->get_essential_block(key).v_u.size();
            else
                n_repeated = 1;

            // calculate phi of one block (possibly multiple blocks when using superposition)
            calculate_phi_one_block(
                d_block.second,       // phi
                d_propagator[dep_v],  // dependency v
                d_propagator[dep_u],  // dependency u
                n_segment_allocated,
                n_segment_offset,
                n_segment_original);

            // normalize concentration
            PolymerChain& pc = mx->get_polymer(p);
            double norm = mx->get_ds()*pc.get_volume_fraction()/pc.get_alpha()/single_partitions[p]*n_repeated;
            lin_comb<<<N_BLOCKS, N_THREADS>>>(d_block.second, norm, d_block.second, 0.0, d_block.second, M);
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}

// Advance propagator using Richardson extrapolation
void CudaPseudoContinuous::advance_one_propagator(const int GPU,
    double *d_q_in, double *d_q_out,
    double *d_boltz_bond, double *d_boltz_bond_half,
    double *d_exp_dw, double *d_exp_dw_half)
{
    // overlapping computations for 1/2 step and 1/4 step
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        // step 1/2: Evaluate exp(-w*ds/2) in real space
        // step 1/4: Evaluate exp(-w*ds/4) in real space
        real_multi_exp_dw_two<<<N_BLOCKS, N_THREADS, 0, streams[GPU][0]>>>(
            &d_q_step_1_two[GPU][0], d_q_in, d_exp_dw,
            &d_q_step_1_two[GPU][M], d_q_in, d_exp_dw_half, 1.0, M);

        // step 1/2: Execute a Forward FFT
        // step 1/4: Execute a Forward FFT
        hipfftExecD2Z(plan_for_two[GPU], d_q_step_1_two[GPU], d_qk_in_1_two[GPU]);

        // step 1/2: Multiply exp(-k^2 ds/6)  in fourier space
        // step 1/4: Multiply exp(-k^2 ds/12) in fourier space
        complex_real_multi_bond_two<<<N_BLOCKS, N_THREADS, 0, streams[GPU][0]>>>(
            &d_qk_in_1_two[GPU][0],         d_boltz_bond,
            &d_qk_in_1_two[GPU][M_COMPLEX], d_boltz_bond_half, M_COMPLEX);

        // step 1/2: Execute a backward FFT
        // step 1/4: Execute a backward FFT
        hipfftExecZ2D(plan_bak_two[GPU], d_qk_in_1_two[GPU], d_q_step_1_two[GPU]);

        // step 1/2: Evaluate exp(-w*ds/2) in real space
        // step 1/4: Evaluate exp(-w*ds/2) in real space
        real_multi_exp_dw_two<<<N_BLOCKS, N_THREADS, 0, streams[GPU][0]>>>(
            d_q_step_1_one[GPU], &d_q_step_1_two[GPU][0], d_exp_dw,
            d_q_step_2_one[GPU], &d_q_step_1_two[GPU][M], d_exp_dw, 1.0/((double)M), M);

        // step 1/4: Execute a Forward FFT
        hipfftExecD2Z(plan_for_one[GPU], d_q_step_2_one[GPU], d_qk_in_2_one[GPU]);

        // step 1/4: Multiply exp(-k^2 ds/12) in fourier space
        multi_complex_real<<<N_BLOCKS, N_THREADS, 0, streams[GPU][0]>>>(d_qk_in_2_one[GPU], d_boltz_bond_half, M_COMPLEX);

        // step 1/4: Execute a backward FFT
        hipfftExecZ2D(plan_bak_one[GPU], d_qk_in_2_one[GPU], d_q_step_2_one[GPU]);

        // step 1/4: Evaluate exp(-w*ds/4) in real space.
        multi_real<<<N_BLOCKS, N_THREADS, 0, streams[GPU][0]>>>(d_q_step_2_one[GPU], d_q_step_2_one[GPU], d_exp_dw_half, 1.0/((double)M), M);

        // compute linear combination with 4/3 and -1/3 ratio
        lin_comb<<<N_BLOCKS, N_THREADS, 0, streams[GPU][0]>>>(d_q_out, 4.0/3.0, d_q_step_2_one[GPU], -1.0/3.0, d_q_step_1_one[GPU], M);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoContinuous::advance_two_propagators(
    double *d_q_in_1, double *d_q_in_2,
    double *d_q_out_1, double *d_q_out_2,
    double *d_boltz_bond_1, double *d_boltz_bond_2, 
    double *d_boltz_bond_half_1, double *d_boltz_bond_half_2,         
    double *d_exp_dw_1, double *d_exp_dw_2,
    double *d_exp_dw_half_1, double *d_exp_dw_half_2)
{
    // overlapping computations for 1/2 step and 1/4 step using 4-batch cuFFT
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        // step 1/2: Evaluate exp(-w*ds/2) in real space
        // step 1/4: Evaluate exp(-w*ds/4) in real space
        real_multi_exp_dw_four<<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(
            &d_q_step_1_four[0],   d_q_in_1, d_exp_dw_1,
            &d_q_step_1_four[M],   d_q_in_2, d_exp_dw_2,
            &d_q_step_1_four[2*M], d_q_in_1, d_exp_dw_half_1,
            &d_q_step_1_four[3*M], d_q_in_2, d_exp_dw_half_2, 1.0, M);

        // step 1/2: Execute a Forward FFT
        // step 1/4: Execute a Forward FFT
        hipfftExecD2Z(plan_for_four, d_q_step_1_four, d_qk_in_1_four);

        // step 1/2: Multiply exp(-k^2 ds/6)  in fourier space
        // step 1/4: Multiply exp(-k^2 ds/12) in fourier space
        complex_real_multi_bond_four<<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(
            &d_qk_in_1_four[0],           d_boltz_bond_1,
            &d_qk_in_1_four[M_COMPLEX],   d_boltz_bond_2,
            &d_qk_in_1_four[2*M_COMPLEX], d_boltz_bond_half_1,
            &d_qk_in_1_four[3*M_COMPLEX], d_boltz_bond_half_2, M_COMPLEX);

        // step 1/2: Execute a backward FFT
        // step 1/4: Execute a backward FFT
        hipfftExecZ2D(plan_bak_four, d_qk_in_1_four, d_q_step_1_four);

        // step 1/2: Evaluate exp(-w*ds/2) in real space
        // step 1/4: Evaluate exp(-w*ds/2) in real space
        real_multi_exp_dw_four<<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(
            &d_q_step_1_two[0][0], &d_q_step_1_four[0],   d_exp_dw_1,
            &d_q_step_1_two[0][M], &d_q_step_1_four[M],   d_exp_dw_2,
            &d_q_step_2_two[0][0], &d_q_step_1_four[2*M], d_exp_dw_1,
            &d_q_step_2_two[0][M], &d_q_step_1_four[3*M], d_exp_dw_2, 1.0/((double)M), M);

        // step 1/4: Execute a Forward FFT
        hipfftExecD2Z(plan_for_two[0], d_q_step_2_two[0], d_qk_in_2_two[0]);

        // step 1/4: Multiply exp(-k^2 ds/12) in fourier space
        complex_real_multi_bond_two<<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(
            &d_qk_in_2_two[0][0],         d_boltz_bond_half_1, 
            &d_qk_in_2_two[0][M_COMPLEX], d_boltz_bond_half_2, M_COMPLEX);

        // step 1/4: Execute a backward FFT
        hipfftExecZ2D(plan_bak_two[0], d_qk_in_2_two[0], d_q_step_2_two[0]);

        // step 1/4: Evaluate exp(-w*ds/4) in real space
        real_multi_exp_dw_two<<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(
            &d_q_step_2_two[0][0], &d_q_step_2_two[0][0], d_exp_dw_half_1,
            &d_q_step_2_two[0][M], &d_q_step_2_two[0][M], d_exp_dw_half_2, 1.0/((double)M), M);

        // compute linear combination with 4/3 and -1/3 ratio
        lin_comb<<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(d_q_out_1, 4.0/3.0, &d_q_step_2_two[0][0], -1.0/3.0, &d_q_step_1_two[0][0], M);
        lin_comb<<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(d_q_out_2, 4.0/3.0, &d_q_step_2_two[0][M], -1.0/3.0, &d_q_step_1_two[0][M], M);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoContinuous::calculate_phi_one_block(
    double *d_phi, double **d_q_1, double **d_q_2, const int N, const int N_OFFSET, const int N_ORIGINAL)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        std::vector<double> simpson_rule_coeff = SimpsonRule::get_coeff(N);

        // Compute segment concentration
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_phi, d_q_1[N_ORIGINAL-N_OFFSET], d_q_2[0], simpson_rule_coeff[0], M);
        for(int n=1; n<=N; n++)
        {
            add_multi_real<<<N_BLOCKS, N_THREADS>>>(d_phi, d_q_1[N_ORIGINAL-N_OFFSET-n], d_q_2[n], simpson_rule_coeff[n], M);
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
double CudaPseudoContinuous::get_total_partition(int polymer)
{
    try
    {
        return single_partitions[polymer];
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoContinuous::get_monomer_concentration(std::string monomer_type, double *phi)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();

        // initialize to zero
        gpu_error_check(hipMemset(d_phi, 0, sizeof(double)*M));

        // for each block
        for(const auto& d_block: d_block_phi)
        {
            const auto& key = d_block.first;
            std::string dep_v = std::get<1>(key);
            int n_segment_allocated = mx->get_essential_block(key).n_segment_allocated;
            if (Mixture::get_monomer_type_from_key(dep_v) == monomer_type && n_segment_allocated != 0)
                lin_comb<<<N_BLOCKS, N_THREADS>>>(d_phi, 1.0, d_phi, 1.0, d_block.second, M);
        }
        gpu_error_check(hipMemcpy(phi, d_phi, sizeof(double)*M, hipMemcpyDeviceToHost));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoContinuous::get_polymer_concentration(int p, double *phi)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int P = mx->get_n_polymers();

        if (p < 0 || p > P-1)
            throw_with_line_number("Index (" + std::to_string(p) + ") must be in range [0, " + std::to_string(P-1) + "]");

        if (mx->is_using_superposition())
            throw_with_line_number("Disable 'superposition' option to invoke 'get_polymer_concentration'.");

        // initialize to zero
        gpu_error_check(hipMemset(d_phi, 0, sizeof(double)*M));

        PolymerChain& pc = mx->get_polymer(p);
        std::vector<PolymerChainBlock>& blocks = pc.get_blocks();

        for(size_t b=0; b<blocks.size(); b++)
        {
            std::string dep_v = pc.get_propagator_key(blocks[b].v, blocks[b].u);
            std::string dep_u = pc.get_propagator_key(blocks[b].u, blocks[b].v);
            if (dep_v < dep_u)
                dep_v.swap(dep_u);

            lin_comb<<<N_BLOCKS, N_THREADS>>>(d_phi, 0.0, d_phi, 1.0, d_block_phi[std::make_tuple(p, dep_v, dep_u)], M);
            gpu_error_check(hipMemcpy(&phi[b*M], d_phi, sizeof(double)*M, hipMemcpyDeviceToHost));
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
std::vector<double> CudaPseudoContinuous::compute_stress()
{
    // This method should be invoked after invoking compute_statistics().

    // To calculate stress, we multiply weighted fourier basis to q(k)*q^dagger(-k).
    // We only need the real part of stress calculation.

    try{
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();
        const int N_GPUS = CudaCommon::get_instance().get_n_gpus();

        const int DIM  = cb->get_dim();
        const int M    = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        auto bond_lengths = mx->get_bond_lengths();
        std::vector<double> stress(DIM);
        std::map<std::tuple<int, std::string, std::string>, std::array<double,3>> block_dq_dl[MAX_GPUS];
        double stress_sum_out[MAX_GPUS][3];

        // compute stress for each block
        for(const auto& block: d_block_phi)
        {
            const auto& key = block.first;
            int p                = std::get<0>(key);
            std::string dep_v    = std::get<1>(key);
            std::string dep_u    = std::get<2>(key);

            const int N           = mx->get_essential_block(key).n_segment_allocated;
            const int N_OFFSET    = mx->get_essential_block(key).n_segment_offset;
            const int N_ORIGINAL  = mx->get_essential_block(key).n_segment_original;
            std::string monomer_type = mx->get_essential_block(key).monomer_type;

            // if there is no segment
            if(N == 0)
                continue;

            // contains no '['
            int n_repeated;
            if (dep_u.find('[') == std::string::npos)
                n_repeated = mx->get_essential_block(key).v_u.size();
            else
                n_repeated = 1;

            std::vector<double> s_coeff = SimpsonRule::get_coeff(N);
            double bond_length_sq = bond_lengths[monomer_type]*bond_lengths[monomer_type];
            double** d_q_1 = d_propagator[dep_v];    // dependency v
            double** d_q_2 = d_propagator[dep_u];    // dependency u

            std::array<double,3> _block_dq_dl[MAX_GPUS];
            for(int gpu=0; gpu<N_GPUS; gpu++)
                for(int d=0; d<3; d++)
                    _block_dq_dl[gpu][d] = 0.0;

            int prev, next;
            prev = 0;
            next = 1;

            // copy memory from device to device
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                // index
                int idx = gpu;
                gpu_error_check(hipSetDevice(gpu));
                if (idx <= N)
                {
                    gpu_error_check(hipMemcpy(&d_stress_q[gpu][prev][0], d_q_1[N_ORIGINAL-N_OFFSET-idx],
                            sizeof(double)*M,hipMemcpyDeviceToDevice));
                    gpu_error_check(hipMemcpy(&d_stress_q[gpu][prev][M], d_q_2[idx],
                            sizeof(double)*M,hipMemcpyDeviceToDevice));
                }
            }

            // compute
            for(int n=0; n<=N; n+=N_GPUS)
            {
                // STREAM 1: copy memory from device to device
                for(int gpu=0; gpu<N_GPUS; gpu++)
                {
                    // index
                    const int idx = n + gpu;
                    const int idx_next = idx + N_GPUS;

                    gpu_error_check(hipSetDevice(gpu));
                    if (idx_next <= N)
                    {
                        gpu_error_check(hipMemcpyAsync(&d_stress_q[gpu][next][0], d_q_1[N_ORIGINAL-N_OFFSET-idx_next],
                                sizeof(double)*M,hipMemcpyDeviceToDevice, streams[gpu][1]));
                        gpu_error_check(hipMemcpyAsync(&d_stress_q[gpu][next][M], d_q_2[idx_next],
                                sizeof(double)*M,hipMemcpyDeviceToDevice, streams[gpu][1]));
                    }
                }
                // STREAM 0: execute kernels
                // execute a Forward FFT
                for(int gpu=0; gpu<N_GPUS; gpu++)
                {
                    const int idx = n + gpu;
                    gpu_error_check(hipSetDevice(gpu));
                    if (idx <= N)
                        hipfftExecD2Z(plan_for_two[gpu], d_stress_q[gpu][prev], d_qk_in_1_two[gpu]);
                }
                // multiply two propagators in the fourier spaces
                for(int gpu=0; gpu<N_GPUS; gpu++)
                {
                    const int idx = n + gpu;
                    gpu_error_check(hipSetDevice(gpu));
                    if (idx <= N)
                    {
                        multi_complex_conjugate<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_q_multi[gpu], &d_qk_in_1_two[gpu][0], &d_qk_in_1_two[gpu][M_COMPLEX], M_COMPLEX);
                        if ( DIM == 3 )
                        {
                            // x direction
                            multi_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_stress_sum[gpu], d_q_multi[gpu], d_fourier_basis_x[gpu], bond_length_sq, M_COMPLEX);
                            hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
                            gpu_error_check(hipMemcpyAsync(&stress_sum_out[gpu][0],d_stress_sum_out[gpu],sizeof(double),hipMemcpyDeviceToHost, streams[gpu][0]));

                            // y direction
                            multi_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_stress_sum[gpu], d_q_multi[gpu], d_fourier_basis_y[gpu], bond_length_sq, M_COMPLEX);
                            hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
                            gpu_error_check(hipMemcpyAsync(&stress_sum_out[gpu][1],d_stress_sum_out[gpu],sizeof(double),hipMemcpyDeviceToHost, streams[gpu][0]));

                            // z direction
                            multi_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_stress_sum[gpu], d_q_multi[gpu], d_fourier_basis_z[gpu], bond_length_sq, M_COMPLEX);
                            hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
                            gpu_error_check(hipMemcpyAsync(&stress_sum_out[gpu][2],d_stress_sum_out[gpu],sizeof(double),hipMemcpyDeviceToHost, streams[gpu][0]));
                        }
                        if ( DIM == 2 )
                        {
                            // y direction
                            multi_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_stress_sum[gpu], d_q_multi[gpu], d_fourier_basis_y[gpu], bond_length_sq, M_COMPLEX);
                            hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
                            gpu_error_check(hipMemcpyAsync(&stress_sum_out[gpu][0],d_stress_sum_out[gpu],sizeof(double),hipMemcpyDeviceToHost, streams[gpu][0]));

                            // z direction
                            multi_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_stress_sum[gpu], d_q_multi[gpu], d_fourier_basis_z[gpu], bond_length_sq, M_COMPLEX);
                            hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
                            gpu_error_check(hipMemcpyAsync(&stress_sum_out[gpu][1],d_stress_sum_out[gpu],sizeof(double),hipMemcpyDeviceToHost, streams[gpu][0]));
                        }
                        if ( DIM == 1 )
                        {
                            // z direction
                            multi_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_stress_sum[gpu], d_q_multi[gpu], d_fourier_basis_z[gpu], bond_length_sq, M_COMPLEX);
                            hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
                            gpu_error_check(hipMemcpyAsync(&stress_sum_out[gpu][0],d_stress_sum_out[gpu],sizeof(double),hipMemcpyDeviceToHost, streams[gpu][0]));
                        }
                        // synchronize streams and add results
                        gpu_error_check(hipStreamSynchronize(streams[gpu][0]));
                        for(int d=0; d<DIM; d++)
                            _block_dq_dl[gpu][d] += s_coeff[idx]*stress_sum_out[gpu][d]*n_repeated;
                    }
                }
                // synchronize all GPUs
                for(int gpu=0; gpu<N_GPUS; gpu++)
                {
                    gpu_error_check(hipSetDevice(gpu));
                    gpu_error_check(hipDeviceSynchronize());
                }
                std::swap(prev, next);
            }
            // copy stress data
            for(int gpu=0; gpu<N_GPUS; gpu++)
                block_dq_dl[gpu][key] = _block_dq_dl[gpu];
        }

        // compute total stress
        for(int d=0; d<DIM; d++)
            stress[d] = 0.0;
        for(const auto& block: d_block_phi)
        {
            const auto& key = block.first;
            int p             = std::get<0>(key);
            std::string dep_v = std::get<1>(key);
            std::string dep_u = std::get<2>(key);
            PolymerChain& pc  = mx->get_polymer(p);

            for(int gpu=0; gpu<N_GPUS; gpu++)
                for(int d=0; d<DIM; d++)
                    stress[d] += block_dq_dl[gpu][key][d]*pc.get_volume_fraction()/pc.get_alpha()/single_partitions[p];
        }
        for(int d=0; d<DIM; d++)
            stress[d] /= -3.0*cb->get_lx(d)*M*M/mx->get_ds();
            
        return stress;
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoContinuous::get_chain_propagator(double *q_out, int polymer, int v, int u, int n)
{
    // This method should be invoked after invoking compute_statistics()

    // Get chain propagator for a selected polymer, block and direction.
    // This is made for debugging and testing.
    try
    {
        const int M = cb->get_n_grid();
        PolymerChain& pc = mx->get_polymer(polymer);
        std::string dep = pc.get_propagator_key(v,u);

        if (mx->get_essential_propagator_codes().find(dep) == mx->get_essential_propagator_codes().end())
            throw_with_line_number("Could not find the propagator code '" + dep + "'. Disable 'superposition' option to obtain propagators.");

        const int N = mx->get_essential_propagator_codes()[dep].max_n_segment;
        if (n < 0 || n > N)
            throw_with_line_number("n (" + std::to_string(n) + ") must be in range [0, " + std::to_string(N) + "]");

        gpu_error_check(hipMemcpy(q_out, d_propagator[dep][n], sizeof(double)*M,hipMemcpyDeviceToHost));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}