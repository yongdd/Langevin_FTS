#include "hip/hip_runtime.h"
#define THRUST_IGNORE_DEPRECATED_CPP_DIALECT
#define CUB_IGNORE_DEPRECATED_CPP_DIALECT

#include <complex>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include "CudaPseudoContinuous.h"
#include "CudaComputationBox.h"
#include "SimpsonQuadrature.h"

CudaPseudoContinuous::CudaPseudoContinuous(
    ComputationBox *cb,
    Mixture *mx)
    : Pseudo(cb, mx)
{
    try{
        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        // allocate memory for partition functions
        if( mx->get_unique_branches().size() == 0)
            throw_with_line_number("There is no unique branch. Add polymers first.");
        for(const auto& item: mx->get_unique_branches())
        {
            std::string dep = item.first;
            int max_n_segment = item.second.max_n_segment;
            d_unique_partition[dep] = nullptr;
            gpu_error_check(hipMalloc((void**)&d_unique_partition[dep], sizeof(double)*M*(max_n_segment+1)));
        }

        // allocate memory for concentrations
        if( mx->get_unique_blocks().size() == 0)
            throw_with_line_number("There is no unique block. Add polymers first.");
        for(const auto& item: mx->get_unique_blocks())
        {
            d_unique_phi[item.first] = nullptr;
            gpu_error_check(hipMalloc((void**)&d_unique_phi[item.first], sizeof(double)*M));
        }

        // create boltz_bond, boltz_bond_half, exp_dw, and exp_dw_half
        for(const auto& item: mx->get_bond_lengths()){
            std::string species = item.first;
            d_boltz_bond     [species] = nullptr;
            d_boltz_bond_half[species] = nullptr;
            d_exp_dw         [species] = nullptr;
            d_exp_dw_half    [species] = nullptr;

            gpu_error_check(hipMalloc((void**)&d_exp_dw         [species], sizeof(double)*M));
            gpu_error_check(hipMalloc((void**)&d_exp_dw_half    [species], sizeof(double)*M));
            gpu_error_check(hipMalloc((void**)&d_boltz_bond     [species], sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_boltz_bond_half[species], sizeof(double)*M_COMPLEX));
        }

        // total partition functions for each polymer
        single_partitions = new double[mx->get_n_polymers()];

        // create FFT plan
        const int BATCH{1};
        const int NRANK{cb->get_dim()};
        int n_grid[NRANK];

        if(cb->get_dim() == 3)
        {
            n_grid[0] = cb->get_nx(0);
            n_grid[1] = cb->get_nx(1);
            n_grid[2] = cb->get_nx(2);
        }
        else if(cb->get_dim() == 2)
        {
            n_grid[0] = cb->get_nx(1);
            n_grid[1] = cb->get_nx(2);
        }
        else if(cb->get_dim() == 1)
        {
            n_grid[0] = cb->get_nx(2);
        }
        hipfftPlanMany(&plan_for, NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z,BATCH);
        hipfftPlanMany(&plan_bak, NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D,BATCH);

        // allocate memory for get_concentration
        gpu_error_check(hipMalloc((void**)&d_phi, sizeof(double)*M));

        // allocate memory for pseudo-spectral: one_step()
        gpu_error_check(hipMalloc((void**)&d_q_step1, sizeof(double)*M));
        gpu_error_check(hipMalloc((void**)&d_q_step2, sizeof(double)*M));
        gpu_error_check(hipMalloc((void**)&d_qk_in,  sizeof(ftsComplex)*M_COMPLEX));
        
        // allocate memory for stress calculation: compute_stress()
        gpu_error_check(hipMalloc((void**)&d_fourier_basis_x, sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_fourier_basis_y, sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_fourier_basis_z, sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_qk_1,        sizeof(ftsComplex)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_qk_2,        sizeof(ftsComplex)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_q_multi,         sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_stress_sum,      sizeof(double)*M_COMPLEX));

        update();
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
CudaPseudoContinuous::~CudaPseudoContinuous()
{
    hipfftDestroy(plan_for);
    hipfftDestroy(plan_bak);

    delete[] single_partitions;

    for(const auto& item: d_boltz_bond)
        hipFree(item.second);
    for(const auto& item: d_boltz_bond_half)
        hipFree(item.second);
    for(const auto& item: d_exp_dw)
        hipFree(item.second);
    for(const auto& item: d_exp_dw_half)
        hipFree(item.second);
    for(const auto& item: d_unique_partition)
        hipFree(item.second);
    for(const auto& item: d_unique_phi)
        hipFree(item.second);

    // for get_concentration
    hipFree(d_phi);

    // for pseudo-spectral: one_step()
    hipFree(d_q_step1);
    hipFree(d_q_step2);
    hipFree(d_qk_in);

    // for stress calculation: compute_stress()
    hipFree(d_fourier_basis_x);
    hipFree(d_fourier_basis_y);
    hipFree(d_fourier_basis_z);
    hipFree(d_qk_1);
    hipFree(d_qk_2);
    hipFree(d_q_multi);
    hipFree(d_stress_sum);
}

void CudaPseudoContinuous::update()
{
    try{
        // for pseudo-spectral: one_step()
        const int M_COMPLEX = this->n_complex_grid;
        double boltz_bond[M_COMPLEX], boltz_bond_half[M_COMPLEX];

        for(const auto& item: mx->get_bond_lengths())
        {
            std::string species = item.first;
            double bond_length_sq = item.second*item.second;
            get_boltz_bond(boltz_bond     , bond_length_sq,   cb->get_nx(), cb->get_dx(), mx->get_ds());
            get_boltz_bond(boltz_bond_half, bond_length_sq/2, cb->get_nx(), cb->get_dx(), mx->get_ds());
        
            gpu_error_check(hipMemcpy(d_boltz_bond[species],      boltz_bond,      sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_boltz_bond_half[species], boltz_bond_half, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
        }

        // for stress calculation: compute_stress()
        double fourier_basis_x[M_COMPLEX];
        double fourier_basis_y[M_COMPLEX];
        double fourier_basis_z[M_COMPLEX];
        get_weighted_fourier_basis(fourier_basis_x, fourier_basis_y, fourier_basis_z, cb->get_nx(), cb->get_dx());
        gpu_error_check(hipMemcpy(d_fourier_basis_x, fourier_basis_x, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
        gpu_error_check(hipMemcpy(d_fourier_basis_y, fourier_basis_y, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
        gpu_error_check(hipMemcpy(d_fourier_basis_z, fourier_basis_z, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
    }
    catch(std::exception& exc)
    {
        throw_with_line_number(exc.what());
    }
}
void CudaPseudoContinuous::compute_statistics(
    std::map<std::string, double*> q_init,
    std::map<std::string, double*> w_block)
{
    try{
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const double ds = mx->get_ds();

        for(const auto& item: mx->get_unique_branches())
        {
            if( w_block.count(item.second.species) == 0)
                throw_with_line_number("\"" + item.second.species + "\" species is not in w_block.");
        }

        if( q_init.size() > 0)
            throw_with_line_number("Currently, \'q_init\' is not supported.");

        // exp_dw and exp_dw_half
        double exp_dw[M];
        double exp_dw_half[M];
        for(const auto& item: w_block)
        {
            std::string species = item.first;
            double *w = item.second;
            for(int i=0; i<M; i++)
            { 
                exp_dw     [i] = exp(-w[i]*ds*0.5);
                exp_dw_half[i] = exp(-w[i]*ds*0.25);
            }
            gpu_error_check(hipMemcpy(d_exp_dw     [species], exp_dw,      sizeof(double)*M,hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_exp_dw_half[species], exp_dw_half, sizeof(double)*M,hipMemcpyHostToDevice));
        }

        double q_uniform[M];
        for(int i=0; i<M; i++)
            q_uniform[i] = 1.0;
        for(const auto& item: mx->get_unique_branches())
        {
            auto& key = item.first;
            // calculate one block end
            if (item.second.deps.size() > 0) // if it is not leaf node
            {
                gpu_error_check(hipMemcpy(d_unique_partition[key], q_uniform,
                    sizeof(double)*M, hipMemcpyHostToDevice));

                for(int p=0; p<item.second.deps.size(); p++)
                {
                    std::string sub_dep = item.second.deps[p].first;
                    int sub_n_segment   = item.second.deps[p].second;
                    multi_real<<<N_BLOCKS, N_THREADS>>>(
                        d_unique_partition[key], d_unique_partition[key],
                        &d_unique_partition[sub_dep][sub_n_segment*M], 1.0, M);
                }
            }
            else // if it is leaf node
            {
                gpu_error_check(hipMemcpy(d_unique_partition[key], q_uniform,
                    sizeof(double)*M, hipMemcpyHostToDevice)); //* q_init
            }

            // apply the propagator successively
            for(int n=1; n<=item.second.max_n_segment; n++)
            {
                one_step(&d_unique_partition[key][(n-1)*M],
                         &d_unique_partition[key][n*M],
                         d_boltz_bond[item.second.species],
                         d_boltz_bond_half[item.second.species],
                         d_exp_dw[item.second.species],
                         d_exp_dw_half[item.second.species]);
            }
        }

        // calculate segment concentrations
        for(const auto& item: mx->get_unique_blocks())
        {
            auto& key = item.first;
            calculate_phi_one_type(
                d_unique_phi[key],                     // phi
                d_unique_partition[std::get<0>(key)],  // dependency v
                d_unique_partition[std::get<1>(key)],  // dependency u
                std::get<2>(key));                      // n_segment
        }

        // for each distinct polymers 
        for(int p=0; p<mx->get_n_polymers(); p++)
        {
            PolymerChain& pc = mx->get_polymer(p);
            std::vector<PolymerChainBlock>& blocks = pc.get_blocks();

            // calculate the single chain partition function at block 0
            std::string dep_v = pc.get_dep(blocks[0].v, blocks[0].u);
            std::string dep_u = pc.get_dep(blocks[0].u, blocks[0].v);
            int n_segment = blocks[0].n_segment;
            single_partitions[p] = ((CudaComputationBox *)cb)->inner_product_gpu(
                &d_unique_partition[dep_v][n_segment*M],  // q
                &d_unique_partition[dep_u][0]);           // q^dagger
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}

// Advance partial partition function using Richardson extrapolation.
void CudaPseudoContinuous::one_step(
    double *d_q_in, double *d_q_out,
    double *d_boltz_bond, double *d_boltz_bond_half,
    double *d_exp_dw, double *d_exp_dw_half)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        //-------------- step 1 ----------
        // Evaluate e^(-w*ds/2) in real space
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_step1, d_q_in, d_exp_dw, 1.0, M);

        // Execute a Forw_ard FFT
        hipfftExecD2Z(plan_for, d_q_step1, d_qk_in);

        // Multiply e^(-k^2 ds/6) in fourier space
        multi_complex_real<<<N_BLOCKS, N_THREADS>>>(d_qk_in, d_boltz_bond, M_COMPLEX);

        // Execute a backw_ard FFT
        hipfftExecZ2D(plan_bak, d_qk_in, d_q_step1);

        // Evaluate e^(-w*ds/2) in real space
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_step1, d_q_step1, d_exp_dw, 1.0/((double)M), M);

        //-------------- step 2 ----------
        // Evaluate e^(-w*ds/4) in real space
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_step2, d_q_in, d_exp_dw_half, 1.0, M);

        // Execute a Forw_ard FFT
        hipfftExecD2Z(plan_for, d_q_step2, d_qk_in);

        // Multiply e^(-k^2 ds/12) in fourier space
        multi_complex_real<<<N_BLOCKS, N_THREADS>>>(d_qk_in, d_boltz_bond_half, M_COMPLEX);

        // Execute a backw_ard FFT
        hipfftExecZ2D(plan_bak, d_qk_in, d_q_step2);

        // Evaluate e^(-w*ds/2) in real space
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_step2, d_q_step2, d_exp_dw, 1.0/((double)M), M);
        // Execute a Forw_ard FFT
        hipfftExecD2Z(plan_for, d_q_step2, d_qk_in);

        // Multiply e^(-k^2 ds/12) in fourier space
        multi_complex_real<<<N_BLOCKS, N_THREADS>>>(d_qk_in, d_boltz_bond_half, M_COMPLEX);

        // Execute a backw_ard FFT
        hipfftExecZ2D(plan_bak, d_qk_in, d_q_step2);

        // Evaluate e^(-w*ds/4) in real space.
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_step2, d_q_step2, d_exp_dw_half, 1.0/((double)M), M);
        //-------------- step 3 ----------
        lin_comb<<<N_BLOCKS, N_THREADS>>>(d_q_out, 4.0/3.0, d_q_step2, -1.0/3.0, d_q_step1, M);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoContinuous::calculate_phi_one_type(
    double *d_phi, double *d_q_1, double *d_q_2, const int N)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        std::vector<double> simpson_rule_coeff = SimpsonQuadrature::get_coeff(N);

        // Compute segment concentration
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_phi, &d_q_1[0], &d_q_2[N*M], simpson_rule_coeff[0], M);
        for(int n=1; n<=N; n++)
        {
            add_multi_real<<<N_BLOCKS, N_THREADS>>>(d_phi, &d_q_1[n*M], &d_q_2[(N-n)*M], simpson_rule_coeff[n], M);
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
double CudaPseudoContinuous::get_total_partition(int polymer)
{
    try
    {
        return single_partitions[polymer];
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoContinuous::get_species_concentration(std::string species, double *phi)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        // initialize to zero
        lin_comb<<<N_BLOCKS, N_THREADS>>>(d_phi, 0.0, d_phi, 0.0, d_phi, M);

        // for each distinct polymers 
        for(int p=0; p<mx->get_n_polymers(); p++)
        {
            PolymerChain& pc = mx->get_polymer(p);
            std::vector<PolymerChainBlock>& blocks = pc.get_blocks();
            for(int b=0; b<blocks.size(); b++)
            {
                if (blocks[b].species == species)
                {
                    std::string dep_v = pc.get_dep(blocks[b].v, blocks[b].u);
                    std::string dep_u = pc.get_dep(blocks[b].u, blocks[b].v);
                    if (dep_v > dep_u)
                        dep_v.swap(dep_u);
                        
                    // normalize the concentration
                    double norm = cb->get_volume()*mx->get_ds()*pc.get_volume_fraction()/pc.get_alpha()/single_partitions[p];
                    lin_comb<<<N_BLOCKS, N_THREADS>>>(d_phi, 1.0, d_phi, norm, d_unique_phi[std::make_tuple(dep_v, dep_u, blocks[b].n_segment)], M);
                }
            }
        }
        gpu_error_check(hipMemcpy(phi, d_phi, sizeof(double)*M, hipMemcpyDeviceToHost));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoContinuous::get_polymer_concentration(int polymer, double *phi)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int P = mx->get_n_polymers();

        if (polymer < 0 || polymer > P-1)
            throw_with_line_number("Index (" + std::to_string(polymer) + ") must be in range [0, " + std::to_string(P-1) + "]");

        PolymerChain& pc = mx->get_polymer(polymer);
        std::vector<PolymerChainBlock>& blocks = pc.get_blocks();

        for(int b=0; b<blocks.size(); b++)
        {
            std::string dep_v = pc.get_dep(blocks[b].v, blocks[b].u);
            std::string dep_u = pc.get_dep(blocks[b].u, blocks[b].v);
            if (dep_v > dep_u)
                dep_v.swap(dep_u);

            // copy normalized concentration
            double norm = cb->get_volume()*mx->get_ds()*pc.get_volume_fraction()/pc.get_alpha()/single_partitions[polymer];
            lin_comb<<<N_BLOCKS, N_THREADS>>>(d_phi, 0.0, d_phi, norm, d_unique_phi[std::make_tuple(dep_v, dep_u, blocks[b].n_segment)], M);
            gpu_error_check(hipMemcpy(&phi[b*M], d_phi, sizeof(double)*M, hipMemcpyDeviceToHost));
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
std::array<double,3> CudaPseudoContinuous::compute_stress()
{
    // This method should be invoked after invoking compute_statistics().

    // To calculate stress, we multiply weighted fourier basis to q(k)*q^dagger(-k).
    // We only need the real part of stress calculation.

    try{
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int DIM  = cb->get_dim();
        const int M    = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        std::map<std::string, double>& bond_lengths = mx->get_bond_lengths();
        std::array<double,3> stress;
        std::map<std::tuple<std::string, std::string, int>, std::array<double,3>> unique_dq_dl;
        thrust::device_ptr<double> temp_gpu_ptr(d_stress_sum);

        // compute stress for Unique key pairs
        for(const auto& item: mx->get_unique_blocks())
        {
            auto& key = item.first;
            std::string dep_v = std::get<0>(key);
            std::string dep_u = std::get<1>(key);
            const int N       = std::get<2>(key);
            std::string species = item.second.species;

            std::vector<double> s_coeff = SimpsonQuadrature::get_coeff(N);
            double bond_length_sq = bond_lengths[species]*bond_lengths[species];
            double* d_q_1 = d_unique_partition[dep_v];    // dependency v
            double* d_q_2 = d_unique_partition[dep_u];    // dependency u

            // reset
            for(int d=0; d<3; d++)
                unique_dq_dl[key][d] = 0.0;

            // compute
            for(int n=0; n<=N; n++)
            {
                hipfftExecD2Z(plan_for, &d_q_1[n*M],     d_qk_1);
                hipfftExecD2Z(plan_for, &d_q_2[(N-n)*M], d_qk_2);
                multi_complex_conjugate<<<N_BLOCKS, N_THREADS>>>(d_q_multi, d_qk_1, d_qk_2, M_COMPLEX);
                if ( DIM >= 3 )
                {
                    multi_real<<<N_BLOCKS, N_THREADS>>>(d_stress_sum, d_q_multi, d_fourier_basis_x, bond_length_sq, M_COMPLEX);
                    unique_dq_dl[key][0] += s_coeff[n]*thrust::reduce(temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX);
                }
                if ( DIM >= 2 )
                {
                    multi_real<<<N_BLOCKS, N_THREADS>>>(d_stress_sum, d_q_multi, d_fourier_basis_y, bond_length_sq, M_COMPLEX);
                    unique_dq_dl[key][1] += s_coeff[n]*thrust::reduce(temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX);
                }
                if ( DIM >= 1 )
                {
                    multi_real<<<N_BLOCKS, N_THREADS>>>(d_stress_sum, d_q_multi, d_fourier_basis_z, bond_length_sq, M_COMPLEX);
                    unique_dq_dl[key][2] += s_coeff[n]*thrust::reduce(temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX);
                }
            }
        }

        // compute total stress
        for(int d=0; d<3; d++)
            stress[d] = 0.0;
        for(int p=0; p < mx->get_n_polymers(); p++)
        {
            PolymerChain& pc = mx->get_polymer(p);
            std::vector<PolymerChainBlock>& blocks = pc.get_blocks();
            for(int b=0; b<blocks.size(); b++)
            {
                std::string dep_v = pc.get_dep(blocks[b].v, blocks[b].u);
                std::string dep_u = pc.get_dep(blocks[b].u, blocks[b].v);
                if (dep_v > dep_u)
                    dep_v.swap(dep_u);
                for(int d=0; d<3; d++)
                    stress[d] += unique_dq_dl[std::make_tuple(dep_v, dep_u, blocks[b].n_segment)][d]*pc.get_volume_fraction()/pc.get_alpha()/single_partitions[p];
            }
        }
        for(int d=0; d<3; d++)
            stress[d] /= -3.0*cb->get_lx(d)*M*M/mx->get_ds()/cb->get_volume();
            
        return stress;
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoContinuous::get_partial_partition(double *q_out, int polymer, int v, int u, int n)
{
    // This method should be invoked after invoking compute_statistics()

    // Get partial partition functions
    // This is made for debugging and testing
    try
    {
        const int M = cb->get_n_grid();
        PolymerChain& pc = mx->get_polymer(polymer);
        std::string dep = pc.get_dep(v,u);
        const int N = mx->get_unique_branches()[dep].max_n_segment;
        if (n < 0 || n > N)
            throw_with_line_number("n (" + std::to_string(n) + ") must be in range [0, " + std::to_string(N) + "]");

        double* partition = d_unique_partition[dep];
        gpu_error_check(hipMemcpy(q_out, &partition[n*M], sizeof(double)*M,hipMemcpyDeviceToHost));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}