#include "hip/hip_runtime.h"
#include <complex>
#include <thrust/reduce.h>
#include <iostream>
#include "CudaPseudoDiscrete.h"
#include "CudaComputationBox.h"

CudaPseudoDiscrete::CudaPseudoDiscrete(
    ComputationBox *cb,
    Molecules *molecules)
    : Pseudo(cb, molecules)
{
    try
    {
        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;
        const int N_GPUS = CudaCommon::get_instance().get_n_gpus();

        // Allocate memory for propagators
        gpu_error_check(hipSetDevice(0));
        if( molecules->get_essential_propagator_codes().size() == 0)
            throw_with_line_number("There is no propagator code. Add polymers first.");
        for(const auto& item: molecules->get_essential_propagator_codes())
        {
             // There are N segments

             // Example (N==5)
             // O--O--O--O--O
             // 0  1  2  3  4

             // Legend)
             // -- : full bond
             // O  : full segment

            std::string key = item.first;
            int max_n_segment = item.second.max_n_segment;

            propagator_size[key] = max_n_segment;
            d_propagator[key] = new double*[max_n_segment];
            for(int i=0; i<propagator_size[key]; i++)
                gpu_error_check(hipMalloc((void**)&d_propagator[key][i], sizeof(double)*M));

            #ifndef NDEBUG
            propagator_finished[key] = new bool[max_n_segment];
            for(int i=0; i<max_n_segment;i++)
                propagator_finished[key][i] = false;
            #endif
        }

        // Allocate memory for propagator_junction, which contain propagator at junction of discrete chain
        for(const auto& item: molecules->get_essential_propagator_codes())
        {
            std::string key = item.first;
            d_propagator_junction[key] = nullptr;
            gpu_error_check(hipMalloc((void**)&d_propagator_junction[key], sizeof(double)*M));
        }

        // Allocate memory for concentrations
        if( molecules->get_essential_blocks().size() == 0)
            throw_with_line_number("There is no block. Add polymers first.");
        for(const auto& item: molecules->get_essential_blocks())
        {
            d_block_phi[item.first] = nullptr;
            gpu_error_check(hipMalloc((void**)&d_block_phi[item.first], sizeof(double)*M));
        }

        // Create boltz_bond, boltz_bond_half, and exp_dw
        for(const auto& item: molecules->get_bond_lengths())
        {
            std::string monomer_type = item.first;
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                d_boltz_bond     [gpu][monomer_type] = nullptr;
                d_boltz_bond_half[gpu][monomer_type] = nullptr;
                d_exp_dw         [gpu][monomer_type] = nullptr;

                gpu_error_check(hipMalloc((void**)&d_exp_dw         [gpu][monomer_type], sizeof(double)*M));
                gpu_error_check(hipMalloc((void**)&d_boltz_bond     [gpu][monomer_type], sizeof(double)*M_COMPLEX));
                gpu_error_check(hipMalloc((void**)&d_boltz_bond_half[gpu][monomer_type], sizeof(double)*M_COMPLEX));
            }
        }

        // Total partition functions for each polymer
        single_partitions = new double[molecules->get_n_polymer_types()];

        // Remember one segment for each polymer chain to compute total partition function
        int current_p = 0;
        for(const auto& d_block: d_block_phi)
        {
            const auto& key = d_block.first;
            int p                = std::get<0>(key);
            std::string dep_v    = std::get<1>(key);
            std::string dep_u    = std::get<2>(key);

            // Skip if already found one segment
            if (p != current_p)
                continue;

            int n_superposed;
            int n_segment_offset    = molecules->get_essential_block(key).n_segment_offset;
            int n_segment_original  = molecules->get_essential_block(key).n_segment_original;
            std::string monomer_type = molecules->get_essential_block(key).monomer_type;

            // Contains no '['
            if (dep_u.find('[') == std::string::npos)
                n_superposed = 1;
            else
                n_superposed = molecules->get_essential_block(key).v_u.size();

            single_partition_segment.push_back(std::make_tuple(
                p,
                d_propagator[dep_v][n_segment_original-n_segment_offset-1],  // q
                d_propagator[dep_u][0],                                      // Q_dagger
                monomer_type,       
                n_superposed                   // How many propagators are aggregated
                ));
            current_p++;
        }

       // Find propagators and bond length for each segment to prepare stress computation
        for(const auto& block: d_block_phi)
        {
            const auto& key = block.first;
            int p                = std::get<0>(key);
            std::string dep_v    = std::get<1>(key);
            std::string dep_u    = std::get<2>(key);

            const int N           = molecules->get_essential_block(key).n_segment_allocated;
            const int N_OFFSET    = molecules->get_essential_block(key).n_segment_offset;
            const int N_ORIGINAL  = molecules->get_essential_block(key).n_segment_original;

            double **d_q_1 = d_propagator[dep_v];    // dependency v
            double **d_q_2 = d_propagator[dep_u];    // dependency u

            auto& _block_stress_info_key = block_stress_info[key];

            // Find propagators and bond length
            for(int n=0; n<=N; n++)
            {
                double *d_propagator_v = nullptr;
                double *d_propagator_u = nullptr;
                bool is_half_bond_length = false;

                // At v
                if (n + N_OFFSET == N_ORIGINAL)
                {
                    if (molecules->get_essential_propagator_code(dep_v).deps.size() == 0) // if v is leaf node, skip
                    {
                        _block_stress_info_key.push_back(std::make_tuple(d_propagator_v, d_propagator_u, is_half_bond_length));
                        continue;
                    }
                    
                    d_propagator_v = d_propagator_junction[dep_v];
                    d_propagator_u = d_q_2[N-1];
                    is_half_bond_length = true;
                }
                // At u
                else if (n + N_OFFSET == 0){
                    if (molecules->get_essential_propagator_code(dep_u).deps.size() == 0) // if u is leaf node, skip
                    {
                        _block_stress_info_key.push_back(std::make_tuple(d_propagator_v, d_propagator_u, is_half_bond_length));
                        continue;
                    }

                    d_propagator_v = d_q_1[N_ORIGINAL-1];
                    d_propagator_u = d_propagator_junction[dep_u];
                    is_half_bond_length = true;
                }
                // At superposition junction
                else if (n == 0)
                {
                    _block_stress_info_key.push_back(std::make_tuple(d_propagator_v, d_propagator_u, is_half_bond_length));
                    continue;
                }
                // Within the blocks
                else
                {
                    d_propagator_v = d_q_1[N_ORIGINAL-N_OFFSET-n-1];
                    d_propagator_u = d_q_2[n-1];
                    is_half_bond_length = false;
                }
                _block_stress_info_key.push_back(std::make_tuple(d_propagator_v, d_propagator_u, is_half_bond_length));
            }
        }

        // Create scheduler for computation of propagator
        sc = new Scheduler(molecules->get_essential_propagator_codes(), N_SCHEDULER_STREAMS); 

        // Create streams
        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            gpu_error_check(hipStreamCreate(&streams[gpu][0])); // for kernel execution
            gpu_error_check(hipStreamCreate(&streams[gpu][1])); // for memcpy
        }

        // Create FFT plan
        const int NRANK{cb->get_dim()};
        int n_grid[NRANK];

        if(cb->get_dim() == 3)
        {
            n_grid[0] = cb->get_nx(0);
            n_grid[1] = cb->get_nx(1);
            n_grid[2] = cb->get_nx(2);
        }
        else if(cb->get_dim() == 2)
        {
            n_grid[0] = cb->get_nx(0);
            n_grid[1] = cb->get_nx(1);
        }
        else if(cb->get_dim() == 1)
        {
            n_grid[0] = cb->get_nx(0);
        }

        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            hipfftPlanMany(&plan_for_one[gpu], NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z,1);
            hipfftPlanMany(&plan_for_two[gpu], NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z,2);
            hipfftPlanMany(&plan_bak_one[gpu], NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D,1);
            hipfftPlanMany(&plan_bak_two[gpu], NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D,2);
            hipfftSetStream(plan_for_one[gpu], streams[gpu][0]);
            hipfftSetStream(plan_for_two[gpu], streams[gpu][0]);
            hipfftSetStream(plan_bak_one[gpu], streams[gpu][0]);
            hipfftSetStream(plan_bak_two[gpu], streams[gpu][0]);
        }

        // Allocate memory for pseudo-spectral: advance_propagator()
        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            gpu_error_check(hipMalloc((void**)&d_q_step_1_two[gpu], sizeof(double)*2*M));
            gpu_error_check(hipMalloc((void**)&d_qk_in_1_one[gpu], sizeof(ftsComplex)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_qk_in_1_two[gpu], sizeof(ftsComplex)*2*M_COMPLEX));
        }
        if (N_GPUS > 1)
        {
            gpu_error_check(hipSetDevice(1));
            gpu_error_check(hipMalloc((void**)&d_propagator_device_1[0], sizeof(double)*M));  // prev
            gpu_error_check(hipMalloc((void**)&d_propagator_device_1[1], sizeof(double)*M));  // next
        }

        gpu_error_check(hipSetDevice(0));
        gpu_error_check(hipMalloc((void**)&d_q_half_step, sizeof(double)*M));
        gpu_error_check(hipMalloc((void**)&d_q_junction,  sizeof(double)*M));
        gpu_error_check(hipMalloc((void**)&d_phi, sizeof(double)*M));

        double q_unity[M];
        for(int i=0; i<M; i++)
            q_unity[i] = 1.0;
        gpu_error_check(hipMalloc((void**)&d_q_unity, sizeof(double)*M));
        gpu_error_check(hipMemcpy(d_q_unity, q_unity, sizeof(double)*M, hipMemcpyHostToDevice));

        // Allocate memory for stress calculation: compute_stress()
        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            gpu_error_check(hipMalloc((void**)&d_fourier_basis_x[gpu], sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_fourier_basis_y[gpu], sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_fourier_basis_z[gpu], sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_stress_sum[gpu],      sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_stress_sum_out[gpu],  sizeof(double)*1));
            gpu_error_check(hipMalloc((void**)&d_stress_q[gpu][0],     sizeof(double)*2*M)); // prev
            gpu_error_check(hipMalloc((void**)&d_stress_q[gpu][1],     sizeof(double)*2*M)); // next
            gpu_error_check(hipMalloc((void**)&d_q_multi[gpu],         sizeof(double)*M_COMPLEX));
        }
        // Allocate memory for cub reduction sum
        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            d_temp_storage[gpu] = nullptr;
            temp_storage_bytes[gpu] = 0;
            hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
            gpu_error_check(hipMalloc(&d_temp_storage[gpu], temp_storage_bytes[gpu]));
        }
        update_bond_function();

        gpu_error_check(hipSetDevice(0));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
CudaPseudoDiscrete::~CudaPseudoDiscrete()
{
    const int N_GPUS = CudaCommon::get_instance().get_n_gpus();
    
    for(int gpu=0; gpu<N_GPUS; gpu++)
    {
        hipfftDestroy(plan_for_one[gpu]);
        hipfftDestroy(plan_for_two[gpu]);
        hipfftDestroy(plan_bak_one[gpu]);
        hipfftDestroy(plan_bak_two[gpu]);
    }

    delete sc;

    delete[] single_partitions;

    for(int gpu=0; gpu<N_GPUS; gpu++)
    {
        for(const auto& item: d_boltz_bond[gpu])
            hipFree(item.second);
        for(const auto& item: d_boltz_bond_half[gpu])
            hipFree(item.second);
        for(const auto& item: d_exp_dw[gpu])
            hipFree(item.second);
    }

    for(const auto& item: d_propagator)
    {
        for(int i=0; i<propagator_size[item.first]; i++)
            hipFree(item.second[i]);
        delete[] item.second;
    }
    for(const auto& item: d_block_phi)
        hipFree(item.second);
    for(const auto& item: d_propagator_junction)
        hipFree(item.second);

    #ifndef NDEBUG
    for(const auto& item: propagator_finished)
        delete[] item.second;
    #endif

    // For pseudo-spectral: advance_propagator()
    for(int gpu=0; gpu<N_GPUS; gpu++)
    {
        hipFree(d_q_step_1_two[gpu]);
        hipFree(d_qk_in_1_one[gpu]);
        hipFree(d_qk_in_1_two[gpu]);
    }

    if (N_GPUS > 1)
    {
        hipFree(d_propagator_device_1[0]);
        hipFree(d_propagator_device_1[1]);
    }

    hipFree(d_phi);
    hipFree(d_q_unity);
    hipFree(d_q_half_step);
    hipFree(d_q_junction);

    // For stress calculation: compute_stress()
    for(int gpu=0; gpu<N_GPUS; gpu++)
    {
        hipFree(d_fourier_basis_x[gpu]);
        hipFree(d_fourier_basis_y[gpu]);
        hipFree(d_fourier_basis_z[gpu]);
        hipFree(d_stress_q[gpu][0]);
        hipFree(d_stress_q[gpu][1]);
        hipFree(d_stress_sum[gpu]);
        hipFree(d_stress_sum_out[gpu]);
        hipFree(d_q_multi[gpu]);
        hipFree(d_temp_storage[gpu]);
    }

    // Destroy streams
    for(int gpu=0; gpu<N_GPUS; gpu++)
    {
        hipStreamDestroy(streams[gpu][0]);
        hipStreamDestroy(streams[gpu][1]);
    }
}

void CudaPseudoDiscrete::update_bond_function()
{
    try
    {
        // For pseudo-spectral: advance_propagator()
        const int M_COMPLEX = this->n_complex_grid;
        const int N_GPUS = CudaCommon::get_instance().get_n_gpus();
        double boltz_bond[M_COMPLEX], boltz_bond_half[M_COMPLEX];
        
        for(const auto& item: molecules->get_bond_lengths())
        {
            std::string monomer_type = item.first;
            double bond_length_sq = item.second*item.second;

            get_boltz_bond(boltz_bond     , bond_length_sq,   cb->get_nx(), cb->get_dx(), molecules->get_ds());
            get_boltz_bond(boltz_bond_half, bond_length_sq/2, cb->get_nx(), cb->get_dx(), molecules->get_ds());
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                gpu_error_check(hipMemcpy(d_boltz_bond     [gpu][monomer_type], boltz_bond,      sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
                gpu_error_check(hipMemcpy(d_boltz_bond_half[gpu][monomer_type], boltz_bond_half, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
            }
        }
        // For stress calculation: compute_stress()
        double fourier_basis_x[M_COMPLEX];
        double fourier_basis_y[M_COMPLEX];
        double fourier_basis_z[M_COMPLEX];
        get_weighted_fourier_basis(fourier_basis_x, fourier_basis_y, fourier_basis_z, cb->get_nx(), cb->get_dx());
        for(int gpu=0; gpu<N_GPUS; gpu++)
        {
            gpu_error_check(hipSetDevice(gpu));
            gpu_error_check(hipMemcpy(d_fourier_basis_x[gpu], fourier_basis_x, sizeof(double)*M_COMPLEX, hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_fourier_basis_y[gpu], fourier_basis_y, sizeof(double)*M_COMPLEX, hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_fourier_basis_z[gpu], fourier_basis_z, sizeof(double)*M_COMPLEX, hipMemcpyHostToDevice));
        }
        gpu_error_check(hipSetDevice(0));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoDiscrete::compute_statistics(
    std::map<std::string, const double*> w_input,
    std::map<std::string, const double*> q_init, std::string device)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();
        const int N_GPUS = CudaCommon::get_instance().get_n_gpus();

        const int M = cb->get_n_grid();
        const double ds = molecules->get_ds();

        hipMemcpyKind cudaMemcpyInputToDevice;
        if (device == "gpu")
            cudaMemcpyInputToDevice = hipMemcpyDeviceToDevice;
        else if(device == "cpu")
            cudaMemcpyInputToDevice = hipMemcpyHostToDevice;
        else
        {
            throw_with_line_number("Invalid device \"" + device + "\".");
        }

        for(const auto& item: molecules->get_essential_propagator_codes())
        {
            if( w_input.find(item.second.monomer_type) == w_input.end())
                throw_with_line_number("monomer_type \"" + item.second.monomer_type + "\" is not in w_input.");
        }

        for(const auto& item: w_input)
        {
            if( d_exp_dw[0].find(item.first) == d_exp_dw[0].end())
                throw_with_line_number("monomer_type \"" + item.first + "\" is not in d_exp_dw.");     
        }

        // Compute exp_dw
        for(const auto& item: w_input)
        {
            std::string monomer_type = item.first;
            const double *w = item.second;

            // Copy field configurations from host to device
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                gpu_error_check(hipMemcpyAsync(
                    d_exp_dw[gpu][monomer_type], w,      
                    sizeof(double)*M, cudaMemcpyInputToDevice, streams[gpu][1]));
            }

            // Compute exp_dw and exp_dw_half
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                exp_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][1]>>>
                    (d_exp_dw[gpu][monomer_type], d_exp_dw[gpu][monomer_type], 1.0, -1*ds, M);
                
            }
            // Synchronize all GPUs
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                gpu_error_check(hipDeviceSynchronize());
            }
        }

        // For each time span
        auto& branch_schedule = sc->get_schedule();
        for (auto parallel_job = branch_schedule.begin(); parallel_job != branch_schedule.end(); parallel_job++)
        {
            gpu_error_check(hipSetDevice(0));
            // For each propagator
            for(size_t job=0; job<parallel_job->size(); job++)
            {
                auto& key = std::get<0>((*parallel_job)[job]);
                int n_segment_from = std::get<1>((*parallel_job)[job]);
                int n_segment_to = std::get<2>((*parallel_job)[job]);
                auto& deps = molecules->get_essential_propagator_code(key).deps;
                auto monomer_type = molecules->get_essential_propagator_code(key).monomer_type;

                // Check key
                #ifndef NDEBUG
                if (d_propagator.find(key) == d_propagator.end())
                    throw_with_line_number("Could not find key '" + key + "'. ");
                #endif
                double **_d_propagator = d_propagator[key];

                // Calculate one block end
                if(n_segment_from == 1 && deps.size() == 0) // if it is leaf node
                {
                     // Q_init
                    if (key[0] == '{')
                    {
                        std::string g = Molecules::get_q_input_idx_from_key(key);
                        if (q_init.find(g) == q_init.end())
                            throw_with_line_number("Could not find q_init[\"" + g + "\"].");
                        gpu_error_check(hipMemcpy(_d_propagator[0], q_init[g], sizeof(double)*M, cudaMemcpyInputToDevice));
                        multi_real<<<N_BLOCKS, N_THREADS>>>(_d_propagator[0], _d_propagator[0], d_exp_dw[0][monomer_type], 1.0, M);
                    }
                    else
                    {
                        gpu_error_check(hipMemcpy(_d_propagator[0], d_exp_dw[0][monomer_type], sizeof(double)*M, hipMemcpyDeviceToDevice));
                    }
                    
                    #ifndef NDEBUG
                    propagator_finished[key][0] = true;
                    #endif
                }
                // If it is not leaf node
                else if (n_segment_from == 1 && deps.size() > 0)
                {
                    // If it is superposed
                    if (key[0] == '[')
                    {
                        // Initialize to zero
                        gpu_error_check(hipMemset(_d_propagator[0], 0, sizeof(double)*M));

                        for(size_t d=0; d<deps.size(); d++)
                        {
                            std::string sub_dep = std::get<0>(deps[d]);
                            int sub_n_segment   = std::get<1>(deps[d]);
                            int sub_n_repeated  = std::get<2>(deps[d]);

                            // Check sub key
                            #ifndef NDEBUG
                            if (d_propagator.find(sub_dep) == d_propagator.end())
                                throw_with_line_number("Could not find sub key '" + sub_dep + "'. ");
                            if (!propagator_finished[sub_dep][sub_n_segment-1])
                                throw_with_line_number("Could not compute '" + key +  "', since '"+ sub_dep + std::to_string(sub_n_segment) + "' is not prepared.");
                            #endif

                            lin_comb<<<N_BLOCKS, N_THREADS>>>(
                                _d_propagator[0], 1.0, _d_propagator[0],
                                sub_n_repeated, d_propagator[sub_dep][sub_n_segment-1], M);
                        }
                        advance_one_propagator(0,
                            _d_propagator[0],
                            _d_propagator[0],
                            d_boltz_bond[0][monomer_type],
                            d_exp_dw[0][monomer_type]);

                        #ifndef NDEBUG
                        propagator_finished[key][0] = true;
                        #endif
                    }
                    else
                    {
                        // Example (four branches)
                        //     A
                        //     |
                        // O - . - B
                        //     |
                        //     C

                        // Legend)
                        // .       : junction
                        // O       : full segment
                        // -, |    : half bonds
                        // A, B, C : other full segments

                        // Combine branches
                        gpu_error_check(hipMemcpy(d_q_junction, d_q_unity, sizeof(double)*M, hipMemcpyDeviceToDevice));

                        for(size_t d=0; d<deps.size(); d++)
                        {
                            std::string sub_dep = std::get<0>(deps[d]);
                            int sub_n_segment   = std::get<1>(deps[d]);

                            // Check sub key
                            #ifndef NDEBUG
                            if (d_propagator.find(sub_dep) == d_propagator.end())
                                throw_with_line_number("Could not find sub key '" + sub_dep + "'. ");
                            if (!propagator_finished[sub_dep][sub_n_segment-1])
                                throw_with_line_number("Could not compute '" + key +  "', since '"+ sub_dep + std::to_string(sub_n_segment) + "' is not prepared.");
                            #endif

                            advance_propagator_half_bond_step(0,
                                d_propagator[sub_dep][sub_n_segment-1],
                                d_q_half_step, d_boltz_bond_half[0][molecules->get_essential_propagator_code(sub_dep).monomer_type]);

                            multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_junction, d_q_junction, d_q_half_step, 1.0, M);
                        }
                        gpu_error_check(hipMemcpy(d_propagator_junction[key], d_q_junction, sizeof(double)*M, hipMemcpyDeviceToDevice));

                        // Add half bond
                        advance_propagator_half_bond_step(0, d_q_junction, _d_propagator[0], d_boltz_bond_half[0][monomer_type]);

                        // Add full segment
                        multi_real<<<N_BLOCKS, N_THREADS>>>(_d_propagator[0], _d_propagator[0], d_exp_dw[0][monomer_type], 1.0, M);

                        #ifndef NDEBUG
                        propagator_finished[key][0] = true;
                        #endif
                    }
                }
                else
                {
                    int n = n_segment_from-1;

                    #ifndef NDEBUG
                    if (!propagator_finished[key][n-1])
                        throw_with_line_number("unfinished, key: " + key + ", " + std::to_string(n-1));
                    #endif

                    advance_one_propagator(0, 
                        _d_propagator[n-1],
                        _d_propagator[n],
                        d_boltz_bond[0][monomer_type],
                        d_exp_dw[0][monomer_type]);

                    #ifndef NDEBUG
                    propagator_finished[key][n] = true;
                    #endif
                }
                hipDeviceSynchronize();
            }

            // Synchronize all GPUs
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                gpu_error_check(hipDeviceSynchronize());
            }

            // Advance propagator successively
            if(parallel_job->size()==1)
            {
                gpu_error_check(hipSetDevice(0));
                auto& key = std::get<0>((*parallel_job)[0]);
                int n_segment_from = std::get<1>((*parallel_job)[0]);
                int n_segment_to = std::get<2>((*parallel_job)[0]);
                auto monomer_type = molecules->get_essential_propagator_code(key).monomer_type;
                double **_d_propagator_key = d_propagator[key];

                for(int n=n_segment_from; n<n_segment_to; n++)
                {
                    #ifndef NDEBUG
                    if (!propagator_finished[key][n-1])
                        throw_with_line_number("unfinished, key: " + key + ", " + std::to_string(n-1));
                    #endif

                    advance_one_propagator(0, 
                        _d_propagator_key[n-1],
                        _d_propagator_key[n],
                        d_boltz_bond[0][monomer_type],
                        d_exp_dw[0][monomer_type]);

                    #ifndef NDEBUG
                    propagator_finished[key][n] = true;
                    #endif
                }
            }
            else if(parallel_job->size()==2)
            {

                const int N_JOBS = 2;
                std::string keys[N_JOBS];
                int n_segment_froms[N_JOBS];
                int n_segment_tos[N_JOBS];
                std::string monomer_types[N_JOBS];
                double **_d_propagator_keys[N_JOBS];
                
                for(int j=0; j<N_JOBS; j++)
                {
                    keys[j] = std::get<0>((*parallel_job)[j]);
                    n_segment_froms[j] = std::get<1>((*parallel_job)[j]);
                    n_segment_tos[j] = std::get<2>((*parallel_job)[j]);
                    monomer_types[j] = molecules->get_essential_propagator_code(keys[j]).monomer_type;
                    _d_propagator_keys[j] = d_propagator[keys[j]];
                }

                if (N_GPUS > 1)
                {
                    int prev, next;
                    prev = 0;
                    next = 1;

                    // Copy propagator of key1 from device0 to device1
                    gpu_error_check(hipMemcpy(
                        d_propagator_device_1[prev],
                        _d_propagator_keys[1][n_segment_froms[1]-1],
                        sizeof(double)*M, hipMemcpyDeviceToDevice));

                    for(int n=0; n<n_segment_tos[0]-n_segment_froms[0]; n++)
                    {
                        #ifndef NDEBUG
                        if (!propagator_finished[keys[0]][n-1+n_segment_froms[0]])
                            throw_with_line_number("unfinished, key: " + keys[0] + ", " + std::to_string(n-1+n_segment_froms[0]));
                        if (!propagator_finished[keys[1]][n-1+n_segment_froms[1]])
                            throw_with_line_number("unfinished, key: " + keys[1] + ", " + std::to_string(n-1+n_segment_froms[1]));
                        #endif

                        // DEVICE 0,1, STREAM 0: calculate propagators 
                        advance_two_propagators_two_gpus(
                            _d_propagator_keys[0][n-1+n_segment_froms[0]],
                            d_propagator_device_1[prev],
                            _d_propagator_keys[0][n+n_segment_froms[0]],
                            d_propagator_device_1[next],
                            d_boltz_bond[0][monomer_types[0]],
                            d_boltz_bond[1][monomer_types[1]],
                            d_exp_dw[0][monomer_types[0]],
                            d_exp_dw[1][monomer_types[1]]);

                        // DEVICE 1, STREAM 1: copy memory from device 1 to device 0
                        if (n > 0)
                        {
                            gpu_error_check(hipMemcpyAsync(
                                _d_propagator_keys[1][n-1+n_segment_froms[1]],
                                d_propagator_device_1[prev],
                                sizeof(double)*M, hipMemcpyDeviceToDevice, streams[1][1]));
                        }

                        // Synchronize all GPUs
                        for(int gpu=0; gpu<N_GPUS; gpu++)
                        {
                            gpu_error_check(hipSetDevice(gpu));
                            gpu_error_check(hipDeviceSynchronize());
                        }

                        std::swap(prev, next);

                        #ifndef NDEBUG
                        propagator_finished[keys[0]][n+n_segment_froms[0]] = true;
                        propagator_finished[keys[1]][n+n_segment_froms[1]] = true;
                        #endif
                    }
                    gpu_error_check(hipMemcpy(
                        _d_propagator_keys[1][n_segment_tos[1]-1],
                        d_propagator_device_1[prev],
                        sizeof(double)*M, hipMemcpyDeviceToDevice));
                }
                else
                {
                    gpu_error_check(hipSetDevice(0));
                    for(int n=0; n<n_segment_tos[0]-n_segment_froms[0]; n++)
                    {
                        #ifndef NDEBUG
                        if (!propagator_finished[keys[0]][n-1+n_segment_froms[0]])
                            throw_with_line_number("unfinished, key: " + keys[0] + ", " + std::to_string(n-n_segment_froms[0]));
                        if (!propagator_finished[keys[1]][n-1+n_segment_froms[1]])
                            throw_with_line_number("unfinished, key: " + keys[1] + ", " + std::to_string(n-n_segment_froms[1]));
                        #endif

                        advance_two_propagators(
                            _d_propagator_keys[0][n-1+n_segment_froms[0]],
                            _d_propagator_keys[1][n-1+n_segment_froms[1]],
                            _d_propagator_keys[0][n+n_segment_froms[0]],
                            _d_propagator_keys[1][n+n_segment_froms[1]],
                            d_boltz_bond[0][monomer_types[0]],
                            d_boltz_bond[0][monomer_types[1]],
                            d_exp_dw[0][monomer_types[0]],
                            d_exp_dw[0][monomer_types[1]]);

                        #ifndef NDEBUG
                        propagator_finished[keys[0]][n+n_segment_froms[0]] = true;
                        propagator_finished[keys[1]][n+n_segment_froms[1]] = true;
                        #endif
                    }
                    gpu_error_check(hipDeviceSynchronize());
                }
            }

            // Synchronize all GPUs
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                gpu_error_check(hipSetDevice(gpu));
                gpu_error_check(hipDeviceSynchronize());
            }
        }
        gpu_error_check(hipSetDevice(0));

        // Compute total partition function of each distinct polymers
        for(const auto& segment_info: single_partition_segment)
        {
            int p                    = std::get<0>(segment_info);
            double *d_propagator_v   = std::get<1>(segment_info);
            double *d_propagator_u   = std::get<2>(segment_info);
            std::string monomer_type = std::get<3>(segment_info);
            int n_superposed         = std::get<4>(segment_info);

            single_partitions[p] = cb->inner_product_inverse_weight_device(
                d_propagator_v,  // q
                d_propagator_u, // q^dagger
                d_exp_dw[0][monomer_type])/n_superposed/cb->get_volume();
        }

        // Calculate segment concentrations
        for(const auto& block: d_block_phi)
        {
            const auto& key = block.first;
            int p                = std::get<0>(key);
            std::string dep_v    = std::get<1>(key);
            std::string dep_u    = std::get<2>(key);

            int n_repeated;
            int n_segment_allocated = molecules->get_essential_block(key).n_segment_allocated;
            int n_segment_offset    = molecules->get_essential_block(key).n_segment_offset;
            int n_segment_original  = molecules->get_essential_block(key).n_segment_original;
            std::string monomer_type = molecules->get_essential_block(key).monomer_type;

            // Contains no '['
            if (dep_u.find('[') == std::string::npos)
                n_repeated = molecules->get_essential_block(key).v_u.size();
            else
                n_repeated = 1;

            // Check keys
            #ifndef NDEBUG
            if (d_propagator.find(dep_v) == d_propagator.end())
                throw_with_line_number("Could not find dep_v key'" + dep_v + "'. ");
            if (d_propagator.find(dep_u) == d_propagator.end())
                throw_with_line_number("Could not find dep_u key'" + dep_u + "'. ");
            #endif

            // Calculate phi of one block (possibly multiple blocks when using superposition)
            calculate_phi_one_block(
                block.second,              // Phi
                d_propagator[dep_v],       // dependency v
                d_propagator[dep_u],       // dependency u
                d_exp_dw[0][monomer_type], // exp_dw
                n_segment_allocated,
                n_segment_offset,
                n_segment_original);
            
            // Normalize concentration
            Polymer& pc = molecules->get_polymer(p);
            double norm = molecules->get_ds()*pc.get_volume_fraction()/pc.get_alpha()/single_partitions[p]*n_repeated;
            lin_comb<<<N_BLOCKS, N_THREADS>>>(block.second, norm, block.second, 0.0, block.second, M);
        }
        gpu_error_check(hipSetDevice(0));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoDiscrete::advance_one_propagator(
    const int GPU,
    double *d_q_in, double *d_q_out,
    double *d_boltz_bond, double *d_exp_dw)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        // Execute a Forward FFT
        hipfftExecD2Z(plan_for_one[GPU], d_q_in, d_qk_in_1_one[GPU]);

        // Multiply exp(-k^2 ds/6) in fourier space
        multi_complex_real<<<N_BLOCKS, N_THREADS, 0, streams[GPU][0]>>>(d_qk_in_1_one[GPU], d_boltz_bond, M_COMPLEX);

        // Execute a backward FFT
        hipfftExecZ2D(plan_bak_one[GPU], d_qk_in_1_one[GPU], d_q_out);

        // Evaluate exp(-w*ds) in real space
        multi_real<<<N_BLOCKS, N_THREADS, 0, streams[GPU][0]>>>(d_q_out, d_q_out, d_exp_dw, 1.0/((double)M), M);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoDiscrete::advance_two_propagators(
    double *d_q_in_1, double *d_q_in_2,
    double *d_q_out_1, double *d_q_out_2,
    double *d_boltz_bond_1, double *d_boltz_bond_2,  
    double *d_exp_dw_1, double *d_exp_dw_2)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        gpu_error_check(hipMemcpyAsync(&d_q_step_1_two[0][0], d_q_in_1, sizeof(double)*M, hipMemcpyDeviceToDevice, streams[0][0]));
        gpu_error_check(hipMemcpyAsync(&d_q_step_1_two[0][M], d_q_in_2, sizeof(double)*M, hipMemcpyDeviceToDevice, streams[0][0]));

        // Execute a Forward FFT
        hipfftExecD2Z(plan_for_two[0], d_q_step_1_two[0], d_qk_in_1_two[0]);

        // Multiply exp(-k^2 ds/6) in fourier space
        complex_real_multi_bond_two<<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(
            &d_qk_in_1_two[0][0],         d_boltz_bond_1, 
            &d_qk_in_1_two[0][M_COMPLEX], d_boltz_bond_2, M_COMPLEX);

        // Execute a backward FFT
        hipfftExecZ2D(plan_bak_two[0], d_qk_in_1_two[0], d_q_step_1_two[0]);

        // Evaluate exp(-w*ds) in real space
        real_multi_exp_dw_two<<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(
            d_q_out_1, &d_q_step_1_two[0][0], d_exp_dw_1,
            d_q_out_2, &d_q_step_1_two[0][M], d_exp_dw_2, 1.0/((double)M), M);

    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoDiscrete::advance_two_propagators_two_gpus(
    double *d_q_in_1, double *d_q_in_2,
    double *d_q_out_1, double *d_q_out_2,
    double *d_boltz_bond_1, double *d_boltz_bond_2,  
    double *d_exp_dw_1, double *d_exp_dw_2)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        // Execute a Forward FFT
        gpu_error_check(hipSetDevice(0));
        hipfftExecD2Z(plan_for_one[0], d_q_in_1, d_qk_in_1_one[0]);
        gpu_error_check(hipSetDevice(1));
        hipfftExecD2Z(plan_for_one[1], d_q_in_2, d_qk_in_1_one[1]);

        // Multiply exp(-k^2 ds/6) in fourier space
        gpu_error_check(hipSetDevice(0));
        multi_complex_real<<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(d_qk_in_1_one[0], d_boltz_bond_1, M_COMPLEX);
        gpu_error_check(hipSetDevice(1));
        multi_complex_real<<<N_BLOCKS, N_THREADS, 0, streams[1][0]>>>(d_qk_in_1_one[1], d_boltz_bond_2, M_COMPLEX);

        // Execute a backward FFT
        gpu_error_check(hipSetDevice(0));
        hipfftExecZ2D(plan_bak_one[0], d_qk_in_1_one[0], d_q_out_1);
        gpu_error_check(hipSetDevice(1));
        hipfftExecZ2D(plan_bak_one[1], d_qk_in_1_one[1], d_q_out_2);

        // Evaluate exp(-w*ds) in real space
        gpu_error_check(hipSetDevice(0));
        multi_real<<<N_BLOCKS, N_THREADS, 0, streams[0][0]>>>(d_q_out_1, d_q_out_1, d_exp_dw_1, 1.0/((double)M), M);
        gpu_error_check(hipSetDevice(1));
        multi_real<<<N_BLOCKS, N_THREADS, 0, streams[1][0]>>>(d_q_out_2, d_q_out_2, d_exp_dw_2, 1.0/((double)M), M);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoDiscrete::advance_propagator_half_bond_step(const int GPU, double *d_q_in, double *d_q_out, double *d_boltz_bond_half)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        // 3D fourier discrete transform, forward and inplace
        hipfftExecD2Z(plan_for_one[GPU], d_q_in, d_qk_in_1_one[GPU]);
        // Multiply exp(-k^2 ds/12) in fourier space, in all 3 directions
        multi_complex_real<<<N_BLOCKS, N_THREADS, 0, streams[GPU][0]>>>(d_qk_in_1_one[GPU], d_boltz_bond_half, 1.0/((double)M), M_COMPLEX);
        // 3D fourier discrete transform, backward and inplace
        hipfftExecZ2D(plan_bak_one[GPU], d_qk_in_1_one[GPU], d_q_out);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoDiscrete::calculate_phi_one_block(
    double *d_phi, double **d_q_1, double **d_q_2, double *d_exp_dw, const int N, const int N_OFFSET, const int N_ORIGINAL)
{
    try
    {
        gpu_error_check(hipSetDevice(0));

        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        // Compute segment concentration
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_phi,d_q_1[N_ORIGINAL-N_OFFSET-1], d_q_2[0], 1.0, M);
        for(int n=1; n<N; n++)
        {
            add_multi_real<<<N_BLOCKS, N_THREADS>>>(d_phi, d_q_1[N_ORIGINAL-N_OFFSET-n-1], d_q_2[n], 1.0, M);
        }
        divide_real<<<N_BLOCKS, N_THREADS>>>(d_phi, d_phi, d_exp_dw, 1.0, M);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
double CudaPseudoDiscrete::get_total_partition(int polymer)
{
    try
    {
        return single_partitions[polymer];
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoDiscrete::get_total_concentration(std::string monomer_type, double *phi)
{
    try
    {
        gpu_error_check(hipSetDevice(0));

        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();
        const int M = cb->get_n_grid();

        // Initialize to zero
        gpu_error_check(hipMemset(d_phi, 0, sizeof(double)*M));

        // For each block
        for(const auto& d_block: d_block_phi)
        {
            const auto& key = d_block.first;
            std::string dep_v = std::get<1>(key);
            int n_segment_allocated = molecules->get_essential_block(key).n_segment_allocated;
            if (Molecules::get_monomer_type_from_key(dep_v) == monomer_type && n_segment_allocated != 0)
                lin_comb<<<N_BLOCKS, N_THREADS>>>(d_phi, 1.0, d_phi, 1.0, d_block.second, M);
        }
        gpu_error_check(hipMemcpy(phi, d_phi, sizeof(double)*M, hipMemcpyDeviceToHost));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoDiscrete::get_total_concentration(int p, std::string monomer_type, double *phi)
{
    try
    {
        gpu_error_check(hipSetDevice(0));

        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int P = molecules->get_n_polymer_types();

        if (p < 0 || p > P-1)
            throw_with_line_number("Index (" + std::to_string(p) + ") must be in range [0, " + std::to_string(P-1) + "]");

        // Initialize to zero
        gpu_error_check(hipMemset(d_phi, 0, sizeof(double)*M));

        // For each block
        for(const auto& d_block: d_block_phi)
        {
            const auto& key = d_block.first;
            int polymer_idx = std::get<0>(key);
            std::string dep_v = std::get<1>(key);
            int n_segment_allocated = molecules->get_essential_block(key).n_segment_allocated;
            if (polymer_idx == p && Molecules::get_monomer_type_from_key(dep_v) == monomer_type && n_segment_allocated != 0)
                lin_comb<<<N_BLOCKS, N_THREADS>>>(d_phi, 1.0, d_phi, 1.0, d_block.second, M);
        }
        gpu_error_check(hipMemcpy(phi, d_phi, sizeof(double)*M, hipMemcpyDeviceToHost));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoDiscrete::get_block_concentration(int p, double *phi)
{
    try
    {
        gpu_error_check(hipSetDevice(0));

        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int P = molecules->get_n_polymer_types();

        if (p < 0 || p > P-1)
            throw_with_line_number("Index (" + std::to_string(p) + ") must be in range [0, " + std::to_string(P-1) + "]");

        if (molecules->is_using_superposition())
            throw_with_line_number("Disable 'superposition' option to obtain concentration of each block.");

        // Initialize to zero
        gpu_error_check(hipMemset(d_phi, 0, sizeof(double)*M));

        Polymer& pc = molecules->get_polymer(p);
        std::vector<Block>& blocks = pc.get_blocks();

        for(size_t b=0; b<blocks.size(); b++)
        {
            std::string dep_v = pc.get_propagator_key(blocks[b].v, blocks[b].u);
            std::string dep_u = pc.get_propagator_key(blocks[b].u, blocks[b].v);
            if (dep_v < dep_u)
                dep_v.swap(dep_u);

            lin_comb<<<N_BLOCKS, N_THREADS>>>(d_phi, 0.0, d_phi, 1.0, d_block_phi[std::make_tuple(p, dep_v, dep_u)], M);
            gpu_error_check(hipMemcpy(&phi[b*M], d_phi, sizeof(double)*M, hipMemcpyDeviceToHost));
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
std::vector<double> CudaPseudoDiscrete::compute_stress()
{
    // This method should be invoked after invoking compute_statistics().

    // To calculate stress, we multiply weighted fourier basis to q(k)*q^dagger(-k).
    // We only need the real part of stress calculation.
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();
        const int N_GPUS = CudaCommon::get_instance().get_n_gpus();

        const int DIM  = cb->get_dim();
        const int M    = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        auto bond_lengths = molecules->get_bond_lengths();
        std::vector<double> stress(DIM);
        std::map<std::tuple<int, std::string, std::string>, std::array<double,3>> block_dq_dl[MAX_GPUS];
        double stress_sum_out[MAX_GPUS][3];

        // Compute stress for each block
        for(const auto& d_block: d_block_phi)
        {
            const auto& key = d_block.first;
            int p                = std::get<0>(key);
            std::string dep_v    = std::get<1>(key);
            std::string dep_u    = std::get<2>(key);

            const int N           = molecules->get_essential_block(key).n_segment_allocated;
            const int N_OFFSET    = molecules->get_essential_block(key).n_segment_offset;
            const int N_ORIGINAL  = molecules->get_essential_block(key).n_segment_original;
            std::string monomer_type = molecules->get_essential_block(key).monomer_type;

            // Contains no '['
            int n_repeated;
            if (dep_u.find('[') == std::string::npos)
                n_repeated = molecules->get_essential_block(key).v_u.size();
            else
                n_repeated = 1;

            double bond_length_sq[MAX_GPUS][2];       // one for prev, the other for next
            double *d_boltz_bond_now[MAX_GPUS][2];    // one for prev, the other for next
            double **d_q_1 = d_propagator[dep_v];     // Propagator q
            double **d_q_2 = d_propagator[dep_u];     // Propagator q^dagger

            std::array<double,3> _block_dq_dl[MAX_GPUS];
            for(int gpu=0; gpu<N_GPUS; gpu++)
                for(int d=0; d<3; d++)
                    _block_dq_dl[gpu][d] = 0.0;

            // Check block_stress_info
            const auto& _block_stress_info_key = block_stress_info[key];
            if(_block_stress_info_key.size() != (unsigned int) (N+1))
            {
                throw_with_line_number("Mismatch of block_stress_info("
                    + std::to_string(p) + "," + dep_v + "," + dep_u + ") "
                    + std::to_string(_block_stress_info_key.size()) + ".size() with N+1 (" + std::to_string(N+1) + ")");
            }

            // Variables for block_stress_info
            double *d_propagator_v;
            double *d_propagator_u;
            bool is_half_bond_length;

            int prev, next;
            prev = 0;
            next = 1;

            // Copy memory from device to device
            for(int gpu=0; gpu<N_GPUS; gpu++)
            {
                // Index
                int idx = gpu;
                gpu_error_check(hipSetDevice(gpu));
                if (idx <= N)
                {
                    d_propagator_v = std::get<0>(_block_stress_info_key[idx]);
                    d_propagator_u = std::get<1>(_block_stress_info_key[idx]);
                    is_half_bond_length = std::get<2>(_block_stress_info_key[idx]);

                    if (d_propagator_v != nullptr)
                    {
                        gpu_error_check(hipMemcpy(&d_stress_q[gpu][prev][0], d_propagator_v, sizeof(double)*M, hipMemcpyDeviceToDevice));
                        gpu_error_check(hipMemcpy(&d_stress_q[gpu][prev][M], d_propagator_u, sizeof(double)*M, hipMemcpyDeviceToDevice));

                        if(is_half_bond_length)
                        {
                            bond_length_sq[gpu][prev] = 0.5*bond_lengths[monomer_type]*bond_lengths[monomer_type];
                            d_boltz_bond_now[gpu][prev] = d_boltz_bond_half[gpu][monomer_type];
                        }
                        else
                        {
                            bond_length_sq[gpu][prev] = bond_lengths[monomer_type]*bond_lengths[monomer_type];
                            d_boltz_bond_now[gpu][prev] = d_boltz_bond[gpu][monomer_type];
                        }
                    }
                }
            }

            // Compute stress
            for(int n=0; n<=N; n+=N_GPUS)
            {
                // STREAM 1: copy memory from device to device
                for(int gpu=0; gpu<N_GPUS; gpu++)
                {
                    const int idx = n + gpu;
                    const int idx_next = idx + N_GPUS;
                    
                    gpu_error_check(hipSetDevice(gpu));
                    if (idx_next <= N)
                    {
                        d_propagator_v = std::get<0>(_block_stress_info_key[idx_next]);
                        d_propagator_u = std::get<1>(_block_stress_info_key[idx_next]);
                        is_half_bond_length = std::get<2>(_block_stress_info_key[idx_next]);

                        if (d_propagator_v != nullptr)
                        {
                            gpu_error_check(hipMemcpyAsync(&d_stress_q[gpu][next][0], d_propagator_v, sizeof(double)*M, hipMemcpyDeviceToDevice, streams[gpu][1]));
                            gpu_error_check(hipMemcpyAsync(&d_stress_q[gpu][next][M], d_propagator_u, sizeof(double)*M, hipMemcpyDeviceToDevice, streams[gpu][1]));

                            if(is_half_bond_length)
                            {
                                bond_length_sq[gpu][next] = 0.5*bond_lengths[monomer_type]*bond_lengths[monomer_type];
                                d_boltz_bond_now[gpu][next] = d_boltz_bond_half[gpu][monomer_type];
                            }
                            else
                            {
                                bond_length_sq[gpu][next] = bond_lengths[monomer_type]*bond_lengths[monomer_type];
                                d_boltz_bond_now[gpu][next] = d_boltz_bond[gpu][monomer_type];
                            }
                        }
                    }
                }

                // STREAM 0: execute kernels
                for(int gpu=0; gpu<N_GPUS; gpu++)
                {
                    const int idx = n + gpu;
                    gpu_error_check(hipSetDevice(gpu));
                    if (idx <= N)
                    {
                        d_propagator_v = std::get<0>(_block_stress_info_key[idx]);
                        d_propagator_u = std::get<1>(_block_stress_info_key[idx]);
                        is_half_bond_length = std::get<2>(_block_stress_info_key[idx]);

                        if (d_propagator_v != nullptr)
                        {
                            // Execute a Forward FFT
                            hipfftExecD2Z(plan_for_two[gpu], d_stress_q[gpu][prev], d_qk_in_1_two[gpu]);
                            // Multiply two propagators in the fourier spaces
                            multi_complex_conjugate<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_q_multi[gpu], &d_qk_in_1_two[gpu][0], &d_qk_in_1_two[gpu][M_COMPLEX], M_COMPLEX);
                            multi_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_q_multi[gpu], d_q_multi[gpu], d_boltz_bond_now[gpu][prev], bond_length_sq[gpu][prev], M_COMPLEX);
                        }
                    }
                }

                // STREAM 0: reduction sum
                for(int gpu=0; gpu<N_GPUS; gpu++)
                {
                    const int idx = n + gpu;
                    gpu_error_check(hipSetDevice(gpu));
                    if (idx <= N)
                    {
                        d_propagator_v = std::get<0>(_block_stress_info_key[idx]);
                        d_propagator_u = std::get<1>(_block_stress_info_key[idx]);
                        is_half_bond_length = std::get<2>(_block_stress_info_key[idx]);
                        if (d_propagator_v != nullptr)
                        {
                            if ( DIM == 3 )
                            {
                                // x direction
                                multi_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_stress_sum[gpu], d_q_multi[gpu], d_fourier_basis_x[gpu], 1.0, M_COMPLEX);
                                hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
                                gpu_error_check(hipMemcpyAsync(&stress_sum_out[gpu][0],d_stress_sum_out[gpu],sizeof(double),hipMemcpyDeviceToHost, streams[gpu][0]));

                                // y direction
                                multi_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_stress_sum[gpu], d_q_multi[gpu], d_fourier_basis_y[gpu], 1.0, M_COMPLEX);
                                hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
                                gpu_error_check(hipMemcpyAsync(&stress_sum_out[gpu][1],d_stress_sum_out[gpu],sizeof(double),hipMemcpyDeviceToHost, streams[gpu][0]));

                                // z direction
                                multi_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_stress_sum[gpu], d_q_multi[gpu], d_fourier_basis_z[gpu], 1.0, M_COMPLEX);
                                hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
                                gpu_error_check(hipMemcpyAsync(&stress_sum_out[gpu][2],d_stress_sum_out[gpu],sizeof(double),hipMemcpyDeviceToHost, streams[gpu][0]));
                            }
                            if ( DIM == 2 )
                            {
                                // y direction
                                multi_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_stress_sum[gpu], d_q_multi[gpu], d_fourier_basis_y[gpu], 1.0, M_COMPLEX);
                                hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
                                gpu_error_check(hipMemcpyAsync(&stress_sum_out[gpu][0],d_stress_sum_out[gpu],sizeof(double),hipMemcpyDeviceToHost, streams[gpu][0]));

                                // z direction
                                multi_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_stress_sum[gpu], d_q_multi[gpu], d_fourier_basis_z[gpu], 1.0, M_COMPLEX);
                                hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
                                gpu_error_check(hipMemcpyAsync(&stress_sum_out[gpu][1],d_stress_sum_out[gpu],sizeof(double),hipMemcpyDeviceToHost, streams[gpu][0]));
                            }
                            if ( DIM == 1 )
                            {
                                // z direction
                                multi_real<<<N_BLOCKS, N_THREADS, 0, streams[gpu][0]>>>(d_stress_sum[gpu], d_q_multi[gpu], d_fourier_basis_z[gpu], 1.0, M_COMPLEX);
                                hipcub::DeviceReduce::Sum(d_temp_storage[gpu], temp_storage_bytes[gpu], d_stress_sum[gpu], d_stress_sum_out[gpu], M_COMPLEX, streams[gpu][0]);
                                gpu_error_check(hipMemcpyAsync(&stress_sum_out[gpu][0],d_stress_sum_out[gpu],sizeof(double),hipMemcpyDeviceToHost, streams[gpu][0]));
                            }
                            // Synchronize streams and add results
                            gpu_error_check(hipStreamSynchronize(streams[gpu][0]));
                            for(int d=0; d<DIM; d++)
                                _block_dq_dl[gpu][d] += stress_sum_out[gpu][d]*n_repeated;
                        }
                    }
                }
                // Synchronize all GPUs
                for(int gpu=0; gpu<N_GPUS; gpu++)
                {
                    gpu_error_check(hipSetDevice(gpu));
                    gpu_error_check(hipDeviceSynchronize());
                }
                std::swap(prev, next);
            }
            // Copy stress data
            for(int gpu=0; gpu<N_GPUS; gpu++)
                block_dq_dl[gpu][key] = _block_dq_dl[gpu];
        }
        gpu_error_check(hipSetDevice(0));

        // Compute total stress
        for(int d=0; d<DIM; d++)
            stress[d] = 0.0;
        for(const auto& d_block: d_block_phi)
        {
            const auto& key = d_block.first;
            int p             = std::get<0>(key);
            std::string dep_v = std::get<1>(key);
            std::string dep_u = std::get<2>(key);
            Polymer& pc  = molecules->get_polymer(p);

            for(int gpu=0; gpu<N_GPUS; gpu++)
                for(int d=0; d<DIM; d++)
                    stress[d] += block_dq_dl[gpu][key][d]*pc.get_volume_fraction()/pc.get_alpha()/single_partitions[p];
        }
        for(int d=0; d<DIM; d++)
            stress[d] /= -3.0*cb->get_lx(d)*M*M/molecules->get_ds();
            
        return stress;
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoDiscrete::get_chain_propagator(double *q_out, int polymer, int v, int u, int n)
{ 
    // This method should be invoked after invoking compute_statistics()

    // Get chain propagator for a selected polymer, block and direction.
    // This is made for debugging and testing.
    try
    {
        const int M = cb->get_n_grid();
        Polymer& pc = molecules->get_polymer(polymer);
        std::string dep = pc.get_propagator_key(v,u);

        if (molecules->get_essential_propagator_codes().find(dep) == molecules->get_essential_propagator_codes().end())
            throw_with_line_number("Could not find the propagator code '" + dep + "'. Disable 'superposition' option to obtain propagators.");

        const int N = molecules->get_essential_propagator_codes()[dep].max_n_segment;
        if (n < 1 || n > N)
            throw_with_line_number("n (" + std::to_string(n) + ") must be in range [1, " + std::to_string(N) + "]");

        gpu_error_check(hipMemcpy(q_out, d_propagator[dep][n-1], sizeof(double)*M,hipMemcpyDeviceToHost));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
