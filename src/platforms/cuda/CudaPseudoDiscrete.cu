#include "hip/hip_runtime.h"
#define THRUST_IGNORE_DEPRECATED_CPP_DIALECToptimal
#define CUB_IGNORE_DEPRECATED_CPP_DIALECT

#include <complex>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <iostream>
#include "CudaPseudoDiscrete.h"
#include "CudaComputationBox.h"

CudaPseudoDiscrete::CudaPseudoDiscrete(
    ComputationBox *cb,
    Mixture *mx)
    : Pseudo(cb, mx)
{
    try
    {
        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        // allocate memory for partition functions
        if( mx->get_unique_branches().size() == 0)
            throw_with_line_number("There is no unique branch. Add polymers first.");
        for(const auto& item: mx->get_unique_branches())
        {
            std::string dep = item.first;
            int max_n_segment = item.second.max_n_segment;
             // There are N segments

             // Illustration (N==5)
             // O--O--O--O--O
             // 0  1  2  3  4 unique_blocks

             // Legend)
             // -- : full bond
             // O  : full segment
            d_unique_partition[dep] = new double*[max_n_segment];
            d_unique_partition_size[dep] = max_n_segment;
            for(int i=0; i<d_unique_partition_size[dep]; i++)
                gpu_error_check(hipMalloc((void**)&d_unique_partition[dep][i], sizeof(double)*M));

            #ifndef NDEBUG
            unique_partition_finished[dep] = new bool[max_n_segment];
            for(int i=0; i<max_n_segment;i++)
                unique_partition_finished[dep][i] = false;
            #endif
        }

        // allocate memory for unique_q_junctions, which contain partition function at junction of discrete chain
        for(const auto& item: mx->get_unique_branches())
        {
            std::string dep = item.first;
            d_unique_q_junctions[dep] = nullptr;
            gpu_error_check(hipMalloc((void**)&d_unique_q_junctions[dep], sizeof(double)*M));
        }

        // allocate memory for concentrations
        if( mx->get_unique_blocks().size() == 0)
            throw_with_line_number("There is no unique block. Add polymers first.");
        for(const auto& item: mx->get_unique_blocks())
        {
            d_unique_phi[item.first] = nullptr;
            gpu_error_check(hipMalloc((void**)&d_unique_phi[item.first], sizeof(double)*M));
        }

        // create boltz_bond, boltz_bond_half, and exp_dw
        for(const auto& item: mx->get_bond_lengths())
        {
            std::string monomer_type = item.first;
            d_boltz_bond     [monomer_type] = nullptr;
            d_boltz_bond_half[monomer_type] = nullptr;
            d_exp_dw         [monomer_type] = nullptr;

            gpu_error_check(hipMalloc((void**)&d_boltz_bond     [monomer_type], sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_boltz_bond_half[monomer_type], sizeof(double)*M_COMPLEX));
            gpu_error_check(hipMalloc((void**)&d_exp_dw         [monomer_type], sizeof(double)*M));
        }

        // total partition functions for each polymer
        single_partitions = new double[mx->get_n_polymers()];

        // create scheduler for computation of partial partition function
        sc = new Scheduler(mx->get_unique_branches(), N_STREAM); 

        // Create FFT plan
        const int NRANK{cb->get_dim()};
        int n_grid[NRANK];

        if(cb->get_dim() == 3)
        {
            n_grid[0] = cb->get_nx(0);
            n_grid[1] = cb->get_nx(1);
            n_grid[2] = cb->get_nx(2);
        }
        else if(cb->get_dim() == 2)
        {
            n_grid[0] = cb->get_nx(0);
            n_grid[1] = cb->get_nx(1);
        }
        else if(cb->get_dim() == 1)
        {
            n_grid[0] = cb->get_nx(0);
        }

        hipfftPlanMany(&plan_for_1, NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z,1);
        hipfftPlanMany(&plan_bak_1, NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D,1);
        hipfftPlanMany(&plan_for_2, NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z,2);
        hipfftPlanMany(&plan_bak_2, NRANK, n_grid, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D,2);

        // allocate memory for get_concentration
        gpu_error_check(hipMalloc((void**)&d_phi, sizeof(double)*M));

        // allocate memory for pseudo-spectral: one_step()
        gpu_error_check(hipMalloc((void**)&d_qk_in_1, sizeof(ftsComplex)*M_COMPLEX));

        gpu_error_check(hipMalloc((void**)&d_qk_in_2, sizeof(ftsComplex)*2*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_q_in_temp_2, sizeof(double)*2*M));
        gpu_error_check(hipMalloc((void**)&d_q_out_temp_2, sizeof(double)*2*M));

        gpu_error_check(hipMalloc((void**)&d_q_half_step, sizeof(double)*M));
        gpu_error_check(hipMalloc((void**)&d_q_junction,  sizeof(ftsComplex)*M_COMPLEX));
        
        // allocate memory for stress calculation: compute_stress()
        gpu_error_check(hipMalloc((void**)&d_fourier_basis_x, sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_fourier_basis_y, sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_fourier_basis_z, sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_qk_1,        sizeof(ftsComplex)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_qk_2,        sizeof(ftsComplex)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_q_multi,         sizeof(double)*M_COMPLEX));
        gpu_error_check(hipMalloc((void**)&d_stress_sum,      sizeof(double)*M_COMPLEX));

        update_bond_function();
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
CudaPseudoDiscrete::~CudaPseudoDiscrete()
{
    hipfftDestroy(plan_for_1);
    hipfftDestroy(plan_bak_1);
    hipfftDestroy(plan_for_2);
    hipfftDestroy(plan_bak_2);

    delete sc;

    delete[] single_partitions;

    for(const auto& item: d_boltz_bond)
        hipFree(item.second);
    for(const auto& item: d_boltz_bond_half)
        hipFree(item.second);
    for(const auto& item: d_exp_dw)
        hipFree(item.second);
    for(const auto& item: d_unique_partition)
    {
        for(int i=0; i<d_unique_partition_size[item.first]; i++)
            hipFree(item.second[i]);
        delete[] item.second;
    }
    for(const auto& item: d_unique_phi)
        hipFree(item.second);
    for(const auto& item: d_unique_q_junctions)
        hipFree(item.second);

    #ifndef NDEBUG
    for(const auto& item: unique_partition_finished)
        delete[] item.second;
    #endif

    // for get_concentration
    hipFree(d_phi);

    // for pseudo-spectral: one_step()
    hipFree(d_qk_in_1);

    hipFree(d_qk_in_2);
    hipFree(d_q_in_temp_2);
    hipFree(d_q_out_temp_2);
    
    hipFree(d_q_half_step);
    hipFree(d_q_junction);

    // for stress calculation: compute_stress()
    hipFree(d_fourier_basis_x);
    hipFree(d_fourier_basis_y);
    hipFree(d_fourier_basis_z);
    hipFree(d_qk_1);
    hipFree(d_qk_2);
    hipFree(d_q_multi);
    hipFree(d_stress_sum);
}

void CudaPseudoDiscrete::update_bond_function()
{
    try
    {
        // for pseudo-spectral: one_step()
        const int M_COMPLEX = this->n_complex_grid;
        double boltz_bond[M_COMPLEX], boltz_bond_half[M_COMPLEX];
        
        for(const auto& item: mx->get_bond_lengths())
        {
            std::string monomer_type = item.first;
            double bond_length_sq = item.second*item.second;

            get_boltz_bond(boltz_bond     , bond_length_sq,   cb->get_nx(), cb->get_dx(), mx->get_ds());
            get_boltz_bond(boltz_bond_half, bond_length_sq/2, cb->get_nx(), cb->get_dx(), mx->get_ds());

            gpu_error_check(hipMemcpy(d_boltz_bond     [monomer_type], boltz_bond,      sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
            gpu_error_check(hipMemcpy(d_boltz_bond_half[monomer_type], boltz_bond_half, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
        }

        // for stress calculation: compute_stress()
        double fourier_basis_x[M_COMPLEX];
        double fourier_basis_y[M_COMPLEX];
        double fourier_basis_z[M_COMPLEX];
        get_weighted_fourier_basis(fourier_basis_x, fourier_basis_y, fourier_basis_z, cb->get_nx(), cb->get_dx());
        gpu_error_check(hipMemcpy(d_fourier_basis_x, fourier_basis_x, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
        gpu_error_check(hipMemcpy(d_fourier_basis_y, fourier_basis_y, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
        gpu_error_check(hipMemcpy(d_fourier_basis_z, fourier_basis_z, sizeof(double)*M_COMPLEX,hipMemcpyHostToDevice));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoDiscrete::compute_statistics(
    std::map<std::string, double*> w_input,
    std::map<std::string, double*> q_init)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const double ds = mx->get_ds();

        for(const auto& item: mx->get_unique_branches())
        {
            if( w_input.find(item.second.monomer_type) == w_input.end())
                throw_with_line_number("monomer_type \"" + item.second.monomer_type + "\" is not in w_input.");
        }

        for(const auto& item: w_input)
        {
            if( d_exp_dw.find(item.first) == d_exp_dw.end())
                throw_with_line_number("monomer_type \"" + item.first + "\" is not in d_exp_dw.");     
        }

        // exp_dw
        double exp_dw[M];
        for(const auto& item: w_input)
        {
            std::string monomer_type = item.first;
            double *w = item.second;
            for(int i=0; i<M; i++)
                exp_dw[i] = exp(-w[i]*ds);
            gpu_error_check(hipMemcpy(d_exp_dw[monomer_type], exp_dw, sizeof(double)*M,hipMemcpyHostToDevice));
        }

        double q_uniform[M];
        for(int i=0; i<M; i++)
            q_uniform[i] = 1.0;

        // for each time span
        auto& branch_schedule = sc->get_schedule();
        for (auto parallel_job = branch_schedule.begin(); parallel_job != branch_schedule.end(); parallel_job++)
        {
            // multiplay all partition functions at junctions if necessary 
            for(size_t job=0; job<parallel_job->size(); job++)
            {
                auto& key = std::get<0>((*parallel_job)[job]);
                int n_segment_from = std::get<1>((*parallel_job)[job]);
                int n_segment_to = std::get<2>((*parallel_job)[job]);
                auto& deps = mx->get_unique_branch(key).deps;
                auto monomer_type = mx->get_unique_branch(key).monomer_type;

                // check key
                #ifndef NDEBUG
                if (d_unique_partition.find(key) == d_unique_partition.end())
                    throw_with_line_number("Could not find key '" + key + "'. ");
                #endif
                double **_d_unique_partition = d_unique_partition[key];

                // calculate one block end
                if(n_segment_from == 1 && deps.size() == 0) // if it is leaf node
                {
                     // q_init
                    if (key[0] == '{')
                    {
                        std::string g = Mixture::get_q_input_idx_from_key(key);
                        if (q_init.find(g) == q_init.end())
                            throw_with_line_number("Could not find q_init[\"" + g + "\"].");
                        gpu_error_check(hipMemcpy(_d_unique_partition[0], q_init[g], sizeof(double)*M, hipMemcpyHostToDevice));
                        multi_real<<<N_BLOCKS, N_THREADS>>>(_d_unique_partition[0], _d_unique_partition[0], d_exp_dw[monomer_type], 1.0, M);
                    }
                    else
                    {
                        gpu_error_check(hipMemcpy(_d_unique_partition[0], d_exp_dw[monomer_type], sizeof(double)*M, hipMemcpyDeviceToDevice));
                    }
                    
                    #ifndef NDEBUG
                    unique_partition_finished[key][0] = true;
                    #endif
                }
                else if (n_segment_from == 1 && deps.size() > 0) // if it is not leaf node
                {
                    // if it is superposed
                    if (key[0] == '[')
                    {
                        // initialize to zero
                        gpu_error_check(hipMemset(_d_unique_partition[0], 0, sizeof(double)*M));

                        for(size_t d=0; d<deps.size(); d++)
                        {
                            std::string sub_dep = std::get<0>(deps[d]);
                            int sub_n_segment   = std::get<1>(deps[d]);
                            int sub_n_repeated  = std::get<2>(deps[d]);

                            // check sub key
                            #ifndef NDEBUG
                            if (d_unique_partition.find(sub_dep) == d_unique_partition.end())
                                throw_with_line_number("Could not find sub key '" + sub_dep + "'. ");
                            if (!unique_partition_finished[sub_dep][sub_n_segment-1])
                                throw_with_line_number("Could not compute '" + key +  "', since '"+ sub_dep + std::to_string(sub_n_segment) + "' is not prepared.");
                            #endif

                            lin_comb<<<N_BLOCKS, N_THREADS>>>(
                                _d_unique_partition[0], 1.0, _d_unique_partition[0],
                                sub_n_repeated, d_unique_partition[sub_dep][sub_n_segment-1], M);
                        }
                        one_step_1(_d_unique_partition[0],
                               _d_unique_partition[0],
                               d_boltz_bond[monomer_type],
                               d_exp_dw[monomer_type]);   

                        #ifndef NDEBUG
                        unique_partition_finished[key][0] = true;
                        #endif
                    }
                    else
                    {
                                                // Illustration (four branches)
                        //     A
                        //     |
                        // O - . - B
                        //     |
                        //     C

                        // Legend)
                        // .       : junction
                        // O       : full segment
                        // -, |    : half bonds
                        // A, B, C : other full segments

                        // combine branches
                        gpu_error_check(hipMemcpy(d_q_junction, q_uniform, sizeof(double)*M, hipMemcpyHostToDevice));

                        for(size_t d=0; d<deps.size(); d++)
                        {
                            std::string sub_dep = std::get<0>(deps[d]);
                            int sub_n_segment   = std::get<1>(deps[d]);

                            // check sub key
                            #ifndef NDEBUG
                            if (d_unique_partition.find(sub_dep) == d_unique_partition.end())
                                throw_with_line_number("Could not find sub key '" + sub_dep + "'. ");
                            if (!unique_partition_finished[sub_dep][sub_n_segment-1])
                                throw_with_line_number("Could not compute '" + key +  "', since '"+ sub_dep + std::to_string(sub_n_segment) + "' is not prepared.");
                            #endif

                            half_bond_step(d_unique_partition[sub_dep][sub_n_segment-1],
                                d_q_half_step, d_boltz_bond_half[mx->get_unique_branch(sub_dep).monomer_type]);

                            multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_junction, d_q_junction, d_q_half_step, 1.0, M);
                        }
                        gpu_error_check(hipMemcpy(d_unique_q_junctions[key], d_q_junction, sizeof(double)*M, hipMemcpyDeviceToDevice));

                        // add half bond
                        half_bond_step(d_q_junction, _d_unique_partition[0], d_boltz_bond_half[monomer_type]);

                        // add full segment
                        multi_real<<<N_BLOCKS, N_THREADS>>>(_d_unique_partition[0], _d_unique_partition[0], d_exp_dw[monomer_type], 1.0, M);

                        #ifndef NDEBUG
                        unique_partition_finished[key][0] = true;
                        #endif
                    }
                }
                else
                {
                    int n = n_segment_from-1;

                    #ifndef NDEBUG
                    if (!unique_partition_finished[key][n-1])
                        throw_with_line_number("unfinished, key: " + key + ", " + std::to_string(n-1));
                    #endif

                    one_step_1(_d_unique_partition[n-1],
                               _d_unique_partition[n],
                               d_boltz_bond[monomer_type],
                               d_exp_dw[monomer_type]);

                    #ifndef NDEBUG
                    unique_partition_finished[key][n] = true;
                    #endif
                }
            }
                
            // apply the propagator successively
            if(parallel_job->size()==1)
            {
                auto& key = std::get<0>((*parallel_job)[0]);
                int n_segment_from = std::get<1>((*parallel_job)[0]);
                int n_segment_to = std::get<2>((*parallel_job)[0]);
                auto monomer_type = mx->get_unique_branch(key).monomer_type;
                double **_d_unique_partition_key = d_unique_partition[key];

                for(int n=n_segment_from; n<n_segment_to; n++)
                {
                    #ifndef NDEBUG
                    if (!unique_partition_finished[key][n-1])
                        throw_with_line_number("unfinished, key: " + key + ", " + std::to_string(n-1));
                    #endif

                    one_step_1(_d_unique_partition_key[n-1],
                            _d_unique_partition_key[n],
                            d_boltz_bond[monomer_type],
                            d_exp_dw[monomer_type]);

                    #ifndef NDEBUG
                    unique_partition_finished[key][n] = true;
                    #endif
                }
            }
            else if(parallel_job->size()==2)
            {
                auto& key_1 = std::get<0>((*parallel_job)[0]);
                int n_segment_from_1 = std::get<1>((*parallel_job)[0]);
                int n_segment_to_1 = std::get<2>((*parallel_job)[0]);
                auto species_1 = mx->get_unique_branch(key_1).monomer_type;

                auto& key_2 = std::get<0>((*parallel_job)[1]);
                int n_segment_from_2 = std::get<1>((*parallel_job)[1]);
                int n_segment_to_2 = std::get<2>((*parallel_job)[1]);
                auto species_2 = mx->get_unique_branch(key_2).monomer_type;

                double **_d_unique_partition_key_1 = d_unique_partition[key_1];
                double **_d_unique_partition_key_2 = d_unique_partition[key_2];

                for(int n=0; n<n_segment_to_1-n_segment_from_1; n++)
                {
                    #ifndef NDEBUG
                    if (!unique_partition_finished[key_1][n-1+n_segment_from_1])
                        throw_with_line_number("unfinished, key: " + key_1 + ", " + std::to_string(n-n_segment_from_1));
                    if (!unique_partition_finished[key_2][n-1+n_segment_from_2])
                        throw_with_line_number("unfinished, key: " + key_2 + ", " + std::to_string(n-n_segment_from_2));
                    #endif

                    one_step_2(
                        _d_unique_partition_key_1[n-1+n_segment_from_1],
                        _d_unique_partition_key_2[n-1+n_segment_from_2],
                        _d_unique_partition_key_1[n+n_segment_from_1],
                        _d_unique_partition_key_2[n+n_segment_from_2],
                        d_boltz_bond[species_1],
                        d_boltz_bond[species_2],
                        d_exp_dw[species_1],
                        d_exp_dw[species_2]);

                    #ifndef NDEBUG
                    unique_partition_finished[key_1][n+n_segment_from_1] = true;
                    unique_partition_finished[key_2][n+n_segment_from_2] = true;
                    #endif

                    // std::cout << "finished, key, n: " + key_1 + ", " << std::to_string(n+n_segment_from_1) << std::endl;
                    // std::cout << "finished, key, n: " + key_2 + ", " << std::to_string(n+n_segment_from_2) << std::endl;
                }
            }
        }

        // compute total partition function of each distinct polymers
        int current_p = 0;
        for(const auto& block: d_unique_phi)
        {
            int p                = std::get<0>(block.first);
            std::string dep_v    = std::get<1>(block.first);
            std::string dep_u    = std::get<2>(block.first);

            // already computed
            if (p != current_p)
                continue;

            int n_superposed;
            // int n_segment_allocated = mx->get_unique_block(block.first).n_segment_allocated;
            int n_segment_offset    = mx->get_unique_block(block.first).n_segment_offset;
            int n_segment_original  = mx->get_unique_block(block.first).n_segment_original;
            std::string monomer_type = mx->get_unique_block(block.first).monomer_type;

            // contains no '['
            if (dep_u.find('[') == std::string::npos)
                n_superposed = 1;
            else
                n_superposed = mx->get_unique_block(block.first).v_u.size();

            // check keys
            #ifndef NDEBUG
            if (d_unique_partition.find(dep_v) == d_unique_partition.end())
                throw_with_line_number("Could not find dep_v key'" + dep_v + "'. ");
            if (d_unique_partition.find(dep_u) == d_unique_partition.end())
                throw_with_line_number("Could not find dep_u key'" + dep_u + "'. ");
            #endif

            single_partitions[p] = ((CudaComputationBox *)cb)->inner_product_inverse_weight_gpu(
                d_unique_partition[dep_v][n_segment_original-n_segment_offset-1],  // q
                d_unique_partition[dep_u][0],                                      // q^dagger
                d_exp_dw[monomer_type])/n_superposed/cb->get_volume();        

            // std::cout << p <<", "<< dep_v <<", "<< dep_u <<", "<< n_segment <<", " << single_partitions[p] << std::endl;
            // std::cout << p <<", "<< n_segment <<", "<< n_segment_offset <<", "<< single_partitions[p] << std::endl;
            current_p++;
        }

        // calculate segment concentrations
        for(const auto& block: d_unique_phi)
        {
            int p                = std::get<0>(block.first);
            std::string dep_v    = std::get<1>(block.first);
            std::string dep_u    = std::get<2>(block.first);

            int n_repeated;
            int n_segment_allocated = mx->get_unique_block(block.first).n_segment_allocated;
            int n_segment_offset    = mx->get_unique_block(block.first).n_segment_offset;
            int n_segment_original  = mx->get_unique_block(block.first).n_segment_original;
            std::string monomer_type = mx->get_unique_block(block.first).monomer_type;

            // contains no '['
            if (dep_u.find('[') == std::string::npos)
                n_repeated = mx->get_unique_block(block.first).v_u.size();
            else
                n_repeated = 1;

            // check keys
            #ifndef NDEBUG
            if (d_unique_partition.find(dep_v) == d_unique_partition.end())
                throw_with_line_number("Could not find dep_v key'" + dep_v + "'. ");
            if (d_unique_partition.find(dep_u) == d_unique_partition.end())
                throw_with_line_number("Could not find dep_u key'" + dep_u + "'. ");
            #endif

            // calculate phi of one block (possibly multiple blocks when using superposition)
            calculate_phi_one_block(
                block.second,             // phi
                d_unique_partition[dep_v],  // dependency v
                d_unique_partition[dep_u],  // dependency u
                d_exp_dw[monomer_type],     // exp_dw
                n_segment_allocated,
                n_segment_offset,
                n_segment_original);
            
            // normalize concentration
            PolymerChain& pc = mx->get_polymer(p);
            double norm = mx->get_ds()*pc.get_volume_fraction()/pc.get_alpha()/single_partitions[p]*n_repeated;
            lin_comb<<<N_BLOCKS, N_THREADS>>>(block.second, norm, block.second, 0.0, block.second, M);
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoDiscrete::one_step_1(
    double *d_q_in, double *d_q_out,
    double *d_boltz_bond, double *d_exp_dw)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        //-------------- step 1 ----------
        // Execute a Forward FFT
        hipfftExecD2Z(plan_for_1, d_q_in, d_qk_in_1);

        // Multiply e^(-k^2 ds/6) in fourier space
        multi_complex_real<<<N_BLOCKS, N_THREADS>>>(d_qk_in_1, d_boltz_bond, M_COMPLEX);

        // Execute a backward FFT
        hipfftExecZ2D(plan_bak_1, d_qk_in_1, d_q_out);

        // Evaluate e^(-w*ds) in real space
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_out, d_q_out, d_exp_dw, 1.0/((double)M), M);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoDiscrete::one_step_2(
    double *d_q_in_1, double *d_q_in_2,
    double *d_q_out_1, double *d_q_out_2,
    double *d_boltz_bond_1, double *d_boltz_bond_2,  
    double *d_exp_dw_1, double *d_exp_dw_2)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        gpu_error_check(hipMemcpy(&d_q_in_temp_2[0], d_q_in_1, sizeof(double)*M, hipMemcpyDeviceToDevice));
        gpu_error_check(hipMemcpy(&d_q_in_temp_2[M], d_q_in_2, sizeof(double)*M, hipMemcpyDeviceToDevice));

        //-------------- step 1 ----------
        // Execute a Forward FFT
        hipfftExecD2Z(plan_for_2, d_q_in_temp_2, d_qk_in_2);

        // Multiply e^(-k^2 ds/6) in fourier space
        multi_complex_real<<<N_BLOCKS, N_THREADS>>>(&d_qk_in_2[0],         d_boltz_bond_1, M_COMPLEX);
        multi_complex_real<<<N_BLOCKS, N_THREADS>>>(&d_qk_in_2[M_COMPLEX], d_boltz_bond_2, M_COMPLEX);

        // Execute a backward FFT
        hipfftExecZ2D(plan_bak_2, d_qk_in_2, d_q_out_temp_2);

        // Evaluate e^(-w*ds) in real space
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_out_1, &d_q_out_temp_2[0], d_exp_dw_1, 1.0/((double)M), M);
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_out_2, &d_q_out_temp_2[M], d_exp_dw_2, 1.0/((double)M), M);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoDiscrete::half_bond_step(double *d_q_in, double *d_q_out, double *d_boltz_bond_half)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        // 3D fourier discrete transform, forward and inplace
        hipfftExecD2Z(plan_for_1, d_q_in, d_qk_in_1);
        // multiply e^(-k^2 ds/12) in fourier space, in all 3 directions
        multi_complex_real<<<N_BLOCKS, N_THREADS>>>(d_qk_in_1, d_boltz_bond_half, 1.0/((double)M), M_COMPLEX);
        // 3D fourier discrete transform, backward and inplace
        hipfftExecZ2D(plan_bak_1, d_qk_in_1, d_q_out);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoDiscrete::calculate_phi_one_block(
    double *d_phi, double **d_q_1, double **d_q_2, double *d_exp_dw, const int N, const int N_OFFSET, const int N_ORIGINAL)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        // Compute segment concentration
        multi_real<<<N_BLOCKS, N_THREADS>>>(d_phi,d_q_1[N_ORIGINAL-N_OFFSET-1], d_q_2[0], 1.0, M);
        for(int n=1; n<N; n++)
        {
            add_multi_real<<<N_BLOCKS, N_THREADS>>>(d_phi, d_q_1[N_ORIGINAL-N_OFFSET-n-1], d_q_2[n], 1.0, M);
        }
        divide_real<<<N_BLOCKS, N_THREADS>>>(d_phi, d_phi, d_exp_dw, 1.0, M);
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
double CudaPseudoDiscrete::get_total_partition(int polymer)
{
    try
    {
        return single_partitions[polymer];
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoDiscrete::get_monomer_concentration(std::string monomer_type, double *phi)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        // initialize to zero
        gpu_error_check(hipMemset(d_phi, 0, sizeof(double)*M));

        // for each block
        for(const auto& block: d_unique_phi)
        {
            std::string dep_v = std::get<1>(block.first);
            int n_segment_allocated = mx->get_unique_block(block.first).n_segment_allocated;
            if (Mixture::get_monomer_type_from_key(dep_v) == monomer_type && n_segment_allocated != 0)
                lin_comb<<<N_BLOCKS, N_THREADS>>>(d_phi, 1.0, d_phi, 1.0, block.second, M);
        }
        gpu_error_check(hipMemcpy(phi, d_phi, sizeof(double)*M, hipMemcpyDeviceToHost));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoDiscrete::get_polymer_concentration(int p, double *phi)
{
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int M = cb->get_n_grid();
        const int P = mx->get_n_polymers();

        if (p < 0 || p > P-1)
            throw_with_line_number("Index (" + std::to_string(p) + ") must be in range [0, " + std::to_string(P-1) + "]");

        if (mx->is_using_superposition())
            throw_with_line_number("Disable 'superposition' option to obtain concentration of each block.");

        PolymerChain& pc = mx->get_polymer(p);
        std::vector<PolymerChainBlock>& blocks = pc.get_blocks();

        for(size_t b=0; b<blocks.size(); b++)
        {
            std::string dep_v = pc.get_dep(blocks[b].v, blocks[b].u);
            std::string dep_u = pc.get_dep(blocks[b].u, blocks[b].v);
            if (dep_v < dep_u)
                dep_v.swap(dep_u);

            // copy normalized concentration
            double norm = cb->get_volume()*mx->get_ds()*pc.get_volume_fraction()/pc.get_alpha()/single_partitions[p];
            lin_comb<<<N_BLOCKS, N_THREADS>>>(d_phi, 0.0, d_phi, norm, d_unique_phi[std::make_tuple(p, dep_v, dep_u)], M);
            gpu_error_check(hipMemcpy(&phi[b*M], d_phi, sizeof(double)*M, hipMemcpyDeviceToHost));
        }
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
std::vector<double> CudaPseudoDiscrete::compute_stress()
{
    // This method should be invoked after invoking compute_statistics().

    // To calculate stress, we multiply weighted fourier basis to q(k)*q^dagger(-k).
    // We only need the real part of stress calculation.
    try
    {
        const int N_BLOCKS  = CudaCommon::get_instance().get_n_blocks();
        const int N_THREADS = CudaCommon::get_instance().get_n_threads();

        const int DIM  = cb->get_dim();
        const int M    = cb->get_n_grid();
        const int M_COMPLEX = this->n_complex_grid;

        auto bond_lengths = mx->get_bond_lengths();
        std::vector<double> stress(cb->get_dim());
        std::map<std::tuple<int, std::string, std::string>, std::array<double,3>> unique_dq_dl;
        thrust::device_ptr<double> temp_gpu_ptr(d_stress_sum);

        // reset stress map
        for(const auto& item: d_unique_phi)
        {
            for(int d=0; d<3; d++)
                unique_dq_dl[item.first][d] = 0.0;
        }

        // compute stress for unique block
        for(const auto& block: d_unique_phi)
        {
            const auto& key      = block.first;
            int p                = std::get<0>(key);
            std::string dep_v    = std::get<1>(key);
            std::string dep_u    = std::get<2>(key);

            const int N           = mx->get_unique_block(block.first).n_segment_allocated;
            const int N_OFFSET    = mx->get_unique_block(block.first).n_segment_offset;
            const int N_ORIGINAL  = mx->get_unique_block(block.first).n_segment_original;
            std::string monomer_type = mx->get_unique_block(key).monomer_type;

            // contains no '['
            int n_repeated;
            if (dep_u.find('[') == std::string::npos)
                n_repeated = mx->get_unique_block(block.first).v_u.size();
            else
                n_repeated = 1;

            double **d_q_1 = d_unique_partition[dep_v];    // dependency v
            double **d_q_2 = d_unique_partition[dep_u];    // dependency u

            double bond_length_sq;
            double *d_boltz_bond_now;

            std::array<double,3> _unique_dq_dl = unique_dq_dl[key];

            // std::cout << "dep_v: " << dep_v << std::endl;
            // std::cout << "dep_u: " << dep_u << std::endl;

            // compute stress
            for(int n=0; n<=N; n++)
            {
                // at v
                if (n + N_OFFSET == N_ORIGINAL)
                {
                    // std::cout << "case 1: " << unique_q_junctions[dep_v][0] << ", " << q_2[(N-1)*M] << std::endl;
                    if (mx->get_unique_branch(dep_v).deps.size() == 0) // if v is leaf node, skip
                        continue;
                    
                    gpu_error_check(hipMemcpy(&d_q_in_temp_2[0], d_unique_q_junctions[dep_v], sizeof(double)*M, hipMemcpyDeviceToDevice));
                    gpu_error_check(hipMemcpy(&d_q_in_temp_2[M], d_q_2[N-1],                  sizeof(double)*M, hipMemcpyDeviceToDevice));

                    bond_length_sq = 0.5*bond_lengths[monomer_type]*bond_lengths[monomer_type];
                    d_boltz_bond_now = d_boltz_bond_half[monomer_type];
                }
                // at u
                else if (n + N_OFFSET == 0){
                    // std::cout << "case 2: " << q_1[(N_ORIGINAL-N_OFFSET-1)*M] << ", " << unique_q_junctions[dep_u][0] << std::endl;
                    if (mx->get_unique_branch(dep_u).deps.size() == 0) // if u is leaf node, skip
                        continue;

                    gpu_error_check(hipMemcpy(&d_q_in_temp_2[0], d_q_1[N_ORIGINAL-1],         sizeof(double)*M, hipMemcpyDeviceToDevice));
                    gpu_error_check(hipMemcpy(&d_q_in_temp_2[M], d_unique_q_junctions[dep_u], sizeof(double)*M, hipMemcpyDeviceToDevice));
                    bond_length_sq = 0.5*bond_lengths[monomer_type]*bond_lengths[monomer_type];
                    d_boltz_bond_now = d_boltz_bond_half[monomer_type];
                }
                // at superposition junction
                else if (n == 0)
                {
                    // std::cout << "case 4" << std::endl;
                    continue;
                }
                // within the blocks
                else
                {
                    gpu_error_check(hipMemcpy(&d_q_in_temp_2[0], d_q_1[N_ORIGINAL-N_OFFSET-n-1], sizeof(double)*M, hipMemcpyDeviceToDevice));
                    gpu_error_check(hipMemcpy(&d_q_in_temp_2[M], d_q_2[n-1],                     sizeof(double)*M, hipMemcpyDeviceToDevice));
                    bond_length_sq = bond_lengths[monomer_type]*bond_lengths[monomer_type];
                    d_boltz_bond_now = d_boltz_bond[monomer_type];
                }

                // execute a Forward FFT
                hipfftExecD2Z(plan_for_2, d_q_in_temp_2, d_qk_in_2);

                // multiplay two partial partition functions in the fourier spaces
                multi_complex_conjugate<<<N_BLOCKS, N_THREADS>>>(d_q_multi, &d_qk_in_2[0], &d_qk_in_2[M_COMPLEX], M_COMPLEX);

                multi_real<<<N_BLOCKS, N_THREADS>>>(d_q_multi, d_q_multi, d_boltz_bond_now, bond_length_sq, M_COMPLEX);
                if ( DIM == 3 )
                {
                    multi_real<<<N_BLOCKS, N_THREADS>>>(d_stress_sum, d_q_multi, d_fourier_basis_x, 1.0, M_COMPLEX);
                    _unique_dq_dl[0] += thrust::reduce(temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX)*n_repeated;

                    multi_real<<<N_BLOCKS, N_THREADS>>>(d_stress_sum, d_q_multi, d_fourier_basis_y, 1.0, M_COMPLEX);
                    _unique_dq_dl[1] += thrust::reduce(temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX)*n_repeated;

                    multi_real<<<N_BLOCKS, N_THREADS>>>(d_stress_sum, d_q_multi, d_fourier_basis_z, 1.0, M_COMPLEX);
                    _unique_dq_dl[2] += thrust::reduce(temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX)*n_repeated;
                }
                if ( DIM == 2 )
                {
                    multi_real<<<N_BLOCKS, N_THREADS>>>(d_stress_sum, d_q_multi, d_fourier_basis_y, 1.0, M_COMPLEX);
                    _unique_dq_dl[0] += thrust::reduce(temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX)*n_repeated;

                    multi_real<<<N_BLOCKS, N_THREADS>>>(d_stress_sum, d_q_multi, d_fourier_basis_z, 1.0, M_COMPLEX);
                    _unique_dq_dl[1] += thrust::reduce(temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX)*n_repeated;
                }
                if ( DIM == 1 )
                {
                    multi_real<<<N_BLOCKS, N_THREADS>>>(d_stress_sum, d_q_multi, d_fourier_basis_z, 1.0, M_COMPLEX);
                    _unique_dq_dl[0] += thrust::reduce(temp_gpu_ptr, temp_gpu_ptr + M_COMPLEX)*n_repeated;
                }
            }
            unique_dq_dl[key] = _unique_dq_dl;
        }

        // compute total stress
        for(int d=0; d<cb->get_dim(); d++)
            stress[d] = 0.0;
        for(const auto& block: d_unique_phi)
        {
            const auto& key      = block.first;
            int p                = std::get<0>(key);
            std::string dep_v    = std::get<1>(key);
            std::string dep_u    = std::get<2>(key);
            PolymerChain& pc = mx->get_polymer(p);

            for(int d=0; d<cb->get_dim(); d++)
                stress[d] += unique_dq_dl[key][d]*pc.get_volume_fraction()/pc.get_alpha()/single_partitions[p];
        }
        for(int d=0; d<cb->get_dim(); d++)
            stress[d] /= -3.0*cb->get_lx(d)*M*M/mx->get_ds();
            
        return stress;
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
void CudaPseudoDiscrete::get_partial_partition(double *q_out, int polymer, int v, int u, int n)
{ 
    // This method should be invoked after invoking compute_statistics()

    // Get partial partition functions
    // This is made for debugging and testing
    try
    {
        const int M = cb->get_n_grid();
        PolymerChain& pc = mx->get_polymer(polymer);
        std::string dep = pc.get_dep(v,u);

        if (mx->get_unique_branches().find(dep) == mx->get_unique_branches().end())
            throw_with_line_number("Could not find the branches '" + dep + "'. Disable 'superposition' option to obtain partial partition functions.");

        const int N = mx->get_unique_branches()[dep].max_n_segment;
        if (n < 1 || n > N)
            throw_with_line_number("n (" + std::to_string(n) + ") must be in range [1, " + std::to_string(N) + "]");

        gpu_error_check(hipMemcpy(q_out, d_unique_partition[dep][n-1], sizeof(double)*M,hipMemcpyDeviceToHost));
    }
    catch(std::exception& exc)
    {
        throw_without_line_number(exc.what());
    }
}
